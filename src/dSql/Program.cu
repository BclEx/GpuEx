#include "hip/hip_runtime.h"
//#define VISUAL
#pragma region PREAMBLE

#if (defined(_WIN32) || defined(WIN32)) && !defined(_CRT_SECURE_NO_WARNINGS)
#define _CRT_SECURE_NO_WARNINGS // This needs to come before any includes for MSVC compiler
#endif

// Enable large-file support for fopen() and friends on unix.
#ifndef DISABLE_LFS
#define _LARGE_FILE       1
#ifndef _FILE_OFFSET_BITS
#define _FILE_OFFSET_BITS 64
#endif
#define _LARGEFILE_SOURCE 1
#endif

#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <assert.h>
#include "..\System.Data.net\Core+Vdbe\VdbeInt.cu.h"
#include <ctype.h>
#include <stdarg.h>

#if !defined(_WIN32) && !defined(WIN32)
#include <signal.h>
#if !defined(__RTP__) && !defined(_WRS_KERNEL)
#include <pwd.h>
#endif
#include <unistd.h>
#include <sys/types.h>
#endif

#ifdef HAVE_EDITLINE
#include <editline/editline.h>
#endif
#if defined(HAVE_READLINE) && HAVE_READLINE==1
#include <readline/readline.h>
#include <readline/history.h>
#endif
#if !defined(HAVE_EDITLINE) && (!defined(HAVE_READLINE) || HAVE_READLINE!=1)
#define readline(p) LocalGetLine(p, stdin, 0)
#define add_history(X)
#define read_history(X)
#define write_history(X)
#define stifle_history(X)
#endif

#if defined(_WIN32) || defined(WIN32)
#include <io.h>
#define isatty(h) _isatty(h)
#define access(f,m) _access((f),(m))
#undef popen
#define popen(a,b) _popen((a),(b))
#undef pclose
#define pclose(x) _pclose(x)
#else
extern int isatty(int); // Make sure isatty() has a prototype.
#endif
#if defined(_WIN32_WCE)
// Windows CE (arm-wince-mingw32ce-gcc) does not provide isatty() thus we always assume that we have a console. That can be overridden with the -batch command line option.
#define isatty(x) 1
#endif

// ctype macros that work with signed characters
#define IsSpace(X)  _isspace((unsigned char)X)
#define IsDigit(X)  _isdigit((unsigned char)X)
#define ToLower(X)  (char)_tolower((unsigned char)X)
#pragma endregion

#pragma region TIMER
static bool _enableTimer = false; // True if the timer is enabled

#if !defined(_WIN32) && !defined(WIN32) && !defined(_WRS_KERNEL) && !defined(__minux)
#include <sys/time.h>
#include <sys/resource.h>

static struct rusage _sBegin; // Saved resource information for the beginning of an operation
static void BeginTimer()
{
	if (_enableTimer)
		getrusage(RUSAGE_SELF, &_sBegin);
}

static double TimeDiff(timeval *start, timeval *end)
{
	return (end->tv_usec - start->tv_usec)*0.000001 + (double)(end->tv_sec - start->tv_sec);
}

static void EndTimer()
{
	if (_enableTimer)
	{
		rusage sEnd;
		getrusage(RUSAGE_SELF, &sEnd);
		printf("CPU Time: user %f sys %f\n",
			TimeDiff(&_sBegin.ru_utime, &sEnd.ru_utime),
			TimeDiff(&_sBegin.ru_stime, &sEnd.ru_stime));
	}
}

#define BEGIN_TIMER BeginTimer()
#define END_TIMER EndTimer()
#define HAS_TIMER 1

#elif (defined(_WIN32) || defined(WIN32))

#include <windows.h>

// Saved resource information for the beginning of an operation
static HANDLE _hProcess;
static FILETIME _ftKernelBegin;
static FILETIME _ftUserBegin;
typedef BOOL (WINAPI *GETPROCTIMES)(HANDLE, LPFILETIME, LPFILETIME, LPFILETIME, LPFILETIME);
static GETPROCTIMES _getProcessTimesAddr = nullptr;

static int HasTimer()
{
	if (_getProcessTimesAddr)
		return true;
	// GetProcessTimes() isn't supported in WIN95 and some other Windows versions. See if the version we are running on has it, and if it does, save off
	// a pointer to it and the current process handle.
	_hProcess = GetCurrentProcess();
	if (_hProcess)
	{
		HINSTANCE hinstLib = LoadLibrary(TEXT("Kernel32.dll"));
		if (hinstLib)
		{
			_getProcessTimesAddr = (GETPROCTIMES)GetProcAddress(hinstLib, "GetProcessTimes");
			if (_getProcessTimesAddr)
				return true;
			FreeLibrary(hinstLib); 
		}
	}
	return false;
}

static void BeginTimer()
{
	if (_enableTimer && _getProcessTimesAddr)
	{
		FILETIME ftCreation, ftExit;
		_getProcessTimesAddr(_hProcess, &ftCreation, &ftExit, &_ftKernelBegin, &_ftUserBegin);
	}
}

static double TimeDiff(FILETIME *start, FILETIME *end)
{
	int64 i64Start = *((int64 *)start);
	int64 i64End = *((int64 *)end);
	return (double)((i64End - i64Start) / 10000000.0);
}

static void EndTimer()
{
	if (_enableTimer && _getProcessTimesAddr)
	{
		FILETIME ftCreation, ftExit, ftKernelEnd, ftUserEnd;
		_getProcessTimesAddr(_hProcess, &ftCreation, &ftExit, &ftKernelEnd, &ftUserEnd);
		printf("CPU Time: user %f sys %f\n", TimeDiff(&_ftUserBegin, &ftUserEnd), TimeDiff(&_ftKernelBegin, &ftKernelEnd));
	}
}

#define BEGIN_TIMER BeginTimer()
#define END_TIMER EndTimer()
#define HAS_TIMER HasTimer()
#else
#define BEGIN_TIMER 
#define END_TIMER
#define HAS_TIMER false
#endif
#pragma endregion

namespace Core
{
	static int _bailOnError = 0; // If the following flag is set, then command execution stops at an error if we are not interactive.
	static int _stdinIsInteractive = 1; // Threat stdin as an interactive input if the following variable is true.  Otherwise, assume stdin is connected to a file or pipe.
	static Context *_ctx = nullptr; // The following is the open SQLite database.  We make a pointer to this database a static variable so that it can be accessed by the SIGINT handler to interrupt database processing.
	static volatile int _seenInterrupt = 0; // True if an interrupt (Control-C) has been received.
	static char *Argv0; // This is the name of our program. It is set in main(), used in a number of other places, mostly for error messages.

	// Prompt strings. Initialized in main. Settable with .prompt main continue
	static char _mainPrompt[20];     // First line prompt. default: "sqlite> "
	static char _continuePrompt[20]; // Continuation prompt. default: "   ...> "

#pragma region Name

	// Write I/O traces to the following stream.
#ifdef ENABLE_IOTRACE
	static FILE *iotrace = nullptr;

	static void iotracePrintf(const char *fmt, ...)
	{
		if (!iotrace) return;
		va_list args;
		va_start(args, fmt);
		char *z = _vmprintf(fmt, ap);
		va_end(ap);
		_fprintf(iotrace, "%s", z);
		_free(z);
	}
#endif

	static bool isNumber(const char *z, int *realnum)
	{
		if (*z == '-' || *z == '+') z++;
		if (!IsDigit(*z))
			return 0;
		z++;
		if (realnum) *realnum = 0;
		while (IsDigit(*z)) { z++; }
		if (*z == '.')
		{
			z++;
			if (!IsDigit(*z)) return 0;
			while (IsDigit(*z)) { z++; }
			if (realnum) *realnum = 1;
		}
		if (*z == 'e' || *z == 'E')
		{
			z++;
			if (*z == '+' || *z == '-') z++;
			if (!IsDigit(*z)) return 0;
			while (IsDigit(*z)) { z++; }
			if (realnum) *realnum = 1;
		}
		return (*z == 0);
	}

	static const char *_shellStatic = nullptr;
	static void ShellStaticFunc(FuncContext *fctx, int argc, Mem **argv)
	{
		_assert(argc == 0);
		_assert(_shellStatic);
		Vdbe::Result_Text(fctx, _shellStatic, -1, DESTRUCTOR_STATIC);
	}

	static char *LocalGetLine(char *prompt, FILE *in, int csvFlag)
	{
		if (prompt && *prompt)
		{
			printf("%s", prompt);
			_fflush(stdout);
		}
		int lineLength = 100;
		char *line = (char *)malloc(lineLength);
		if (!line) return nullptr;
		int n = 0;
		bool inQuote = false;
		while (1)
		{
			if (n+100 > lineLength)
			{
				lineLength = lineLength*2 + 100;
				line = (char *)realloc(line, lineLength);
				if (!line) return nullptr;
			}
			if (!fgets(&line[n], lineLength - n, in))
			{
				if (n == 0)
				{
					free(line);
					return nullptr;
				}
				line[n] = 0;
				break;
			}
			while (line[n])
			{
				if (line[n] == '"') inQuote = !inQuote;
				n++;
			}
			if (n > 0 && line[n-1] == '\n' && (!inQuote || !csvFlag))
			{
				n--;
				if (n > 0 && line[n-1] == '\r') n--;
				line[n] = 0;
				break;
			}
		}
		line = (char *)realloc(line, n+1);
		return line;
	}

	static char *OneInputLine(const char *prior, FILE *in)
	{
		if (in != nullptr)
			return LocalGetLine(nullptr, in, 0);
		char *prompt = (prior && prior[0] ? _continuePrompt : _mainPrompt);
		char *result = readline(prompt);
#if defined(HAVE_READLINE) && HAVE_READLINE==1
		if (result && *result) AddHistory(result);
#endif
		return result;
	}

	struct PreviousModeData
	{
		int Valid;        // Is there legit data in here?
		int Mode;
		int ShowHeader;
		int ColWidth[100];
	};

	enum MODE : uint8
	{
		MODE_Line     = 0,		// One column per line.  Blank line between records
		MODE_Column   = 1,		// One record per line in neat columns
		MODE_List     = 2,		// One record per line with a separator
		MODE_Semi     = 3,		// Same as MODE_List but append ";" to each line
		MODE_Html     = 4,		// Generate an XHTML table
		MODE_Insert   = 5,		// Generate SQL "insert" statements
		MODE_Tcl      = 6,		// Generate ANSI-C or TCL quoted elements
		MODE_Csv      = 7,		// Quote strings, numbers are plain
		MODE_Explain  = 8,		// Like MODE_Column, but do not truncate data
	};

	static const char *modeDescr[] =
	{
		"line",
		"column",
		"list",
		"semi",
		"html",
		"insert",
		"tcl",
		"csv",
		"explain",
	};

	struct CallbackData
	{
		Context *Ctx;				// The database
		int EchoOn;					// True to echo input commands
		int StatsOn;				// True to display memory stats before each finalize
		int Cnt;					// Number of records displayed so far
		FILE *Out;					// Write results here
		FILE *TraceOut;				// Output for sqlite3_trace()
		int Errs;					// Number of errors seen
		MODE Mode;					// An output mode setting
		int WritableSchema;			// True if PRAGMA writable_schema=ON
		int ShowHeader;				// True to show column names in List or Column mode
		char *DestTable;			// Name of destination table when MODE_Insert
		char Separator[20];			// Separator character for MODE_List
		int ColWidth[100];			// Requested width of each column when in column mode
		int ActualWidth[100];		// Actual width of each column
		char NullValue[20];			// The text to print when a NULL comes back from the database
		struct PreviousModeData ExplainPrev;
		// Holds the mode information just before .explain ON
		char Outfile[FILENAME_MAX]; // Filename for *out_
		const char *DbFilename;		// name of the database file
		const char *Vfs;			// Name of VFS to use
		Vdbe *Stmt;					// Current statement if any.
		FILE *Log;					// Write log output here
	};

	//static int _strlen(const char *z)
	//{
	//	const char *z2 = z;
	//	while (*z2) { z2++; }
	//	return 0x3fffffff & (int)(z2 - z);
	//}

	static void ShellLog(void *arg, int errCode, const char *msg)
	{
		struct CallbackData *p = (struct CallbackData*)arg;
		if (!p->Log) return;
		_fprintf(p->Log, "(%d) %s\n", errCode, msg);
		_fflush(p->Log);
	}

#pragma endregion

#pragma region Output

	static void OutputHexBlob(FILE *out_, const void *blob, int blobLength)
	{
		char *blob2 = (char *)blob;
		_fprintf(out_, "X'");
		for (int i = 0; i < blobLength; i++) { _fprintf(out_, "%02x", blob2[i]&0xff); }
		_fprintf(out_, "'");
	}

	static void OutputQuotedString(FILE *out_, const char *z)
	{
		int i;
		int singles = 0;
		for (i = 0; z[i]; i++)
			if (z[i] == '\'' ) singles++;
		if (singles == 0)
			_fprintf(out_, "'%s'", z);
		else
		{
			_fprintf(out_, "'");
			while (*z)
			{
				for (i = 0; z[i] && z[i] != '\''; i++) { }
				if (i == 0) { _fprintf(out_, "''"); z++; }
				else if (z[i] == '\'') { _fprintf(out_, "%.*s''", i, z); z += i+1; }
				else { _fprintf(out_, "%s", z); break; }
			}
			_fprintf(out_, "'");
		}
	}

	static void OutputCString(FILE *out_, const char *z)
	{
		unsigned int c;
		fputc('"', out_);
		while ((c = *(z++)) != 0)
		{
			if (c == '\\') { fputc(c, out_); fputc(c, out_); }
			else if (c == '"') { fputc('\\', out_); fputc('"', out_); }
			else if (c == '\t') { fputc('\\', out_); fputc('t', out_); }
			else if (c == '\n') { fputc('\\', out_); fputc('n', out_); }
			else if (c == '\r') { fputc('\\', out_); fputc('r', out_); }
			else if (!isprint(c)) _fprintf(out_, "\\%03o", c&0xff);
			else fputc(c, out_);
		}
		fputc('"', out_);
	}

	static void OutputHtmlString(FILE *out_, const char *z)
	{
		int i;
		while (*z)
		{
			for (i = 0; z[i] && z[i] != '<' && z[i] != '&' && z[i] != '>' && z[i] != '\"' && z[i] != '\''; i++) { }
			if (i > 0) _fprintf(out_, "%.*s", i, z);
			if (z[i] == '<') _fprintf(out_,"&lt;");
			else if (z[i] == '&') _fprintf(out_,"&amp;");
			else if (z[i] == '>') _fprintf(out_,"&gt;");
			else if (z[i] == '\"') _fprintf(out_,"&quot;");
			else if (z[i] == '\'') _fprintf(out_,"&#39;");
			else break;
			z += i + 1;
		}
	}

	static const char _needCsvQuote[] = {
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 0, 1, 0, 0, 0, 0, 1,   0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0,   0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0,   0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0,   0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0,   0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0,   0, 0, 0, 0, 0, 0, 0, 1, 
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
		1, 1, 1, 1, 1, 1, 1, 1,   1, 1, 1, 1, 1, 1, 1, 1,   
	};

	static void OutputCsv(struct CallbackData *p, const char *z, bool sep)
	{
		FILE *out_ = p->Out;
		if (!z) _fprintf(out_, "%s", p->NullValue);
		else
		{
			int i;
			int sepLength = _strlen(p->Separator);
			for (i = 0; z[i]; i++)
			{
				if (_needCsvQuote[((unsigned char*)z)[i]] || (z[i] == p->Separator[0] &&  (sepLength == 1 || _memcmp(z, p->Separator, sepLength) == 0)))
				{
					i = 0;
					break;
				}
			}
			if (i == 0)
			{
				putc('"', out_);
				for (i = 0; z[i]; i++)
				{
					if (z[i] == '"') putc('"', out_);
					putc(z[i], out_);
				}
				putc('"', out_);
			}
			else _fprintf(out_, "%s", z);
		}
		if (sep) _fprintf(out_, "%s", p->Separator);
	}

#pragma endregion

#pragma region Callback

#ifdef SIGINT
	static void InterruptHandler(int notUsed)
	{
		_seenInterrupt = 1;
		if (ctx) sqlite3_interrupt(ctx);
	}
#endif

	static int ShellCallback(void *args, int argsLength, char **argNames, char **colNames, int *insertTypes)
	{
		int i;
		struct CallbackData *p = (struct CallbackData *)args;
		switch (p->Mode)
		{
		case MODE_Line: {
			int w = 5;
			if (argNames == 0) break;
			for (i = 0; i < argsLength; i++)
			{
				int len = _strlen(colNames[i] ? colNames[i] : "");
				if (len > w) w = len;
			}
			if (p->Cnt++ > 0) _fprintf(p->Out, "\n");
			for (i = 0; i < argsLength; i++)
				_fprintf(p->Out, "%*s = %s\n", w, colNames[i], argNames[i] ? argNames[i] : p->NullValue);
			break; }
		case MODE_Explain:
		case MODE_Column: {
			if (p->Cnt++ == 0)
			{
				for (i = 0; i < argsLength; i++)
				{
					int w = (i < _lengthof(p->ColWidth) ? p->ColWidth[i] : 0);
					if (w == 0)
					{
						w = _strlen(colNames[i] ? colNames[i] : "");
						if (w < 10) w = 10;
						int n = _strlen(argNames && argNames[i] ? argNames[i] : p->NullValue);
						if (w < n) w = n;
					}
					if (i < _lengthof(p->ActualWidth))
						p->ActualWidth[i] = w;
					if (p->ShowHeader)
					{
						if (w < 0) _fprintf(p->Out, "%*.*s%s", -w, -w, colNames[i], (i == argsLength-1 ? "\n" : "  "));
						else _fprintf(p->Out, "%-*.*s%s", w, w, colNames[i], (i == argsLength-1 ? "\n": "  "));
					}
				}
				if (p->ShowHeader)
					for (i = 0; i < argsLength; i++)
					{
						int w;
						if (i < _lengthof(p->ActualWidth))
						{
							w = p->ActualWidth[i];
							if (w < 0) w = -w;
						}
						else
							w = 10;
						_fprintf(p->Out, "%-*.*s%s", w, w, "---------------------------------------------------------------------------------------------", (i == argsLength-1 ? "\n" : "  "));
					}
			}
			if (argNames == 0) break;
			for (i = 0; i < argsLength; i++)
			{
				int w = (i < _lengthof(p->ActualWidth) ? p->ActualWidth[i] : 10);
				if (p->Mode == MODE_Explain && argNames[i] && _strlen(argNames[i]) > w)
					w = _strlen(argNames[i]);
				if (w < 0) _fprintf(p->Out, "%*.*s%s", -w, -w, (argNames[i] ? argNames[i] : p->NullValue), (i == argsLength-1 ? "\n" : "  "));
				else _fprintf(p->Out, "%-*.*s%s", w, w, (argNames[i] ? argNames[i] : p->NullValue), (i == argsLength-1 ? "\n" : "  "));
			}
			break; }
		case MODE_Semi:
		case MODE_List: {
			if (p->Cnt++ == 0 && p->ShowHeader)
				for (i = 0; i < argsLength; i++)
					_fprintf(p->Out, "%s%s", colNames[i], (i == argsLength-1 ? "\n" : p->Separator));
			if (argNames == 0) break;
			for (i = 0; i < argsLength; i++)
			{
				char *z = argNames[i];
				if (!z) z = p->NullValue;
				_fprintf(p->Out, "%s", z);
				if (i < argsLength-1) _fprintf(p->Out, "%s", p->Separator);
				else if (p->Mode == MODE_Semi) _fprintf(p->Out, ";\n");
				else _fprintf(p->Out, "\n");
			}
			break; }
		case MODE_Html: {
			if (p->Cnt++ == 0 && p->ShowHeader)
			{
				_fprintf(p->Out, "<TR>");
				for (i = 0; i < argsLength; i++)
				{
					_fprintf(p->Out, "<TH>");
					OutputHtmlString(p->Out, colNames[i]);
					_fprintf(p->Out, "</TH>\n");
				}
				_fprintf(p->Out, "</TR>\n");
			}
			if (argNames == 0) break;
			_fprintf(p->Out, "<TR>");
			for (i = 0; i < argsLength; i++)
			{
				_fprintf(p->Out, "<TD>");
				OutputHtmlString(p->Out, (argNames[i] ? argNames[i] : p->NullValue));
				_fprintf(p->Out, "</TD>\n");
			}
			_fprintf(p->Out, "</TR>\n");
			break; }
		case MODE_Tcl: {
			if (p->Cnt++ == 0 && p->ShowHeader)
			{
				for (i = 0; i < argsLength; i++)
				{
					OutputCString(p->Out, (colNames[i] ? colNames[i] : ""));
					if (i < argsLength-1) _fprintf(p->Out, "%s", p->Separator);
				}
				_fprintf(p->Out, "\n");
			}
			if (argNames == 0) break;
			for (i = 0; i < argsLength; i++)
			{
				OutputCString(p->Out, (argNames[i] ? argNames[i] : p->NullValue));
				if (i < argsLength-1) _fprintf(p->Out, "%s", p->Separator);
			}
			_fprintf(p->Out, "\n");
			break; }
		case MODE_Csv: {
			if (p->Cnt++ == 0 && p->ShowHeader)
			{
				for (i = 0; i < argsLength; i++)
					OutputCsv(p, (colNames[i] ? colNames[i] : ""), i < argsLength-1);
				_fprintf(p->Out, "\n");
			}
			if (argNames == 0) break;
			for (i = 0; i < argsLength; i++)
				OutputCsv(p, argNames[i], i < argsLength-1);
			_fprintf(p->Out, "\n");
			break; }
		case MODE_Insert: {
			p->Cnt++;
			if (argNames == 0) break;
			_fprintf(p->Out, "INSERT INTO %s VALUES(", p->DestTable);
			for (i = 0; i < argsLength; i++)
			{
				char *sep = (i > 0 ? "," : "");
				if ((argNames[i] == 0) || (insertTypes && insertTypes[i] == TYPE_NULL))
					_fprintf(p->Out, "%sNULL", sep);
				else if (insertTypes && insertTypes[i] == TYPE_TEXT)
				{
					if (sep[0]) _fprintf(p->Out, "%s", sep);
					OutputQuotedString(p->Out, argNames[i]);
				}
				else if (insertTypes && (insertTypes[i] == TYPE_INTEGER || insertTypes[i] == TYPE_FLOAT))
					_fprintf(p->Out, "%s%s", sep, argNames[i]);
				else if (insertTypes && insertTypes[i] == TYPE_BLOB && p->Stmt)
				{
					const void *blob = Vdbe::Column_Blob(p->Stmt, i);
					int blobLength = Vdbe::Column_Bytes(p->Stmt, i);
					if (sep[0]) _fprintf(p->Out, "%s", sep);
					OutputHexBlob(p->Out, blob, blobLength);
				}
				else if (isNumber(argNames[i], 0))
					_fprintf(p->Out, "%s%s", sep, argNames[i]);
				else
				{
					if (sep[0]) _fprintf(p->Out, "%s", sep);
					OutputQuotedString(p->Out, argNames[i]);
				}
			}
			_fprintf(p->Out, ");\n");
			break; }
		}
		return 0;
	}

	static int callback(void *args, int colLength, char **colValues, char **colNames)
	{
		return ShellCallback(args, colLength, colValues, colNames, nullptr); // since we don't have type info, call the ShellCallback with a NULL value
	}

#pragma endregion

#pragma region Helpers

	static void SetTableName(struct CallbackData *p, const char *name)
	{
		if (p->DestTable)
		{
			free(p->DestTable);
			p->DestTable = nullptr;
		}
		if (!name) return;
		bool needQuote = (!isalpha((unsigned char)*name) && *name != '_');
		int i, n;
		for (i = n = 0; name[i]; i++, n++)
		{
			if (!isalnum((unsigned char)name[i]) && name[i] != '_')
			{
				needQuote = true;
				if (name[i] == '\'') n++;
			}
		}
		if (needQuote) n += 2;
		char *z = p->DestTable = (char *)malloc(n+1);
		if (!z)
		{
			_fprintf(stderr, "Error: out_ of memory\n");
			exit(1);
		}
		n = 0;
		if (needQuote) z[n++] = '\'';
		for (i = 0; name[i]; i++)
		{
			z[n++] = name[i];
			if (name[i] == '\'') z[n++] = '\'';
		}
		if (needQuote) z[n++] = '\'';
		z[n] = 0;
	}

	static char *AppendText(char *in, char const *append, char quote)
	{
		int i;
		int appendLength = _strlen(append);
		int inLength = (in ? _strlen(in) : 0);
		int newLength = appendLength+inLength+1;
		if (quote)
		{
			newLength += 2;
			for (i = 0; i < appendLength; i++)
				if (append[i] == quote) newLength++;
		}
		in = (char *)realloc(in, newLength);
		if (!in)
			return nullptr;
		if (quote)
		{
			char *csr = &in[inLength];
			*csr++ = quote;
			for (i = 0; i < appendLength; i++)
			{
				*csr++ = append[i];
				if (append[i] == quote) *csr++ = quote;
			}
			*csr++ = quote;
			*csr++ = '\0';
			_assert((csr-in) == newLength);
		}
		else
		{
			_memcpy(&in[inLength], append, appendLength);
			in[newLength-1] = '\0';
		}
		return in;
	}

	static int RunTableDumpQuery(struct CallbackData *p, const char *selectSql, const char *firstRow)
	{
		int i;
		Vdbe *select;
		RC rc = Prepare::Prepare_(p->Ctx, selectSql, -1, &select, 0);
		if (rc != RC_OK || !select)
		{
			_fprintf(p->Out, "/**** ERROR: (%d) %s *****/\n", rc, Main::ErrMsg(p->Ctx));
			p->Errs++;
			return rc;
		}
		rc = select->Step();
		int results = Vdbe::Column_Count(select);
		while (rc == RC_ROW)
		{
			if (firstRow)
			{
				_fprintf(p->Out, "%s", firstRow);
				firstRow = nullptr;
			}
			const char *z = (const char *)Vdbe::Column_Text(select, 0);
			_fprintf(p->Out, "%s", z);
			for (i = 1; i < results; i++)
				_fprintf(p->Out, ",%s", Vdbe::Column_Text(select, i));
			if (!z) z = "";
			while (z[0] && (z[0] != '-' || z[1] != '-')) z++;
			if (z[0]) _fprintf(p->Out, "\n;\n");
			else _fprintf(p->Out, ";\n");
			rc = select->Step();
		}
		rc = Vdbe::Finalize(select);
		if (rc != RC_OK)
		{
			_fprintf(p->Out, "/**** ERROR: (%d) %s *****/\n", rc, Main::ErrMsg(p->Ctx));
			p->Errs++;
		}
		return rc;
	}

	static char *save_err_msg(Context *ctx)
	{
		int errMsgLength = 1+_strlen(Main::ErrMsg(ctx));
		char *errMsg = (char *)_alloc(errMsgLength);
		if (errMsg)
			_memcpy(errMsg, Main::ErrMsg(ctx), errMsgLength);
		return errMsg;
	}

	static int display_stats(Context *ctx, struct CallbackData *arg, bool reset)
	{
		int cur;
		int high;
		if (arg && arg->Out)
		{
			high = cur = -1;
			_status(STATUS_MEMORY_USED, &cur, &high, reset);
			_fprintf(arg->Out, "Memory Used:                         %d (max %d) bytes\n", cur, high);
			high = cur = -1;
			_status(STATUS_MALLOC_COUNT, &cur, &high, reset);
			_fprintf(arg->Out, "Number of Outstanding Allocations:   %d (max %d)\n", cur, high);
			// Not currently used by the CLI.
			//    high = cur = -1;
			//    _status(STATUS_PAGECACHE_USED, &cur, &high, reset);
			//    _fprintf(arg->Out, "Number of Pcache Pages Used:         %d (max %d) pages\n", cur, high);
			high = cur = -1;
			_status(STATUS_PAGECACHE_OVERFLOW, &cur, &high, reset);
			_fprintf(arg->Out, "Number of Pcache Overflow Bytes:     %d (max %d) bytes\n", cur, high);
			// Not currently used by the CLI.
			//    high = cur = -1;
			//    _status(STATUS_SCRATCH_USED, &cur, &high, reset);
			//    _fprintf(arg->Out, "Number of Scratch Allocations Used:  %d (max %d)\n", cur, high);
			high = cur = -1;
			_status(STATUS_SCRATCH_OVERFLOW, &cur, &high, reset);
			_fprintf(arg->Out, "Number of Scratch Overflow Bytes:    %d (max %d) bytes\n", cur, high);
			high = cur = -1;
			_status(STATUS_MALLOC_SIZE, &cur, &high, reset);
			_fprintf(arg->Out, "Largest Allocation:                  %d bytes\n", high);
			high = cur = -1;
			_status(STATUS_PAGECACHE_SIZE, &cur, &high, reset);
			_fprintf(arg->Out, "Largest Pcache Allocation:           %d bytes\n", high);
			high = cur = -1;
			_status(STATUS_SCRATCH_SIZE, &cur, &high, reset);
			_fprintf(arg->Out, "Largest Scratch Allocation:          %d bytes\n", high);
#ifdef YYTRACKMAXSTACKDEPTH
			high = cur = -1;
			_status(STATUS_PARSER_STACK, &cur, &high, reset);
			_fprintf(arg->Out, "Deepest Parser Stack:                %d (max %d)\n", cur, high);
#endif
		}

		if (arg && arg->Out && ctx)
		{
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_LOOKASIDE_USED, &cur, &high, reset);
			_fprintf(arg->Out, "Lookaside Slots Used:                %d (max %d)\n", cur, high);
			ctx->Status(Context::CTXSTATUS_LOOKASIDE_HIT, &cur, &high, reset);
			_fprintf(arg->Out, "Successful lookaside attempts:       %d\n", high);
			ctx->Status(Context::CTXSTATUS_LOOKASIDE_MISS_SIZE, &cur, &high, reset);
			_fprintf(arg->Out, "Lookaside failures due to size:      %d\n", high);
			ctx->Status(Context::CTXSTATUS_LOOKASIDE_MISS_FULL, &cur, &high, reset);
			_fprintf(arg->Out, "Lookaside failures due to OOM:       %d\n", high);
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_CACHE_USED, &cur, &high, reset);
			_fprintf(arg->Out, "Pager Heap Usage:                    %d bytes\n", cur);
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_CACHE_HIT, &cur, &high, 1);
			_fprintf(arg->Out, "Page cache hits:                     %d\n", cur);
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_CACHE_MISS, &cur, &high, 1);
			_fprintf(arg->Out, "Page cache misses:                   %d\n", cur); 
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_CACHE_WRITE, &cur, &high, 1);
			_fprintf(arg->Out, "Page cache writes:                   %d\n", cur); 
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_SCHEMA_USED, &cur, &high, reset);
			_fprintf(arg->Out, "Schema Heap Usage:                   %d bytes\n", cur); 
			high = cur = -1;
			ctx->Status(Context::CTXSTATUS_STMT_USED, &cur, &high, reset);
			_fprintf(arg->Out, "Statement Heap/Lookaside Usage:      %d bytes\n", cur); 
		}

		if (arg && arg->Out && ctx && arg->Stmt)
		{
			cur = Vdbe::Status(arg->Stmt, Vdbe::STMTSTATUS_FULLSCAN_STEP, reset);
			_fprintf(arg->Out, "Fullscan Steps:                      %d\n", cur);
			cur = Vdbe::Status(arg->Stmt, Vdbe::STMTSTATUS_SORT, reset);
			_fprintf(arg->Out, "Sort Operations:                     %d\n", cur);
			cur = Vdbe::Status(arg->Stmt, Vdbe::STMTSTATUS_AUTOINDEX, reset);
			_fprintf(arg->Out, "Autoindex Inserts:                   %d\n", cur);
		}
		return 0;
	}

	// Execute a statement or set of statements.  Print any result rows/columns depending on the current mode set via the supplied callback.
	// This is very similar to SQLite's built-in sqlite3_exec() function except it takes a slightly different callback and callback data argument.
	// callback // (not the same as sqlite3_exec)
	static int ShellExec(Context *ctx, const char *sql, int (*callback)(void*,int,char**,char**,int*), struct CallbackData *arg, char **errMsgOut)
	{
		Vdbe *stmt = nullptr; // Statement to execute.
		RC rc = RC_OK;
		int rc2;
		const char *leftover; // Tail of unprocessed SQL

		if (errMsgOut)
			*errMsgOut = nullptr;

		while (sql[0] && (rc == RC_OK))
		{
			rc = Prepare::Preparev2(ctx, sql, -1, &stmt, &leftover);
			if (rc != RC_OK)
			{
				if (errMsgOut)
					*errMsgOut = save_err_msg(db);
			}
			else
			{
				if (!stmt)
				{
					sql = leftover; // this happens for a comment or white-space
					while (IsSpace(sql[0])) sql++;
					continue;
				}

				// save off the prepared statment handle and reset row count
				if (arg)
				{
					arg->Stmt = stmt;
					arg->Cnt = 0;
				}

				// echo the sql statement if echo on
				if (arg && arg->EchoOn)
				{
					const char *stmtSql = Vdbe::Sql_(stmt);
					_fprintf(arg->Out, "%s\n", (stmtSql ? stmtSql : sql));
				}

				// Output TESTCTRL_EXPLAIN text of requested
				if (arg && arg->Mode == MODE_Explain)
				{
					const char *explain = nullptr;
					sqlite3_test_control(TESTCTRL_EXPLAIN_STMT, stmt, &explain);
					if (explain && explain[0])
						_fprintf(arg->Out, "%s", explain);
				}

				// perform the first step.  this will tell us if we have a result set or not and how wide it is.
				RC rc = stmt->Step();
				// if we have a result set...
				if (rc == RC_ROW)
				{
					// if we have a callback...
					if (callback)
					{
						// allocate space for col name ptr, value ptr, and type 
						int colLength = Vdbe::Column_Count(stmt);
						void *data = _alloc(3*colLength*sizeof(const char *) + 1);
						if (!data)
							rc = RC_NOMEM;
						else
						{
							char **colNames = (char **)data; // Names of result columns
							char **colValues = &colNames[colLength];       // Results
							int *colTypes = (int *)&colValues[colLength]; // Result types
							int i;
							_assert(sizeof(int) <= sizeof(char *)); 
							// save off ptrs to column names
							for (i = 0; i < colLength; i++)
								colNames[i] = (char *)Vdbe::Column_Name(stmt, i);
							do
							{
								// extract the data and data types
								for (i = 0; i < colLength; i++)
								{
									colValues[i] = (char *)Vdbe::Column_Text(stmt, i);
									colTypes[i] = Vdbe::Column_Type(stmt, i);
									if (!colValues[i] && (colTypes[i] != TYPE_NULL))
									{
										rc = RC_NOMEM;
										break;
									}
								}
								// if data and types extracted successfully...call the supplied callback with the result row data
								if (rc == RC_ROW)
									rc = (callback(arg, colLength, colValues, colNames, colTypes) ? RC_ABORT : stmt->Step());
							} while (rc == RC_ROW);
							_free(data);
						}
					}
					else
						do { rc = stmt->Step(); }
						while (rc == RC_ROW);
				}

				// print Usage stats if stats on
				if (arg && arg->StatsOn)
					display_stats(ctx, arg, 0);

				// Finalize the statement just executed. If this fails, save a copy of the error message. Otherwise, set sql to point to the next statement to execute.
				rc2 = Vdbe::Finalize(stmt);
				if (rc != RC_NOMEM) rc = rc2;
				if (rc == RC_OK)
				{
					sql = leftover;
					while (IsSpace(sql[0])) sql++;
				}
				else if (errMsgOut)
					*errMsgOut = save_err_msg(db);

				// clear saved stmt handle
				if (arg)
					arg->Stmt = nullptr;
			}
		}
		return rc;
	}

	// This is a different callback routine used for dumping the database. Each row received by this callback consists of a table name,
	// the table type ("index" or "table") and SQL to create the table. This routine should print text sufficient to recreate the table.
	static int dump_callback(void *arg, int argsLength, char **args, char **cols)
	{
		RC rc;
		const char *prepStmt = nullptr;
		struct CallbackData *p = (struct CallbackData *)arg;

		if (argsLength != 3) return 1;
		const char *tableName = args[0];
		const char *typeName = args[1];
		const char *sql = args[2];

		if (!strcmp(tableName, "sqlite_sequence")) prepStmt = "DELETE FROM sqlite_sequence;\n";
		else if (!strcmp(tableName, "sqlite_stat1")) _fprintf(p->Out, "ANALYZE sqlite_master;\n");
		else if (!strncmp(tableName, "sqlite_", 7)) return 0;
		else if (!strncmp(sql, "CREATE VIRTUAL TABLE", 20))
		{
			if (!p->WritableSchema)
			{
				_fprintf(p->Out, "PRAGMA writable_schema=ON;\n");
				p->WritableSchema = 1;
			}
			char *ins = _mprintf(
				"INSERT INTO sqlite_master(type,name,tbl_name,rootpage,sql)"
				"VALUES('table','%q','%q',0,'%q');",
				tableName, tableName, sql);
			_fprintf(p->Out, "%s\n", ins);
			_free(ins);
			return 0;
		}
		else _fprintf(p->Out, "%s;\n", sql);

		if (!strcmp(typeName, "table"))
		{
			char *tableInfoSql = AppendText(zTableInfo, "PRAGMA table_info(", 0);
			tableInfoSql = AppendText(zTableInfo, tableName, '"');
			tableInfoSql = AppendText(zTableInfo, ");", 0);

			Vdbe *tableInfo = nullptr;
			rc = Prepare::Prepare_(p->Ctx, tableInfoSql, -1, &tableInfo, 0);
			free(tableInfoSql);
			if (rc != RC_OK || !tableInfo)
				return 1;

			char *select = nullptr;
			select = AppendText(select, "SELECT 'INSERT INTO ' || ", 0);
			// Always quote the table name, even if it appears to be pure ascii, in case it is a keyword. Ex:  INSERT INTO "table" ...
			char *tmp = nullptr;
			tmp = AppendText(tmp, tableName, '"');
			if (tmp)
			{
				select = AppendText(select, tmp, '\'');
				free(tmp);
			}
			select = AppendText(select, " || ' VALUES(' || ", 0);
			rc = tableInfo->Step();
			int rows = 0;
			while (rc == RC_ROW)
			{
				const char *text = (const char *)Vdbe::Column_Text(tableInfo, 1);
				select = AppendText(select, "quote(", nullptr);
				select = AppendText(select, text, '"');
				rc = tableInfo->Step();
				select = AppendText(select, (rc == RC_ROW ? "), " : ") "), nullptr);
				rows++;
			}
			rc = Vdbe::Finalize(tableInfo);
			if (rc != RC_OK || rows == 0)
			{
				free(select);
				return 1;
			}
			select = AppendText(select, "|| ')' FROM  ", 0);
			select = AppendText(select, tableName, '"');

			rc = RunTableDumpQuery(p, select, prepStmt);
			if (rc == RC_CORRUPT)
			{
				select = AppendText(select, " ORDER BY rowid DESC", 0);
				RunTableDumpQuery(p, select, nullptr);
			}
			free(select);
		}
		return 0;
	}

	// Run zQuery.  Use dump_callback() as the callback routine so that the contents of the query are output as SQL statements.
	// If we get a SQLITE_CORRUPT error, rerun the query after appending "ORDER BY rowid DESC" to the end.
	static int RunSchemaDumpQuery(struct CallbackData *p, const char *query)
	{
		char *err = nullptr;
		RC rc = Main::Exec(p->Ctx, query, dump_callback, p, &err);
		if (rc == RC_CORRUPT)
		{
			int length = _strlen(query);
			_fprintf(p->Out, "/****** CORRUPTION ERROR *******/\n");
			if (err)
			{
				_fprintf(p->Out, "/****** %s ******/\n", err);
				_free(err);
				err = nullptr;
			}
			char *q2 = malloc(length+100);
			if (!q2) return rc;
			_snprintf(length+100, q2, "%s ORDER BY rowid DESC", query);
			rc = Main::Exec(p->Ctx, q2, dump_callback, p, &err);
			if (rc)
				_fprintf(p->Out, "/****** ERROR: %s ******/\n", err);
			else
				rc = RC_CORRUPT;
			_free(err);
			free(q2);
		}
		return rc;
	}

	// Text of a help message
	static char _help[] =
		".backup ?DB? FILE      Backup DB (default \"main\") to FILE\n"
		".bail ON|OFF           Stop after hitting an error.  Default OFF\n"
		".databases             List names and files of attached databases\n"
		".dump ?TABLE? ...      Dump the database in an SQL text format\n"
		"                         If TABLE specified, only dump tables matching\n"
		"                         LIKE pattern TABLE.\n"
		".echo ON|OFF           Turn command echo on or off\n"
		".exit                  Exit this program\n"
		".explain ?ON|OFF?      Turn output mode suitable for EXPLAIN on or off.\n"
		"                         With no args, it turns EXPLAIN on.\n"
		".header(s) ON|OFF      Turn display of headers on or off\n"
		".help                  Show this message\n"
		".import FILE TABLE     Import data from FILE into TABLE\n"
		".indices ?TABLE?       Show names of all indices\n"
		"                         If TABLE specified, only show indices for tables\n"
		"                         matching LIKE pattern TABLE.\n"
#ifdef ENABLE_IOTRACE
		".iotrace FILE          Enable I/O diagnostic logging to FILE\n"
#endif
#ifndef OMIT_LOAD_EXTENSION
		".load FILE ?ENTRY?     Load an extension library\n"
#endif
		".log FILE|off          Turn logging on or off.  FILE can be stderr/stdout\n"
		".mode MODE ?TABLE?     Set output mode where MODE is one of:\n"
		"                         csv      Comma-separated values\n"
		"                         column   Left-aligned columns.  (See .width)\n"
		"                         html     HTML <table> code\n"
		"                         insert   SQL insert statements for TABLE\n"
		"                         line     One value per line\n"
		"                         list     Values delimited by .separator string\n"
		"                         tabs     Tab-separated values\n"
		"                         tcl      TCL list elements\n"
		".nullvalue STRING      Use STRING in place of NULL values\n"
		".output FILENAME       Send output to FILENAME\n"
		".output stdout         Send output to the screen\n"
		".print STRING...       Print literal STRING\n"
		".prompt MAIN CONTINUE  Replace the standard prompts\n"
		".quit                  Exit this program\n"
		".read FILENAME         Execute SQL in FILENAME\n"
		".restore ?DB? FILE     Restore content of DB (default \"main\") from FILE\n"
		".schema ?TABLE?        Show the CREATE statements\n"
		"                         If TABLE specified, only show tables matching\n"
		"                         LIKE pattern TABLE.\n"
		".separator STRING      Change separator used by output mode and .import\n"
		".show                  Show the current values for various settings\n"
		".stats ON|OFF          Turn stats on or off\n"
		".tables ?TABLE?        List names of tables\n"
		"                         If TABLE specified, only list tables matching\n"
		"                         LIKE pattern TABLE.\n"
		".timeout MS            Try opening locked tables for MS milliseconds\n"
		".trace FILE|off        Output each SQL statement as it is run\n"
		".vfsname ?AUX?         Print the name of the VFS stack\n"
		".width NUM1 NUM2 ...   Set column widths for \"column\" mode\n"
		;

	static char _timerHelp[] =
		".timer ON|OFF          Turn the CPU timer measurement on or off\n"
		;


	static int ProcessInput(struct CallbackData *p, FILE *in); // Forward reference

	// Make sure the database is open.  If it is not, then open it.  If the database fails to open, print an error message and exit.
	static void OpenCtx(struct CallbackData *p)
	{
		if (!p->Ctx)
		{
			Main::Initialize();
			Main::Open(p->DbFilename, &p->Ctx);
			ctx = p->Ctx;
			if (ctx && Main::ErrCode(ctx) == RC_OK)
				sqlite3_create_function(ctx, "shellstatic", 0, TEXTENCODE_UTF8, 0, shellstaticFunc, 0, 0);
			if (!ctx || Main::ErrCode(ctx) != RC_OK)
			{
				_fprintf(stderr,"Error: unable to open database \"%s\": %s\n", p->DbFilename, Main::ErrMsg(ctx));
				exit(1);
			}
#ifndef OMIT_LOAD_EXTENSION
			//Main::enable_load_extension(p->db, 1);
#endif
#ifdef ENABLE_REGEXP
			{
				extern int sqlite3_add_regexp_func(sqlite3*);
				sqlite3_add_regexp_func(db);
			}
#endif
#ifdef ENABLE_SPELLFIX
			{
				extern int sqlite3_spellfix1_register(sqlite3*);
				sqlite3_spellfix1_register(db);
			}
#endif
		}
	}

	// Do C-language style dequoting.
	//
	//    \t    -> tab
	//    \n    -> newline
	//    \r    -> carriage return
	//    \NNN  -> ascii character NNN in octal
	//    \\    -> backslash
	static void ResolveBackslashes(char *z)
	{
		int i, j;
		char c;
		for (i = j = 0; (c = z[i]) != 0; i++, j++)
		{
			if (c == '\\')
			{
				c = z[++i];
				if (c == 'n') c = '\n';
				else if (c == 't') c = '\t';
				else if (c == 'r') c = '\r';
				else if (c >= '0' && c <= '7')
				{
					c -= '0';
					if (z[i+1] >= '0' && z[i+1] <= '7')
					{
						i++; c = (c<<3) + z[i] - '0';
						if (z[i+1] >= '0' && z[i+1] <= '7') { i++; c = (c<<3) + z[i] - '0'; }
					}
				}
			}
			z[j] = c;
		}
		z[j] = 0;
	}

	// Interpret zArg as a boolean value.  Return either 0 or 1.
	static bool BooleanValue(char *arg)
	{
		int i;
		for (i = 0; arg[i] >= '0' && arg[i] <= '9'; i++) { }
		if (i > 0 && arg[i] == 0) return atoi(arg);
		if (!_strcmp(arg, "on") || !strcmp(arg, "yes")) return true;
		if (!_strcmp(arg, "off") || !_strcmp(arg, "no")) return false;
		_fprintf(stderr, "ERROR: Not a boolean value: \"%s\". Assuming \"no\".\n", arg);
		return false;
	}

	// Close an output file, assuming it is not stderr or stdout
	static void OutputFileClose(FILE *f)
	{
		if (f && f != stdout && f != stderr) fclose(f);
	}

	// Try to open an output file.  The names "stdout" and "stderr" are recognized and do the right thing.  NULL is returned if the output filename is "off".
	static FILE *OutputFileOpen(const char *file)
	{
		FILE *f;
		if (!_strcmp(file, "stdout")) f = stdout;
		else if (!_strcmp(file, "stderr")) f = stderr;
		else if (!_strcmp(file, "off")) f = 0;
		else
		{
			f = fopen(file, "wb");
			if (!f)
				_fprintf(stderr, "Error: cannot open \"%s\"\n", file);
		}
		return f;
	}

	// A routine for handling output from sqlite3_trace().
	static void SqlTraceCallback(void *arg, const char *z)
	{
		FILE *f = (FILE *)arg;
		if (f) _fprintf(f, "%s\n", z);
	}

	// A no-op routine that runs with the ".breakpoint" doc-command.  This is a useful spot to set a debugger breakpoint.
	static void TestBreakpoint()
	{
		static int calls = 0;
		calls++;
	}



#pragma endregion

#pragma region META

	static int DoMetaCommand(char *line, struct CallbackData *p)
	{
		int i = 1;
		int argsLength = 0;
		int rc = 0;
		char *args[50];

		// Parse the input line into tokens.
		while (line[i] && argsLength < _lengthof(args))
		{
			while (IsSpace(line[i])) { i++; }
			if (line[i] == 0) break;
			if (line[i] == '\'' || line[i] == '"')
			{
				int delim = line[i++];
				args[argsLength++] = &line[i];
				while (line[i] && line[i] != delim) { i++; }
				if (line[i] == delim)
					line[i++] = 0;
				if (delim == '"') ResolveBackslashes(args[argsLength-1]);
			}
			else
			{
				args[argsLength++] = &line[i];
				while (line[i] && !IsSpace(line[i])) { i++; }
				if (line[i]) line[i++] = 0;
				ResolveBackslashes(args[argsLength-1]);
			}
		}

		// Process the input line.
		if (argsLength == 0) return 0; // no tokens, no error
		int n = _strlen(args[0]);
		int c = args[0][0];
		if (c == 'b' && n >= 3 && !strncmp(args[0], "backup", n))
		{
			const char *destFile = nullptr;
			const char *dbName = nullptr;
			const char *key = nullptr;
			Context *dest;
			sqlite3_backup *backup;
			for (int j = 1; j < argsLength; j++)
			{
				const char *z = args[j];
				if (z[0] == '-')
				{
					while (z[0] == '-') z++;
					if (!strcmp(z, "key") && j < argsLength-1)
						key = args[++j];
					else
					{
						_fprintf(stderr, "unknown option: %s\n", args[j]);
						return 1;
					}
				}
				else if (!destFile)
					destFile = args[j];
				else if (!dbName)
				{
					dbName = destFile;
					destFile = args[j];
				}
				else
				{
					_fprintf(stderr, "too many arguments to .backup\n");
					return 1;
				}
			}
			if (!destFile)
			{
				_fprintf(stderr, "missing FILENAME argument on .backup\n");
				return 1;
			}
			if (!dbName) dbName = "main";
			rc = sqlite3_open(destFile, &dest);
			if (rc != RC_OK)
			{
				_fprintf(stderr, "Error: cannot open \"%s\"\n", destFile);
				Main::Close(dest);
				return 1;
			}
#ifdef HAS_CODEC
			sqlite3_key(dest, key, (int)strlen(key));
#else
			(void)zKey;
#endif
			OpenCtx(p);
			backup = sqlite3_backup_init(pDest, "main", p->Ctx, dbName);
			if (!backup)
			{
				_fprintf(stderr, "Error: %s\n", Main::ErrMsg(dest));
				Main::Close(dest);
				return 1;
			}
			while ((rc = sqlite3_backup_step(backup, 100)) == RC_OK) { }
			sqlite3_backup_finish(backup);
			if (rc == RC_DONE)
				rc = 0;
			else
			{
				_fprintf(stderr, "Error: %s\n", Main::ErrMsg(dest));
				rc = 1;
			}
			Main::Close(dest);
		}
		else if (c == 'b' && n >= 3 && !strncmp(args[0], "bail", n) && argsLength > 1 && argsLength < 3)
		{
			_bailOnError = BooleanValue(args[1]);
		}
		else if (c == 'b' && n >= 3 && !strncmp(args[0], "breakpoint", n))
			// The undocumented ".breakpoint" command causes a call to the no-op routine named TestBreakpoint().
		{
			TestBreakpoint();
		}
		else if (c == 'd' && n > 1 && !strncmp(args[0], "databases", n) && argsLength == 1)
		{
			OpenCtx(p);
			struct CallbackData data;
			memcpy(&data, p, sizeof(data));
			data.ShowHeader = 1;
			data.Mode = MODE_Column;
			data.ColWidth[0] = 3;
			data.ColWidth[1] = 15;
			data.ColWidth[2] = 58;
			data.Cnt = 0;
			char *errMsg = 0;
			Main::Exec(p->Ctx, "PRAGMA database_list; ", callback, &data, &errMsg);
			if (errMsg)
			{
				_fprintf(stderr,"Error: %s\n", errMsg);
				_free(errMsg);
				rc = 1;
			}
		}
		else if (c == 'd' && !strncmp(args[0], "dump", n) && argsLength < 3)
		{
			OpenCtx(p);
			// When playing back a "dump", the content might appear in an order which causes immediate foreign key constraints to be violated.
			// So disable foreign-key constraint enforcement to prevent problems.
			_fprintf(p->Out, "PRAGMA foreign_keys=OFF;\n");
			_fprintf(p->Out, "BEGIN TRANSACTION;\n");
			p->WritableSchema = 0;
			Main::Exec(p->Ctx, "SAVEPOINT dump; PRAGMA writable_schema=ON", 0, 0, 0);
			p->Errs = 0;
			if (argsLength == 1)
			{
				RunSchemaDumpQuery(p, 
					"SELECT name, type, sql FROM sqlite_master "
					"WHERE sql NOT NULL AND type=='table' AND name!='sqlite_sequence'");
				RunSchemaDumpQuery(p, 
					"SELECT name, type, sql FROM sqlite_master "
					"WHERE name=='sqlite_sequence'");
				RunTableDumpQuery(p,
					"SELECT sql FROM sqlite_master "
					"WHERE sql NOT NULL AND type IN ('index','trigger','view')", 0);
			}
			else
			{
				for (int i = 1; i < argsLength; i++)
				{
					shellStatic = args[i];
					RunSchemaDumpQuery(p,
						"SELECT name, type, sql FROM sqlite_master "
						"WHERE tbl_name LIKE shellstatic() AND type=='table'"
						"  AND sql NOT NULL");
					RunTableDumpQuery(p,
						"SELECT sql FROM sqlite_master "
						"WHERE sql NOT NULL"
						"  AND type IN ('index','trigger','view')"
						"  AND tbl_name LIKE shellstatic()", 0);
					shellStatic = nullptr;
				}
			}
			if (p->WritableSchema)
			{
				_fprintf(p->Out, "PRAGMA writable_schema=OFF;\n");
				p->WritableSchema = 0;
			}
			Main::Exec(p->Ctx, "PRAGMA writable_schema=OFF;", 0, 0, 0);
			Main::Exec(p->Ctx, "RELEASE dump;", 0, 0, 0);
			_fprintf(p->Out, (p->Errs ? "ROLLBACK; -- due to errors\n" : "COMMIT;\n"));
		}
		else if (c == 'e' && !strncmp(args[0], "echo", n) && argsLength > 1 && argsLength < 3)
		{
			p->EchoOn = BooleanValue(args[1]);
		}
		else if (c == 'e' && !strncmp(args[0], "exit", n))
		{
			if (argsLength > 1 && (rc = atoi(args[1])) != 0) exit(rc);
			rc = 2;
		}
		else if (c == 'e' && !strncmp(args[0], "explain", n) && argsLength < 3)
		{
			int val = (argsLength >= 2 ? BooleanValue(args[1]) : 1);
			if (val == 1)
			{
				if (!p->ExplainPrev.Valid)
				{
					p->ExplainPrev.Valid = 1;
					p->ExplainPrev.Mode = p->Mode;
					p->ExplainPrev.ShowHeader = p->ShowHeader;
					memcpy(p->ExplainPrev.ColWidth, p->ColWidth, sizeof(p->ColWidth));
				}
				// We could put this code under the !p->explainValid condition so that it does not execute if we are already in
				// explain mode. However, always executing it allows us an easy was to reset to explain mode in case the user previously
				// did an .explain followed by a .width, .mode or .header command.
				p->Mode = MODE_Explain;
				p->ShowHeader = 1;
				memset(p->ColWidth, 0, _lengthof(p->ColWidth));
				p->ColWidth[0] = 4;		// addr
				p->ColWidth[1] = 13;	// opcode
				p->ColWidth[2] = 4;		// P1
				p->ColWidth[3] = 4;		// P2
				p->ColWidth[4] = 4;		// P3
				p->ColWidth[5] = 13;	// P4
				p->ColWidth[6] = 2;		// P5
				p->ColWidth[7] = 13;	// Comment
			}
			else if (p->ExplainPrev.Valid)
			{
				p->ExplainPrev.Valid = 0;
				p->Mode = p->ExplainPrev.Mode;
				p->ShowHeader = p->ExplainPrev.ShowHeader;
				memcpy(p->ColWidth, p->ExplainPrev.ColWidth, sizeof(p->ColWidth));
			}
		}
		else if (c == 'h' && (!strncmp(args[0], "header", n) || !strncmp(args[0], "headers", n)) && argsLength > 1 && argsLength < 3)
		{
			p->ShowHeader = BooleanValue(args[1]);
		}
		else if (c == 'h' && !strncmp(args[0], "help", n))
		{
			_fprintf(stderr, "%s", _help);
			if (HAS_TIMER)
				_fprintf(stderr, "%s", _timerHelp);
		}
		else if (c == 'i' && !strncmp(args[0], "import", n) && argsLength == 3)
		{
			char *tableName = args[2];    // Insert data into this table
			char *file = args[1];     // The file from which to extract data
			Vdbe *stmt = nullptr; // A statement
			int i;                   // Loop counters
			OpenCtx(p);
			int sepLength = _strlen(p->Separator); // Number of bytes in p->separator[]
			if (sepLength == 0)
			{
				_fprintf(stderr, "Error: non-null separator required for import\n");
				return 1;
			}
			char *sql = _mprintf("SELECT * FROM %s", tableName); // An SQL statement
			if (!sql)
			{
				_fprintf(stderr, "Error: out_ of memory\n");
				return 1;
			}
			int bytes = _strlen(sql); // Number of bytes in an SQL string
			rc = Prepare::Prepare_(p->Ctx, sql, -1, &stmt, 0);
			_free(sql);
			if (rc)
			{
				if (stmt) Vdbe::Finalize(stmt);
				_fprintf(stderr,"Error: %s\n", Main::ErrMsg(ctx));
				return 1;
			}
			int colsLength = Vdbe::Column_Count(stmt); // Number of columns in the table
			Vdbe::Finalize(stmt);
			stmt = nullptr;
			if (colsLength == 0) return 0; // no columns, no error
			sql = malloc(bytes + 20 + colsLength*2);
			if (!sql)
			{
				_fprintf(stderr, "Error: out_ of memory\n");
				return 1;
			}
			_snprintf(bytes+20, sql, "INSERT INTO %s VALUES(?", tableName);
			int j = _strlen(sql);
			for (i = 1; i < colsLength; i++)
			{
				sql[j++] = ',';
				sql[j++] = '?';
			}
			sql[j++] = ')';
			sql[j] = 0;
			rc = Prepare::Prepare_(p->Ctx, sql, -1, &stmt, 0);
			free(sql);
			if (rc)
			{
				_fprintf(stderr, "Error: %s\n", Main::ErrMsg(ctx));
				if (stmt) Vdbe::Finalize(stmt);
				return 1;
			}
			FILE *in = fopen(file, "rb") // The input file;
				if (!in)
				{
					_fprintf(stderr, "Error: cannot open \"%s\"\n", file);
					Vdbe::Finalize(stmt);
					return 1;
				}
				char **cols; // line[] broken up into columns
				cols = malloc(sizeof(cols[0])*(colsLength+1));
				if (!cols)
				{
					_fprintf(stderr, "Error: out_ of memory\n");
					fclose(in);
					Vdbe::Finalize(stmt);
					return 1;
				}
				Main::Exec(p->Ctx, "BEGIN", 0, 0, 0);
				char *commit = "COMMIT"; // How to commit changes
				int lineno = 0; // Line number of input file
				char *line; // A single line of input from the file
				while ((line = local_getline(0, in, 1)) != 0)
				{
					char *z, c;
					bool inQuote = false;
					lineno++;
					cols[0] = line;
					for (i = 0, z = line; (c = *z) != 0; z++)
					{
						if (c == '"') inQuote = !inQuote;
						if (c == '\n') lineno++;
						if (!inQuote && c == p->Separator[0] && !strncmp(z, p->Separator, sepLength))
						{
							*z = 0;
							i++;
							if (i < colsLength)
							{
								cols[i] = &z[sepLength];
								z += sepLength-1;
							}
						}
					} // end for
					*z = 0;
					if (i+1 != colsLength)
					{
						_fprintf(stderr, "Error: %s line %d: expected %d columns of data but found %d\n", file, lineno, colsLength, i+1);
						commit = "ROLLBACK";
						free(line);
						rc = 1;
						break; // from while
					}
					for (i = 0; i < colsLength; i++)
					{
						if (cols[i][0]=='"')
						{
							int k;
							for (z = cols[i], j = 1, k = 0; z[j]; j++){
								if (z[j] == '"' ) { j++; if (z[j] == 0) break; }
								z[k++] = z[j];
							}
							z[k] = 0;
						}
						Vdbe::Bind_Text(stmt, i+1, cols[i], -1, DESTRUCTOR_STATIC);
					}
					stmt->Step();
					rc = stmt->Reset();
					free(line);
					if (rc != RC_OK)
					{
						_fprintf(stderr,"Error: %s\n", Main::ErrMsg(ctx));
						commit = "ROLLBACK";
						rc = 1;
						break; // from while
					}
				} // end while
				free(cols);
				fclose(in);
				Vdbe::Finalize(stmt);
				Main::Exec(p->Ctx, commit, 0, 0, 0);
		}
		else if (c == 'i' && !strncmp(args[0], "indices", n) && argsLength < 3)
		{
			struct CallbackData data;
			char *errMsg = nullptr;
			OpenCtx(p);
			memcpy(&data, p, sizeof(data));
			data.ShowHeader = 0;
			data.Mode = MODE_List;
			if (argsLength == 1)
				rc = Main::Exec(p->Ctx,
				"SELECT name FROM sqlite_master "
				"WHERE type='index' AND name NOT LIKE 'sqlite_%' "
				"UNION ALL "
				"SELECT name FROM sqlite_temp_master "
				"WHERE type='index' "
				"ORDER BY 1",
				callback, &data, &errMsg);
			else
			{
				shellStatic = args[1];
				rc = Main::Exec(p->Ctx,
					"SELECT name FROM sqlite_master "
					"WHERE type='index' AND tbl_name LIKE shellstatic() "
					"UNION ALL "
					"SELECT name FROM sqlite_temp_master "
					"WHERE type='index' AND tbl_name LIKE shellstatic() "
					"ORDER BY 1",
					callback, &data, &errMsg);
				shellStatic = nullptr;
			}
			if (errMsg)
			{
				_fprintf(stderr,"Error: %s\n", errMsg);
				_free(errMsg);
				rc = 1;
			}
			else if( rc != RC_OK)
			{
				_fprintf(stderr,"Error: querying sqlite_master and sqlite_temp_master\n");
				rc = 1;
			}
		}
#ifdef ENABLE_IOTRACE
		else if (c == 'i' && !strncmp(args[0], "iotrace", n))
		{
			extern void (*sqlite3IoTrace)(const char*, ...);
			if (iotrace && iotrace != stdout) fclose(iotrace);
			iotrace = 0;
			if (argsLength < 2)
				sqlite3IoTrace = 0;
			else if (!strcmp(args[1], "-"))
			{
				sqlite3IoTrace = iotracePrintf;
				iotrace = stdout;
			}
			else
			{
				iotrace = fopen(args[1], "w");
				if (iotrace == 0)
				{
					_fprintf(stderr, "Error: cannot open \"%s\"\n", args[1]);
					sqlite3IoTrace = 0;
					rc = 1;
				}
				else
					sqlite3IoTrace = iotracePrintf;
			}
		}
#endif
#ifndef OMIT_LOAD_EXTENSION
		else if (c == 'l' && !strncmp(args[0], "load", n) && argsLength >= 2)
		{
			const char *file = args[1];
			const char *proc = (argsLength >= 3 ? args[2] : 0);
			OpenCtx(p);
			char *errMsg = 0;
			rc = sqlite3_load_extension(p->Ctx, file, proc, &errMsg);
			if (rc != RC_OK)
			{
				_fprintf(stderr, "Error: %s\n", errMsg);
				_free(errMsg);
				rc = 1;
			}
		}
#endif
		else if (c == 'l' && !strncmp(args[0], "log", n) && argsLength >= 2)
		{
			const char *file = args[1];
			OutputFileClose(p->Log);
			p->Log = OutputFileOpen(file);
		}
		else if (c == 'm' && !strncmp(args[0], "mode", n) && argsLength == 2)
		{
			int n2 = _strlen(args[1]);
			if ((n2 == 4 && !strncmp(args[1],"line",n2)) || (n2 == 5 && !strncmp(args[1],"lines",n2))) p->Mode = MODE_Line;
			else if ((n2 == 6 && !strncmp(args[1],"column",n2)) || (n2 == 7 && !strncmp(args[1],"columns",n2))) p->Mode = MODE_Column;
			else if (n2 == 4 && !strncmp(args[1],"list",n2)) p->Mode = MODE_List;
			else if (n2 == 4 && !strncmp(args[1],"html",n2)) p->Mode = MODE_Html;
			else if (n2 == 3 && !strncmp(args[1],"tcl",n2)) { p->Mode = MODE_Tcl; _snprintf(sizeof(p->Separator), p->Separator, " "); }
			else if (n2 == 3 && !strncmp(args[1],"csv",n2)) { p->Mode = MODE_Csv; _snprintf(sizeof(p->Separator), p->Separator, ","); }
			else if (n2 == 4 && !strncmp(args[1],"tabs",n2)) { p->Mode = MODE_List; _snprintf(sizeof(p->Separator), p->Separator, "\t"); }
			else if (n2 == 6 && !strncmp(args[1],"insert",n2)) { p->Mode = MODE_Insert; SetTableName(p, "table"); }
			else
			{
				_fprintf(stderr,"Error: mode should be one of: column csv html insert line list tabs tcl\n");
				rc = 1;
			}
		}
		else if (c == 'm' && !strncmp(args[0], "mode", n) && argsLength == 3)
		{
			int n2 = _strlen(args[1]);
			if (n2 == 6 && !strncmp(args[1],"insert",n2)) { p->Mode = MODE_Insert; SetTableName(p, args[2]); }
			else
			{
				_fprintf(stderr, "Error: invalid arguments:  \"%s\". Enter \".help\" for help\n", args[2]);
				rc = 1;
			}
		}
		else if (c == 'n' && !strncmp(args[0], "nullvalue", n) && argsLength == 2)
		{
			_snprintf(sizeof(p->NullValue), p->NullValue, "%.*s", (int)_lengthof(p->NullValue)-1, args[1]);
		}
		else if (c == 'o' && !strncmp(args[0], "output", n) && argsLength == 2)
		{
			if (p->Outfile[0] == '|') pclose(p->Out);
			else OutputFileClose(p->Out);
			p->Outfile[0] = 0;
			if (args[1][0] == '|')
			{
				p->Out = popen(&args[1][1], "w");
				if (!p->Out)
				{
					_fprintf(stderr,"Error: cannot open pipe \"%s\"\n", &args[1][1]);
					p->Out = stdout;
					rc = 1;
				}
				else
					_snprintf(sizeof(p->Outfile), p->Outfile, "%s", args[1]);
			}
			else
			{
				p->Out = OutputFileOpen(args[1]);
				if (!p->Out)
				{
					if (strcmp(args[1], "off"))
						_fprintf(stderr, "Error: cannot write to \"%s\"\n", args[1]);
					p->Out = stdout;
					rc = 1;
				}
				else
					_snprintf(sizeof(p->Outfile), p->Outfile, "%s", args[1]);
			}
		}
		else if (c == 'p' && n >= 3 && !strncmp(args[0], "print", n))
		{
			for (int i = 1; i < argsLength; i++)
			{
				if (i > 1) _fprintf(p->Out, " ");
				_fprintf(p->Out, "%s", args[i]);
			}
			_fprintf(p->Out, "\n");
		}
		else if (c == 'p' && !strncmp(args[0], "prompt", n) && (argsLength == 2 || argsLength == 3))
		{
			if (argsLength >= 2)
				strncpy(_mainPrompt, args[1], (int)_lengthof(_mainPrompt)-1);
			if (argsLength >= 3)
				strncpy(_continuePrompt, args[2], (int)_lengthof(_continuePrompt)-1);
		}
		else if (c == 'q' && !strncmp(args[0], "quit", n) && argsLength == 1)
		{
			rc = 2;
		}
		else if (c == 'r' && n >= 3 && !strncmp(args[0], "read", n) && argsLength == 2)
		{
			FILE *alt = fopen(args[1], "rb");
			if (!alt)
			{
				_fprintf(stderr, "Error: cannot open \"%s\"\n", args[1]);
				rc = 1;
			}
			else
			{
				rc = ProcessInput(p, alt);
				fclose(alt);
			}
		}
		else if (c == 'r' && n >= 3 && !strncmp(args[0], "restore", n) && argsLength > 1 && argsLength < 4)
		{
			const char *srcFile;
			const char *dbName;
			if (argsLength == 2)
			{
				srcFile = args[1];
				dbName = "main";
			}
			else
			{
				srcFile = args[2];
				dbName = args[1];
			}
			Context *src;
			rc = sqlite3_open(srcFile, &src);
			if (rc != RC_OK)
			{
				_fprintf(stderr, "Error: cannot open \"%s\"\n", srcFile);
				Main::Close(src);
				return 1;
			}
			OpenCtx(p);
			sqlite3_backup *backup = sqlite3_backup_init(p->db, dbName, src, "main");
			if (!backup)
			{
				_fprintf(stderr, "Error: %s\n", Main::ErrMsg(p->Ctx));
				Main::Close(src);
				return 1;
			}
			int timeout = 0;
			while ((rc = sqlite3_backup_step(backup, 100)) == RC_OK || rc == RC_BUSY)
			{
				if (rc == RC_BUSY)
				{
					if (timeout++ >= 3) break;
					sqlite3_sleep(100);
				}
			}
			sqlite3_backup_finish(backup);
			if (rc == RC_DONE) rc = 0;
			else if (rc == RC_BUSY || rc == RC_LOCKED) { _fprintf(stderr, "Error: source database is busy\n"); rc = 1; }
			else { _fprintf(stderr, "Error: %s\n", Main::ErrMsg(p->Ctx)); rc = 1; }
			Main::Close(src);
		}
		else if (c == 's' && !strncmp(args[0], "schema", n) && argsLength < 3)
		{
			struct CallbackData data;
			char *errMsg = 0;
			OpenCtx(p);
			memcpy(&data, p, sizeof(data));
			data.ShowHeader = 0;
			data.Mode = MODE_Semi;
			if ( rgsLength > 1)
			{
				int i;
				for (i = 0; args[1][i]; i++) args[1][i] = ToLower(args[1][i]);
				if (!strcmp(args[1], "sqlite_master"))
				{
					char *new_argv[2], *new_colv[2];
					new_argv[0] = "CREATE TABLE sqlite_master (\n"
						"  type text,\n"
						"  name text,\n"
						"  tbl_name text,\n"
						"  rootpage integer,\n"
						"  sql text\n"
						")";
					new_argv[1] = 0;
					new_colv[0] = "sql";
					new_colv[1] = 0;
					callback(&data, 1, new_argv, new_colv);
					rc = RC_OK;
				}
				else if (!strcmp(args[1], "sqlite_temp_master"))
				{
					char *new_argv[2], *new_colv[2];
					new_argv[0] = "CREATE TEMP TABLE sqlite_temp_master (\n"
						"  type text,\n"
						"  name text,\n"
						"  tbl_name text,\n"
						"  rootpage integer,\n"
						"  sql text\n"
						")";
					new_argv[1] = 0;
					new_colv[0] = "sql";
					new_colv[1] = 0;
					callback(&data, 1, new_argv, new_colv);
					rc = RC_OK;
				}
				else
				{
					shellStatic = args[1];
					rc = Main:Exec(p->Ctx,
						"SELECT sql FROM "
						"  (SELECT sql sql, type type, tbl_name tbl_name, name name, rowid x"
						"     FROM sqlite_master UNION ALL"
						"   SELECT sql, type, tbl_name, name, rowid FROM sqlite_temp_master) "
						"WHERE lower(tbl_name) LIKE shellstatic()"
						"  AND type!='meta' AND sql NOTNULL "
						"ORDER BY substr(type,2,1), "
						" CASE type WHEN 'view' THEN rowid ELSE name END",
						callback, &data, &errMsg);
					shellStatic = nullptr;
				}
			}
			else
			{
				rc = Main::Exec(p->Ctx,
					"SELECT sql FROM "
					"  (SELECT sql sql, type type, tbl_name tbl_name, name name, rowid x"
					"     FROM sqlite_master UNION ALL"
					"   SELECT sql, type, tbl_name, name, rowid FROM sqlite_temp_master) "
					"WHERE type!='meta' AND sql NOTNULL AND name NOT LIKE 'sqlite_%'"
					"ORDER BY substr(type,2,1),"
					" CASE type WHEN 'view' THEN rowid ELSE name END",
					callback, &data, &errMsg);
			}
			if (errMsg) { _fprintf(stderr,"Error: %s\n", errMsg); _free(errMsg); rc = 1; }
			else if (rc != RC_OK) { _fprintf(stderr,"Error: querying schema information\n"); rc = 1; }
			else rc = 0;
		}
		else if (c == 's' && !strncmp(args[0], "separator", n) && argsLength == 2)
		{
			_snprintf(sizeof(p->Separator), p->Separator, "%.*s", (int)sizeof(p->Separator)-1, args[1]);
		}
		else if (c == 's' && !strncmp(args[0], "show", n) && argsLength == 1)
		{
			_fprintf(p->Out,"%9.9s: %s\n","echo", p->EchoOn ? "on" : "off");
			_fprintf(p->Out,"%9.9s: %s\n","explain", p->ExplainPrev.Valid ? "on" :"off");
			_fprintf(p->Out,"%9.9s: %s\n","headers", p->ShowHeader ? "on" : "off");
			_fprintf(p->Out,"%9.9s: %s\n","mode", modeDescr[p->Mode]);
			_fprintf(p->Out,"%9.9s: ", "nullvalue");
			OutputCString(p->Out, p->NullValue);
			_fprintf(p->Out, "\n");
			_fprintf(p->Out,"%9.9s: %s\n","output", _strlen(p->Outfile) ? p->Outfile : "stdout");
			_fprintf(p->Out,"%9.9s: ", "separator");
			OutputCString(p->Out, p->Separator);
			_fprintf(p->Out, "\n");
			_fprintf(p->Out,"%9.9s: %s\n","stats", p->StatsOn ? "on" : "off");
			_fprintf(p->Out,"%9.9s: ","width");
			for (int i = 0; i < (int)_lengthof(p->ColWidth) && p->ColWidth[i] != 0; i++)
				_fprintf(p->Out, "%d ", p->ColWidth[i]);
			_fprintf(p->Out,"\n");
		}
		else if (c == 's' && !strncmp(args[0], "stats", n) && argsLength > 1 && argsLength < 3)
		{
			p->StatsOn = BooleanValue(args[1]);
		}
		else if (c == 't' && n > 1 && !strncmp(args[0], "tables", n) && argsLength < 3)
		{
			OpenCtx(p);
			Vdbe *stmt;
			rc = Prepare::PreparevV(p->Ctx, "PRAGMA database_list", -1, &stmt, 0);
			if (rc) return rc;
			char *sql = _mprintf(
				"SELECT name FROM sqlite_master"
				" WHERE type IN ('table','view')"
				"   AND name NOT LIKE 'sqlite_%%'"
				"   AND name LIKE ?1");
			while (stmt->Step() == RC_ROW)
			{
				const char *dbName = (const char *)Vdbe::Column_Text(stmt, 1);
				if (!dbName || !strcmp(dbName, "main")) continue;
				if (!strcmp(dbName, "temp"))
					sql = _mprintf(
					"%z UNION ALL "
					"SELECT 'temp.' || name FROM sqlite_temp_master"
					" WHERE type IN ('table','view')"
					"   AND name NOT LIKE 'sqlite_%%'"
					"   AND name LIKE ?1", sql);
				else
					sql = sqlite3_mprintf(
					"%z UNION ALL "
					"SELECT '%q.' || name FROM \"%w\".sqlite_master"
					" WHERE type IN ('table','view')"
					"   AND name NOT LIKE 'sqlite_%%'"
					"   AND name LIKE ?1", sql, dbName, dbName);
			}
			Vdbe::Finalize(stmt);
			sql = _mprintf("%z ORDER BY 1", sql);
			rc = Prepare::PrepareV2(p->Ctx, sql, -1, &stmt, 0);
			_free(sql);
			if (rc) return rc;
			int rows, allocs;
			rows = allocs = 0;
			char **newResults = nullptr;
			if (argsLength > 1)
				Vdbe::Bind_Text(stmt, 1, args[1], -1, DESTRUCTOR_TRANSIENT);
			else
				Vdbe::Bind_Text(stmt, 1, "%", -1, DESTRUCTOR_STATIC);
			while (stmt->Step() == RC_ROW)
			{
				if (rows >= allocs)
				{
					int n = allocs*2 + 10;
					char **newResults = _realloc(results, sizeof(results[0])*n);
					if (!newResults)
					{
						_fprintf(stderr, "Error: out_ of memory\n");
						break;
					}
					allocs = n;
					results = newResults;
				}
				results[rows] = _mprintf("%s", Vdbe::Column_Text(stmt, 0));
				if (results[rows]) rows++;
			}
			Vdbe::Finalize(stmt);        
			if (rows > 0)
			{
				int i;
				int maxlen = 0;
				for (i = 0; i < rows; i++)
				{
					int len = _strlen(results[i]);
					if (len > maxlen) maxlen = len;
				}
				int printCols = 80/(maxlen+2);
				if (printCols < 1) printCols = 1;
				int printRows = (rows + printCols - 1)/printCols;
				for (i = 0; i < printRows; i++)
				{
					for (int j = i; j < rows; j += pintRows)
					{
						char *sp = (j < printRow ? "" : "  ");
						printf("%s%-*s", sp, maxlen, (results[j] ? rResults[j] : ""));
					}
					printf("\n");
				}
			}
			for (int ii = 0; ii < rows; ii++) _free(results[ii]);
			_free(results);
		}
		else if (c == 't' && n >= 8 && !strncmp(args[0], "testctrl", n) && argsLength >= 2)
		{
			static const struct
			{
				const char *CtrlName;   // Name of a test-control option
				int CtrlCode;           // Integer code for that option
			} _ctrls[] = {
				{ "prng_save",             TESTCTRL_PRNG_SAVE              },
				{ "prng_restore",          TESTCTRL_PRNG_RESTORE           },
				{ "prng_reset",            TESTCTRL_PRNG_RESET             },
				{ "bitvec_test",           TESTCTRL_BITVEC_TEST            },
				{ "fault_install",         TESTCTRL_FAULT_INSTALL          },
				{ "benign_malloc_hooks",   TESTCTRL_BENIGN_MALLOC_HOOKS    },
				{ "pending_byte",          TESTCTRL_PENDING_BYTE           },
				{ "assert",                TESTCTRL_ASSERT                 },
				{ "always",                TESTCTRL_ALWAYS                 },
				{ "reserve",               TESTCTRL_RESERVE                },
				{ "optimizations",         TESTCTRL_OPTIMIZATIONS          },
				{ "iskeyword",             TESTCTRL_ISKEYWORD              },
				{ "scratchmalloc",         TESTCTRL_SCRATCHMALLOC          },
			};
			int testctrl = -1;
			int rc = 0;
			int i, n;
			OpenCtx(p);
			// convert testctrl text option to value. allow any unique prefix of the option name, or a numerical value.
			int n = _strlen(args[1]);
			for (i = 0; i < (int)_lengthof(_ctrls); i++)
			{
				if (!strncmp(args[1], _ctrls[i].CtrlName, n))
				{
					if (testctrl < 0)
						testctrl = _ctrls[i].CtrlCode;
					else
					{
						_fprintf(stderr, "ambiguous option name: \"%s\"\n", args[1]);
						testctrl = -1;
						break;
					}
				}
			}
			if (testctrl < 0) testctrl = atoi(args[1]);
			if ((testctrl < TESTCTRL_FIRST) || (testctrl > TESTCTRL_LAST))
				_fprintf(stderr,"Error: invalid testctrl option: %s\n", args[1]);
			else
			{
				switch (testctrl)
				{
				case TESTCTRL_OPTIMIZATIONS:
				case TESTCTRL_RESERVE:
					// sqlite3_test_control(int, db, int)
					if (argsLength == 3)
					{
						int opt = (int)strtol(args[2], 0, 0);
						rc = sqlite3_test_control(testctrl, p->Ctx, opt);
						printf("%d (0x%08x)\n", rc, rc);
					}
					else
						_fprintf(stderr,"Error: testctrl %s takes a single int option\n", args[1]);
					break;
				case TESTCTRL_PRNG_SAVE:
				case TESTCTRL_PRNG_RESTORE:
				case TESTCTRL_PRNG_RESET:
					// sqlite3_test_control(int)
					if (argsLength == 2)
					{
						rc = sqlite3_test_control(testctrl);
						printf("%d (0x%08x)\n", rc, rc);
					}
					else
						_fprintf(stderr,"Error: testctrl %s takes no options\n", args[1]);
					break;
				case TESTCTRL_PENDING_BYTE:
					// sqlite3_test_control(int, uint)
					if (argsLength == 3)
					{
						unsigned int opt = (unsigned int)atoi(args[2]);
						rc = sqlite3_test_control(testctrl, opt);
						printf("%d (0x%08x)\n", rc, rc);
					}
					else
						_fprintf(stderr,"Error: testctrl %s takes a single unsigned int option\n", args[1]);
					break;
				case TESTCTRL_ASSERT:
				case TESTCTRL_ALWAYS:
					// sqlite3_test_control(int, int)
					if (argsLength == 3)
					{
						int opt = atoi(args[2]);        
						rc = sqlite3_test_control(testctrl, opt);
						printf("%d (0x%08x)\n", rc, rc);
					}
					else
						_fprintf(stderr,"Error: testctrl %s takes a single int option\n", args[1]);
					break;
#ifdef SQLITE_N_KEYWORD
				case TESTCTRL_ISKEYWORD:
					// sqlite3_test_control(int, char *)
					if (argsLength == 3)
					{
						const char *opt = args[2];
						rc = sqlite3_test_control(testctrl, opt);
						printf("%d (0x%08x)\n", rc, rc);
					}
					else
						_fprintf(stderr,"Error: testctrl %s takes a single char * option\n", args[1]);
					break;
#endif
				case TESTCTRL_BITVEC_TEST:         
				case TESTCTRL_FAULT_INSTALL:       
				case TESTCTRL_BENIGN_MALLOC_HOOKS: 
				case TESTCTRL_SCRATCHMALLOC:       
				default:
					_fprintf(stderr,"Error: CLI support for testctrl %s not implemented\n", args[1]);
					break;
				}
			}
		}
		else if (c == 't' && n > 4 && !strncmp(args[0], "timeout", n) && argsLength == 2)
		{
			OpenCtx(p);
			sqlite3_busy_timeout(p->Ctx, atoi(args[1]));
		}
		else if (HAS_TIMER && c == 't' && n >= 5 && !strncmp(args[0], "timer", n) && argsLength == 2)
		{
			enableTimer = BooleanValue(args[1]);
		}
		else if (c == 't' && !strncmp(args[0], "trace", n) && argsLength > 1)
		{
			OpenCtx(p);
			OutputFileClose(p->traceOut);
			p->TraceOut = OutputFileOpen(args[1]);
#if !defined(OMIT_TRACE) && !defined(OMIT_FLOATING_POINT)
			if (p->TraceOut == 0)
				sqlite3_trace(p->Ctx, 0, 0);
			else
				sqlite3_trace(p->Ctx, SqlTraceCallback, p->TraceOut);
#endif
		}
		else if (c == 'v' && !strncmp(args[0], "version", n))
		{
			printf("SQLite %s %s\n", sqlite3_libversion(), sqlite3_sourceid());
		}
		else if (c == 'v' && !strncmp(args[0], "vfsname", n))
		{
			const char *dbName = (argsLength == 2 ? args[1] : "main");
			char *vfsName = 0;
			if (p->Ctx)
			{
				sqlite3_file_control(p->Ctx, dbName, SQLITE_FCNTL_VFSNAME, &vfsName);
				if (vfsName)
				{
					printf("%s\n", vfsName);
					_free(vfsName);
				}
			}
		}
#if defined(_DEBUG) && defined(ENABLE_WHERETRACE)
		else if (c == 'w' && !strncmp(args[0], "wheretrace", n))
		{
			extern int sqlite3WhereTrace;
			sqlite3WhereTrace = atoi(args[1]);
		}
#endif
		else if (c == 'w' && !strncmp(args[0], "width", n) && argsLength > 1)
		{
			assert(argsLength <= _lengthof(args));
			for (int j = 1; j < argsLength && j < _lengthof(p->colWidth); j++)
				p->ColWidth[j-1] = atoi(args[j]);
		}
		else
		{
			_fprintf(stderr, "Error: unknown command or invalid arguments:  \"%s\". Enter \".help\" for help\n", args[0]);
			rc = 1;
		}
		return rc;
	}

#pragma endregion

#pragma region Parse

	// Return TRUE if a semicolon occurs anywhere in the first N characters of string z[].
	static bool _contains_semicolon(const char *z, int length)
	{
		for (int i = 0; i < length; i++) { if (z[i] == ';') return true; }
		return false;
	}

	// Test to see if a line consists entirely of whitespace.
	static bool _all_whitespace(const char *z)
	{
		for (; *z; z++)
		{
			if (IsSpace(z[0])) continue;
			if (*z == '/' && z[1] == '*')
			{
				z += 2;
				while (*z && (*z != '*' || z[1] != '/')) { z++; }
				if (!*z) return false;
				z++;
				continue;
			}
			if (*z == '-' && z[1] == '-')
			{
				z += 2;
				while (*z && *z != '\n') { z++; }
				if (!*z) return true;
				continue;
			}
			return false;
		}
		return true;
	}

	// Return TRUE if the line typed in is an SQL command terminator other than a semi-colon.  The SQL Server style "go" command is understood as is the Oracle "/".
	static bool _is_command_terminator(const char *line)
	{
		while (IsSpace(line[0])) { line++; };
		if (line[0] == '/' && _all_whitespace(&line[1])) return true; // Oracle
		if (ToLower(line[0]) == 'g' && ToLower(line[1]) == 'o' && _all_whitespace(&line[2])) return true; // SQL Server
		return false;
	}

	// Return true if sql is a complete SQL statement.  Return false if it ends in the middle of a string literal or C-style comment.
	static int _is_complete(char *sql, int sqlLength)
	{
		if (!sql) return 1;
		sql[sqlLength] = ';';
		sql[sqlLength+1] = 0;
		RC rc = Complete::Complete_(sql);
		sql[sqlLength] = 0;
		return rc;
	}

	// Read input from *in and process it.  If *in==0 then input is interactive - the user is typing it it.  Otherwise, input
	// is coming from a file or device.  A prompt is issued and history is saved only if input is interactive.  An interrupt signal will
	// cause this routine to exit immediately, unless input is interactive.
	//
	// Return the number of errors.
	static bool ProcessInput(struct CallbackData *p, FILE *in)
	{
		char *line = 0;
		char *sql = 0;
		int sqlLength = 0;
		int sqlLengthPrior = 0;
		char *errMsg;
		int rc;
		int errCnt = 0;
		int lineno = 0;
		int startline = 0;
		while (errCnt == 0 || !_bailOnError || (in == 0 && _stdinIsInteractive))
		{
			_fflush(p->Out);
			free(line);
			line = OneInputLine(sql, in);
			if (!line) // End of input
			{
				if (_stdinIsInteractive) printf("\n");
				break;
			}
			if (_seenInterrupt)
			{
				if (in) break;
				_seenInterrupt = 0;
			}
			lineno++;
			if ((!sql || sql[0] == 0) && _all_whitespace(line)) continue;
			if (line && line[0] == '.' && sqlLength == 0)
			{
				if (p->EchoOn) printf("%s\n", line);
				rc = DoMetaCommand(line, p);
				if (rc == 2) break; // exit requested
				else if (rc) errCnt++;
				continue;
			}
			if (_is_command_terminator(line) && _is_complete(sql, sqlLength))
				memcpy(line, ";", 2);
			sqlLengthPrior = sqlLength;
			if (!sql)
			{
				int i;
				for (i = 0; line[i] && IsSpace(line[i]); i++) { }
				if (line[i] != 0)
				{
					sqlLength = _strlen(line);
					sql = malloc(sqlLength+3);
					if (!sql)
					{
						_fprintf(stderr, "Error: out_ of memory\n");
						exit(1);
					}
					memcpy(sql, line, sqlLength+1);
					startline = lineno;
				}
			}
			else
			{
				int lineLength = _strlen(line);
				sql = realloc(sql, sqlLength + lineLength + 4);
				if (!sql)
				{
					_fprintf(stderr,"Error: out_ of memory\n");
					exit(1);
				}
				sql[sqlLength++] = '\n';
				memcpy(&sql[sqlLength], line, lineLength+1);
				sqlLength += lineLength;
			}
			if (sql && _contains_semicolon(&sql[sqlLengthPrior], sqlLength-sqlLengthPrior) && Complete::Complete_(sql))
			{
				p->Cnt = 0;
				OpenCtx(p);
				BEGIN_TIMER;
				rc = ShellExec(p->Ctx, sql, ShellCallback, p, &errMsg);
				END_TIMER;
				if (rc || errMsg)
				{
					char prefix[100];
					if (in != 0 || !_stdinIsInteractive)
						_snprintf(sizeof(prefix), prefix, "Error: near line %d:", startline);
					else
						_snprintf(sizeof(prefix), prefix, "Error:");
					if (errMsg)
					{
						_fprintf(stderr, "%s %s\n", prefix, errMsg);
						_free(errMsg);
						errMsg = nullptr;
					}
					else
						_fprintf(stderr, "%s %s\n", prefix, Main::ErrMsg(p->Ctx));
					errCnt++;
				}
				free(sql);
				sql = nullptr;
				sqlLength = 0;
			}
		}
		if (sql)
		{
			if (!_all_whitespace(sql))
				_fprintf(stderr, "Error: incomplete SQL: %s\n", sql);
			free(sql);
		}
		free(line);
		return (errCnt > 0);
	}

#pragma endregion

#pragma region Name2

	// Return a pathname which is the user's home directory.  A 0 return indicates an error of some kind.
	static char *FindHomeDir()
	{
		static char *home_dir = NULL;
		if (home_dir) return home_dir;
#if !defined(_WIN32) && !defined(WIN32) && !defined(_WIN32_WCE) && !defined(__RTP__) && !defined(_WRS_KERNEL)
		{
			struct passwd *pwent;
			uid_t uid = getuid();
			if ((pwent = getpwuid(uid)) != NULL)
				home_dir = pwent->pw_dir;
		}
#endif
#if defined(_WIN32_WCE)
		home_dir = "/"; // Windows CE (arm-wince-mingw32ce-gcc) does not provide getenv()
#else
#if defined(_WIN32) || defined(WIN32)
		if (!home_dir) home_dir = getenv("USERPROFILE");
#endif
		if (!home_dir) home_dir = getenv("HOME");

#if defined(_WIN32) || defined(WIN32)
		if (!home_dir)
		{
			int n;
			char *drive = getenv("HOMEDRIVE");
			char *path = getenv("HOMEPATH");
			if (drive && path)
			{
				int n = _strlen(drive) + _strlen(path) + 1;
				home_dir = malloc(n);
				if (!home_dir) return nullptr;
				_snprintf(n, home_dir, "%s%s", drive, path);
				return home_dir;
			}
			home_dir = "c:\\";
		}
#endif
#endif // !_WIN32_WCE
		if (home_dir)
		{
			int n = _strlen(home_dir) + 1;
			char *z = malloc(n);
			if (z) memcpy(z, home_dir, n);
			home_dir = z;
		}
		return home_dir;
	}

	// Read input from the file given by sqliterc_override.  Or if that parameter is NULL, take input from ~/.sqliterc
	// Returns the number of errors.
	static int ProcessSqliteRC(struct CallbackData *p,  const char *sqliterc_override)
	{
		char *home_dir = nullptr;
		const char *sqliterc = sqliterc_override;
		char *buf = 0;
		FILE *in = nullptr;
		int rc = 0;
		if (sqliterc == nullptr)
		{
			home_dir = FindHomeDir();
			if (!home_dir)
			{
#if !defined(__RTP__) && !defined(_WRS_KERNEL)
				_fprintf(stderr,"%s: Error: cannot locate your home directory\n", Argv0);
#endif
				return 1;
			}
			Main::Initialize();
			buf = _mprintf("%s/.sqliterc", home_dir);
			sqliterc = buf;
		}
		in = fopen(sqliterc, "rb");
		if (in)
		{
			if (_stdinIsInteractive)
				_fprintf(stderr,"-- Loading resources from %s\n", sqliterc);
			rc = ProcessInput(p, in);
			fclose(in);
		}
		_free(buf);
		return rc;
	}

#pragma endregion

#pragma region Main

	// Show available command line options
	static const char _options[] = 
		"   -bail                stop after hitting an error\n"
		"   -batch               force batch I/O\n"
		"   -column              set output mode to 'column'\n"
		"   -cmd COMMAND         run \"COMMAND\" before reading stdin\n"
		"   -csv                 set output mode to 'csv'\n"
		"   -echo                print commands before execution\n"
		"   -init FILENAME       read/process named file\n"
		"   -[no]header          turn headers on or off\n"
#if defined(ENABLE_MEMSYS3) || defined(ENABLE_MEMSYS5)
		"   -heap SIZE           Size of heap for memsys3 or memsys5\n"
#endif
		"   -help                show this message\n"
		"   -html                set output mode to HTML\n"
		"   -interactive         force interactive I/O\n"
		"   -line                set output mode to 'line'\n"
		"   -list                set output mode to 'list'\n"
#ifdef ENABLE_MULTIPLEX
		"   -multiplex           enable the multiplexor VFS\n"
#endif
		"   -nullvalue TEXT      set text string for NULL values. Default ''\n"
		"   -separator SEP       set output field separator. Default: '|'\n"
		"   -stats               print memory stats before each finalize\n"
		"   -version             show SQLite version\n"
		"   -vfs NAME            use NAME as the default VFS\n"
#ifdef ENABLE_VFSTRACE
		"   -vfstrace            enable tracing of all VFS calls\n"
#endif
		;
	static void Usage(bool showDetail)
	{
		_fprintf(stderr,
			"Usage: %s [OPTIONS] FILENAME [SQL]\n"  
			"FILENAME is the name of an SQLite database. A new database is created\n"
			"if the file does not previously exist.\n", Argv0);
		if (showDetail)
			_fprintf(stderr, "OPTIONS include:\n%s", _options);
		else
			_fprintf(stderr, "Use the -help option for additional information\n");
		exit(1);
	}

	// Initialize the state information in data
	static void MainInit(struct CallbackData *data)
	{
		memset(data, 0, sizeof(*data));
		data->Mode = MODE_List;
		memcpy(data->Separator,"|", 2);
		data->ShowHeader = 0;
		Main::Config(CONFIG_URI, 1);
		Main::Config(CONFIG_LOG, shellLog, data);
		_snprintf(sizeof(_mainPrompt), _mainPrompt, "sqlite> ");
		_snprintf(sizeof(_continuePrompt), _continuePrompt,"   ...> ");
		Main::Config(CONFIG_SINGLETHREAD);
	}

	// Get the argument to an --option.  Throw an error and die if no argument is available.
	static char *CmdlineOptionValue(int argc, char **argv, int i)
	{
		if (i == argc)
		{
			_fprintf(stderr, "%s: Error: missing argument to %s\n", argv[0], argv[argc-1]);
			exit(1);
		}
		return argv[i];
	}

	int main(int argc, char **argv)
	{
		char *errMsg = 0;
		struct CallbackData data;
		const char *initFile = 0;
		char *firstCmd = 0;
		int i;
		int rc = 0;

		if (strcmp(sqlite3_sourceid(),SQLITE_SOURCE_ID))
		{
			_fprintf(stderr, "SQLite header and source version mismatch\n%s\n%s\n", sqlite3_sourceid(), SQLITE_SOURCE_ID);
			exit(1);
		}
		Argv0 = argv[0];
		MainInit(&data);
		_stdinIsInteractive = isatty(0);

		// Make sure we have a valid signal handler early, before anything else is done.
#ifdef SIGINT
		signal(SIGINT, interrupt_handler);
#endif

		// Do an initial pass through the command-line argument to locate the name of the database file, the name of the initialization file,
		// the size of the alternative malloc heap, and the first command to execute.
		for (i = 1; i < argc; i++)
		{
			char *z;
			z = argv[i];
			if (z[0] != '-')
			{
				if (!data.DbFilename)
				{
					data.DbFilename = z;
					continue;
				}
				if (!firstCmd)
				{
					firstCmd = z;
					continue;
				}
				_fprintf(stderr, "%s: Error: too many options: \"%s\"\n", Argv0, argv[i]);
				_fprintf(stderr, "Use -help for a list of options.\n");
				return 1;
			}
			if (z[1] == '-') z++;
			if (!strcmp(z, "-separator") || !strcmp(z, "-nullvalue") || !strcmp(z, "-cmd")) CmdlineOptionValue(argc, argv, ++i);
			else if (!strcmp(z, "-init")) initFile = CmdlineOptionValue(argc, argv, ++i);
			// Need to check for batch mode here to so we can avoid printing informational messages (like from ProcessSqliteRC) before 
			// we do the actual processing of arguments later in a second pass.
			else if (!strcmp(z, "-batch")) _stdinIsInteractive = 0;
			else if (!strcmp(z, "-heap"))
			{
#if defined(ENABLE_MEMSYS3) || defined(ENABLE_MEMSYS5)
				const char *sizeAsString = CmdlineOptionValue(argc, argv, ++i);
				int64 sizeHeap = atoi(sizeAsString);
				int c;
				for (int j = 0; (c = sizeAsString[j]) != 0; j++)
				{
					if (c == 'M') { sizeHeap *= 1000000; break; }
					if (c == 'K') { sizeHeap *= 1000; break; }
					if (c == 'G') { sizeHeap *= 1000000000; break; }
				}
				if (sizeHeap > 0x7fff0000) sizeHeap = 0x7fff0000;
				Main::Config(CONFIG_HEAP, malloc((int)sizeHeap), (int)sizeHeap, 64);
			}
#endif
#ifdef ENABLE_VFSTRACE
			else if (!strcmp(z, "-vfstrace"))
			{
				extern int vfstrace_register(const char *zTraceName, const char *zOldVfsName, int (*xOut)(const char*,void*), void *pOutArg, int makeDefault);
				vfstrace_register("trace", 0, (int(*)(const char*,void*))fputs, stderr, 1);
			}
#endif
#ifdef ENABLE_MULTIPLEX
			else if (!strcmp(z, "-multiplex"))
			{
				extern int sqlite3_multiple_initialize(const char*,int);
				sqlite3_multiplex_initialize(0, 1);
			}
#endif
			else if (!strcmp(z, "-vfs"))
			{
				VSystem *vfs = VSystem::Find(CmdlineOptionValue(argc, argv, ++i));
				if (vfs)
					sqlite3_vfs_register(vfs, 1);
				else
				{
					_fprintf(stderr, "no such VFS: \"%s\"\n", argv[i]);
					exit(1);
				}
			}
		}
		if (!data.DbFilename)
		{
#ifndef OMIT_MEMORYDB
			data.DbFilename = ":memory:";
#else
			_fprintf(stderr,"%s: Error: no database filename specified\n", Argv0);
			return 1;
#endif
		}
		data.out_ = stdout;

		// Go ahead and open the database file if it already exists.  If the file does not exist, delay opening it.  This prevents empty database
		// files from being created if a user mistypes the database name argument to the sqlite command-line tool.
		if (!access(data.DbFilename, 0))
			OpenCtx(&data);

		// Process the initialization file if there is one.  If no -init option is given on the command line, look for a file named ~/.sqliterc and
		// try to process it.
		rc = ProcessSqliteRC(&data, initFile);
		if (rc > 0)
			return rc;

		// Make a second pass through the command-line argument and set options.  This second pass is delayed until after the initialization
		// file is processed so that the command-line arguments will override
		// settings in the initialization file.
		for (i = 1; i < argc; i++)
		{
			char *z = argv[i];
			if (z[0] != '-' ) continue;
			if (z[1] == '-' ) z++;
			if (!strcmp(z, "-init")) i++;
			else if (!strcmp(z, "-html")) data.Mode = MODE_Html;
			else if (!strcmp(z, "-list")) data.Mode = MODE_List;
			else if (!strcmp(z, "-line")) data.Mode = MODE_Line;
			else if (!strcmp(z, "-column")) data.Mode = MODE_Column;
			else if (!strcmp(z, "-csv")) { data.Mode = MODE_Csv; memcpy(data.Separator, ",", 2); }
			else if (!strcmp(z, "-separator")) _snprintf(sizeof(data.Separator), data.Separator, "%s", CmdlineOptionValue(argc,argv,++i));
			else if (!strcmp(z, "-nullvalue")) _snprintf(sizeof(data.NullValue), data.NullValue, "%s", CmdlineOptionValue(argc,argv,++i));
			else if (!strcmp(z, "-header")) data.ShowHeader = 1;
			else if (!strcmp(z, "-noheader")) data.ShowHeader = 0;
			else if (!strcmp(z, "-echo")) data.EchoOn = 1;
			else if (!strcmp(z, "-stats")) data.StatsOn = 1;
			else if (!strcmp(z, "-bail")) _bailOnError = 1;
			else if (!strcmp(z, "-version")) { printf("%s %s\n", sqlite3_libversion(), sqlite3_sourceid()); return 0; }
			else if (!strcmp(z, "-interactive")) _stdinIsInteractive = 1;
			else if (!strcmp(z, "-batch")) _stdinIsInteractive = 0;
			else if (!strcmp(z, "-heap")) i++;
			else if (!strcmp(z, "-vfs")) i++;
#ifdef ENABLE_VFSTRACE
			else if (!strcmp(z, "-vfstrace")) i++;
#endif
#ifdef ENABLE_MULTIPLEX
			else if (!strcmp(z, "-multiplex")) i++;
#endif
			else if (!strcmp(z, "-help")) Usage(1);
			else if (!strcmp(z, "-cmd")=)
			{
				if (i == argc-1) break;
				z = CmdlineOptionValue(argc,argv,++i);
				if (z[0] == '.')
				{
					rc = DoMetaCommand(z, &data);
					if (rc && _bailOnError) return rc;
				}
				else
				{
					OpenCtx(&data);
					rc = ShellExec(data.Ctx, z, ShellCallback, &data, &errMsg);
					if (errMsg)
					{
						_fprintf(stderr,"Error: %s\n", errMsg);
						if (_bailOnError) return (rc ? rc : 1);
					}
					else if (rc)
					{
						_fprintf(stderr, "Error: unable to process SQL \"%s\"\n", z);
						if (_bailOnError) return rc;
					}
				}
			}
			else
			{
				_fprintf(stderr, "%s: Error: unknown option: %s\n", Argv0, z);
				_fprintf(stderr, "Use -help for a list of options.\n");
				return 1;
			}
		}

		if (firstCmd)
		{
			// Run just the command that follows the database name
			if (firstCmd[0] == '.')
				rc = DoMetaCommand(firstCmd, &data);
			else
			{
				OpenCtx(&data);
				rc = ShellExec(data.db, firstCmd, ShellCallback, &data, &errMsg);
				if (errMsg)
				{
					_fprintf(stderr,"Error: %s\n", errMsg);
					return (rc ? rc : 1);
				}
				else if (rc)
				{
					_fprintf(stderr,"Error: unable to process SQL \"%s\"\n", firstCmd);
					return rc;
				}
			}
		}
		else
		{
			// Run commands received from standard input
			if (_stdinIsInteractive)
			{
				printf(
					"SQLite version %s %.19s\n" /*extra-version-info*/
					"Enter \".help\" for instructions\n"
					"Enter SQL statements terminated with a \";\"\n",
					sqlite3_libversion(), sqlite3_sourceid() );
				char *history = 0;
				char *home = FindHomeDir();
				if (home)
				{
					int historyLength = _strlen(home) + 20;
					if ((history = malloc(historyLength))!=0 ){
						sqlite3_snprintf(historyLength, history,"%s/.sqlite_history", home);
					}
				}
#if defined(HAVE_READLINE) && HAVE_READLINE == 1
				if (history) read_history(history);
#endif
				rc = ProcessInput(&data, 0);
				if (history)
				{
					stifle_history(100);
					write_history(history);
					free(history);
				}
			}
			else
				rc = ProcessInput(&data, stdin);
		}
		SetTableName(&data, 0);
		if (data.Ctx)
			Main::Close(data.Ctx);
		return rc;
	}

#pragma endregion
}
