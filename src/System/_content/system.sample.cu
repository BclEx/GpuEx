#include "hip/hip_runtime.h"
#include <RuntimeHost.h>
#include <Core\Core.cu.h>

void fileAccessWithCuda(const char *path);

__global__ void fileAccessKernel(const char *path)
{
	auto vfs = VSystem::FindVfs(nullptr);
	auto file = (VFile *)_allocZero(vfs->SizeOsFile);
	auto rc = vfs->Open(path, file, (VSystem::OPEN)(VSystem::OPEN_CREATE|VSystem::OPEN_READWRITE|VSystem::OPEN_MAIN_DB), nullptr);
	file->Write4(0, 123145);
	file->Close();
}

int main()
{	
	// Write values to filesytem.
	fileAccessWithCuda("\\T_\\Test.db");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaErrorCheck(hipDeviceReset());

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void fileAccessWithCuda(const char *path)
{
	cudaDeviceHeap deviceHeap; memset(&deviceHeap, 0, sizeof(deviceHeap));

	// Set DeviceFlags for hipDeviceMapHost
	cudaErrorCheckF(hipSetDeviceFlags(hipDeviceMapHost), goto Error);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaErrorCheckF(hipSetDevice(0), goto Error);

	deviceHeap = cudaDeviceHeapCreate();
	cudaDeviceHeapSelect(deviceHeap);

	// Initialize Sentinel if required.
#if OS_MAP
	CoreS::VSystemSentinel::Initialize();
#endif

	// Launch a kernel on the GPU with one thread for each element.
	fileAccessKernel<<<1, 1>>>(path);

	// Shutdown Sentinel if required.
#if OS_MAP
	CoreS::VSystemSentinel::Shutdown();
#endif

	// Check for any errors launching the kernel
	cudaErrorCheckF(hipGetLastError(), goto Error);

	// cudaDeviceHeapSynchronize..
	cudaErrorCheckF(cudaDeviceHeapSynchronize(deviceHeap), goto Error);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaErrorCheckF(hipDeviceSynchronize(), goto Error);

Error:
	cudaDeviceHeapDestroy(deviceHeap);
}
