#include "hip/hip_runtime.h"
#include "..\System.net\Core\Core.cu.h"

__device__ static void TestVFS()
{
}

// NATIVE: assert
__global__ static void testSystemIO0(void *r)
{
	_runtimeSetHeap(r);
	MutexEx masterMutex;
	RC rc = SysEx::PreInitialize(masterMutex);
	SysEx::PostInitialize(masterMutex);
	//
	TestVFS();
	//
	SysEx::Shutdown();
	printf("System.IO: 0\n");
}

#if __HIPCC__
void __testSystemIO(cudaDeviceHeap &r)
{
	testSystemIO0<<<1, 1>>>(r.heap); cudaDeviceHeapSynchronize(r);
}
#else
void __testSystemIO(cudaDeviceHeap &r)
{
	testSystemIO0(r.heap);
}
#endif