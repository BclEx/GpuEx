#include "hip/hip_runtime.h"
#include "..\System.net\Core\Core.cu.h"

__device__ static void TestVFS()
{
	auto vfs = VSystem::FindVfs(nullptr);
	auto file = (VFile *)_allocZero(vfs->SizeOsFile);
	auto rc = vfs->Open("C:\\T_\\Test.db", file, (VSystem::OPEN)(VSystem::OPEN_CREATE|VSystem::OPEN_READWRITE|VSystem::OPEN_MAIN_DB), nullptr);
	file->Write4(0, 123145);
	file->Close();
}

// NATIVE: assert
__global__ static void testSystemIO0(void *r)
{
	_runtimeSetHeap(r);
	MutexEx masterMutex;
	RC rc = SysEx::PreInitialize(masterMutex);
	SysEx::PostInitialize(masterMutex);
	//
	TestVFS();
	//
	SysEx::Shutdown();
	printf("System.IO: 0\n");
}

#if __HIPCC__
void __testSystemIO(cudaDeviceHeap &r)
{
	testSystemIO0<<<1, 1>>>(r.heap); cudaDeviceHeapSynchronize(r);
}
#else
void __testSystemIO(cudaDeviceHeap &r)
{
	testSystemIO0(r.heap);
}
#endif