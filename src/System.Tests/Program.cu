#include <RuntimeHost.h>
#include "..\System.net\Core\Core.cu.h"

void __testSystem(cudaDeviceHeap &r);

#if __HIPCC__
void GMain(cudaDeviceHeap &r) {
#else
void main(int argc, char **argv) { cudaDeviceHeap r; memset(&r, 0, sizeof(r));
#endif
#if OS_MAP
	CoreS::VSystemSentinel::Initialize();
#endif
__testSystem(r);
#if OS_MAP
	CoreS::VSystemSentinel::Shutdown();
#endif
}

#if __HIPCC__
void __main(cudaDeviceHeap &r)
{	
	cudaDeviceHeapSelect(r);
	GMain(r); cudaDeviceHeapSynchronize(r);
}

int main(int argc, char **argv)
{
	cudaErrorCheck(hipSetDeviceFlags(hipDeviceMapHost));
	int deviceId = gpuGetMaxGflopsDeviceId();
	cudaErrorCheck(hipSetDevice(deviceId));
	hipDeviceReset();

	cudaDeviceHeap deviceHeap = cudaDeviceHeapCreate(256, 4096);
	//cudaDeviceFalloc fallocHost = cudaDeviceFallocCreate(100, 1024);

	// First initialize OpenGL context, so we can properly set the GL for CUDA. This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	//IVisualRender *render = new RuntimeVisualRender(deviceHeap);
	//if (!Visual::InitGL(render, &argc, argv)) return 0;
	//cudaErrorCheck(cudaGLSetGLDevice(deviceId));

	// run
	__main(deviceHeap);
	//Visual::Main();
	//Visual::Dispose();

	cudaDeviceHeapDestroy(deviceHeap);

	hipDeviceReset();
	printf("\nEnd"); char c; scanf("%c", &c);
	return 0;
}
#endif