#include "hip/hip_runtime.h"
#include "Tcl+Int.h"
#if 0 && OS_GPU
#include "Tcl+Gpu.h"

// The variable below caches the name of the current working directory in order to avoid repeated calls to getwd.  The string is malloc-ed. NULL means the cache needs to be refreshed.
__device__ static char *currentDir = NULL;

// Prototypes for local procedures defined in this file:
__device__ static int CleanupChildren(Tcl_Interp *interp, int numPids, int *pidPtr, FILE *errorId);
__device__ static char *GetFileType(int mode);
__device__ static int StoreStatData(Tcl_Interp *interp, char *varName, struct stat *statPtr);

/*
*----------------------------------------------------------------------
*
* Tcl_CdCmd --
*	This procedure is invoked to process the "cd" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_CdCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc > 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " dirName\"", (char *)NULL);
		return TCL_ERROR;
	}
	char *dirName;
	if (argc == 2) {
		dirName = argv[1];
	} else {
		dirName = "~";
	}
	dirName = Tcl_TildeSubst(interp, dirName);
	if (dirName == NULL) {
		return TCL_ERROR;
	}
	if (currentDir != NULL) {
		_freeFast(currentDir);
		currentDir = NULL;
	}
	if (_chdir(dirName) != 0) {
		Tcl_AppendResult(interp, "couldn't change working directory to \"", dirName, "\": ", Tcl_OSError(interp), (char *)NULL);
		return TCL_ERROR;
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_CloseCmd --
*	This procedure is invoked to process the "close" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_CloseCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId\"", (char *)NULL);
		return TCL_ERROR;
	}
	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[1], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	((Interp *)interp)->filePtrArray[fileno(filePtr->f)] = NULL;
	// First close the file (in the case of a process pipeline, there may be two files, one for the pipe at each end of the pipeline).
	int result = TCL_OK;
	if (filePtr->f2 != NULL) {
		if (_fclose(filePtr->f2)) {
			Tcl_AppendResult(interp, "error closing \"", argv[1], "\": ", Tcl_OSError(interp), "\n", (char *)NULL);
			result = TCL_ERROR;
		}
	}
	if (_fclose(filePtr->f)) {
		Tcl_AppendResult(interp, "error closing \"", argv[1], "\": ", Tcl_OSError(interp), "\n", (char *)NULL);
		result = TCL_ERROR;
	}
	// If the file was a connection to a pipeline, clean up everything associated with the child processes.
	if (filePtr->numPids > 0) {
		if (CleanupChildren(interp, filePtr->numPids, filePtr->pidPtr, filePtr->errorId) != TCL_OK) {
			result = TCL_ERROR;
		}
	}
	_freeFast((char *)filePtr);
	return result;
}

/*
*----------------------------------------------------------------------
*
* Tcl_EofCmd --
*	This procedure is invoked to process the "eof" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_EofCmd(ClientData notUsed, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId\"", (char *)NULL);
		return TCL_ERROR;
	}
	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[1], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	if (feof(filePtr->f)) {
		interp->result = "1";
	} else {
		interp->result = "0";
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_ExecCmd --
*	This procedure is invoked to process the "exec" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_ExecCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	int *pidPtr;
	int numPids;
	// See if the command is to be run in background;  if so, create the command, detach it, and return.
	if (argv[argc-1][0] == '&' && argv[argc-1][1] == 0) {
		argc--;
		argv[argc] = NULL;
		numPids = Tcl_CreatePipeline(interp, argc-1, argv+1, &pidPtr, (FILE **)NULL, (FILE **)NULL, (FILE **)NULL);
		if (numPids < 0) {
			return TCL_ERROR;
		}
		Tcl_DetachPids(numPids, pidPtr);
		_freeFast((char *)pidPtr);
		return TCL_OK;
	}

	// Create the command's pipeline.
	FILE *outputId; // File id for output pipe.  -1 means command overrode.
	FILE *errorId; // File id for temporary file containing error output.
	numPids = Tcl_CreatePipeline(interp, argc-1, argv+1, &pidPtr, (FILE **)NULL, &outputId, &errorId);
	if (numPids < 0) {
		return TCL_ERROR;
	}

	// Read the child's output (if any) and put it into the result.
	int result = TCL_OK;
	if (outputId) {
		while (true) {
#define BUFFER_SIZE 1000
			char buffer[BUFFER_SIZE+1];
			int count = fread(buffer, BUFFER_SIZE, 1, outputId);
			if (count == 0) {
				break;
			}
			if (count < 0) {
				Tcl_ResetResult(interp);
				Tcl_AppendResult(interp, "error reading from output pipe: ", Tcl_OSError(interp), (char *)NULL);
				result = TCL_ERROR;
				break;
			}
			buffer[count] = 0;
			Tcl_AppendResult(interp, buffer, (char *)NULL);
		}
		_fclose(outputId);
	}

	if (CleanupChildren(interp, numPids, pidPtr, errorId) != TCL_OK) {
		result = TCL_ERROR;
	}
	return result;
}

/*
*----------------------------------------------------------------------
*
* Tcl_ExitCmd --
*	This procedure is invoked to process the "exit" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_ExitCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 1 && argc != 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " ?returnCode?\"", (char *)NULL);
		return TCL_ERROR;
	}
	if (argc == 1) {
		exit(0);
	}
	int value;
	if (Tcl_GetInt(interp, argv[1], &value) != TCL_OK) {
		return TCL_ERROR;
	}
	exit(value);
	return TCL_OK; // Better not ever reach this!
}

/*
*----------------------------------------------------------------------
*
* Tcl_FileCmd --
*	This procedure is invoked to process the "file" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_FileCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	char *p;
	if (argc < 3) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " option name ?arg ...?\"", (char *)NULL);
		return TCL_ERROR;
	}
	char c = argv[1][0];
	int length = _strlen(argv[1]);

	// First handle operations on the file name.
	char *fileName = Tcl_TildeSubst(interp, argv[2]);
	if (fileName == NULL) {
		return TCL_ERROR;
	}
	if (c == 'd' && !_strncmp(argv[1], "dirname", length)) {
		if (argc != 3) {
			argv[1] = "dirname";
not3Args:
			Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " ", argv[1], " name\"", (char *)NULL);
			return TCL_ERROR;
		}
		p = _strrchr(fileName, '/');
		if (p == NULL) {
			interp->result = ".";
		} else if (p == fileName) {
			interp->result = "/";
		} else {
			*p = 0;
			Tcl_SetResult(interp, fileName, TCL_VOLATILE);
			*p = '/';
		}
		return TCL_OK;
	} else if (c == 'r' && !_strncmp(argv[1], "rootname", length) && length >= 2) {
		if (argc != 3) {
			argv[1] = "rootname";
			goto not3Args;
		}
		p = _strrchr(fileName, '.');
		char *lastSlash = _strrchr(fileName, '/');
		if (p == NULL || (lastSlash != NULL && lastSlash > p)) {
			Tcl_SetResult(interp, fileName, TCL_VOLATILE);
		} else {
			*p = 0;
			Tcl_SetResult(interp, fileName, TCL_VOLATILE);
			*p = '.';
		}
		return TCL_OK;
	} else if (c == 'e' && !_strncmp(argv[1], "extension", length) && length >= 3) {
		if (argc != 3) {
			argv[1] = "extension";
			goto not3Args;
		}
		p = _strrchr(fileName, '.');
		char *lastSlash = _strrchr(fileName, '/');
		if (p != NULL && (lastSlash == NULL || lastSlash < p)) {
			Tcl_SetResult(interp, p, TCL_VOLATILE);
		}
		return TCL_OK;
	} else if (c == 't' && !_strncmp(argv[1], "tail", length) && length >= 2) {
		if (argc != 3) {
			argv[1] = "tail";
			goto not3Args;
		}
		p = _strrchr(fileName, '/');
		if (p != NULL) {
			Tcl_SetResult(interp, p+1, TCL_VOLATILE);
		} else {
			Tcl_SetResult(interp, fileName, TCL_VOLATILE);
		}
		return TCL_OK;
	}

	// Next, handle operations that can be satisfied with the "access" kernel call.
	if (fileName == NULL) {
		return TCL_ERROR;
	}
	int mode = 0; // Initialized only to prevent compiler warning message.
	if (c == 'r' && !_strncmp(argv[1], "readable", length) && length >= 5) {
		if (argc != 3) {
			argv[1] = "readable";
			goto not3Args;
		}
		mode = R_OK;
checkAccess:
		if (_access(fileName, mode) == -1) {
			interp->result = "0";
		} else {
			interp->result = "1";
		}
		return TCL_OK;
	} else if (c == 'w' && !_strncmp(argv[1], "writable", length)) {
		if (argc != 3) {
			argv[1] = "writable";
			goto not3Args;
		}
		mode = W_OK;
		goto checkAccess;
	} else if (c == 'e' && !_strncmp(argv[1], "executable", length) && length >= 3) {
		if (argc != 3) {
			argv[1] = "executable";
			goto not3Args;
		}
		mode = X_OK;
		goto checkAccess;
	} else if (c == 'e' && !_strncmp(argv[1], "exists", length) && length >= 3) {
		if (argc != 3) {
			argv[1] = "exists";
			goto not3Args;
		}
		mode = F_OK;
		goto checkAccess;
	}

	// Next, handle operations on the file
	if (c == 'd' && !_strncmp(argv[1], "delete", length) && length >= 3) {
		if (argc != 3) {
			argv[1] = "delete";
			goto not3Args;
		}
		if (_unlink(fileName) == -1 && __errno != ENOENT) {
			Tcl_AppendResult(interp, "couldn't delete \"", argv[2], "\": ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		return TCL_OK;
	}
	else if (!_strcmp(argv[1], "rename")) {
		if (argc != 4) {
			Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " ", argv[1], " source target\"", (char *)NULL);
			return TCL_ERROR;
		}
		if (!_rename(argv[2], argv[3])) {
			Tcl_AppendResult(interp, "couldn't rename \"", argv[2], "\": ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		return TCL_OK;
	}

	// Lastly, check stuff that requires the file to be stat-ed.
	int statOp;
	struct stat statBuf;
	if (c == 'a' && !_strncmp(argv[1], "atime", length)) {
		if (argc != 3) {
			argv[1] = "atime";
			goto not3Args;
		}
		if (__stat(fileName, &statBuf) == -1) {
			goto badStat;
		}
		_sprintf(interp->result, "%ld", statBuf.st_atime);
		return TCL_OK;
	} else if (c == 'i' && !_strncmp(argv[1], "isdirectory", length) && length >= 3) {
		if (argc != 3) {
			argv[1] = "isdirectory";
			goto not3Args;
		}
		statOp = 2;
	} else if (c == 'i' && !_strncmp(argv[1], "isfile", length) && length >= 3) {
		if (argc != 3) {
			argv[1] = "isfile";
			goto not3Args;
		}
		statOp = 1;
	} else if (c == 'l' && !_strncmp(argv[1], "lstat", length)) {
		if (argc != 4) {
			Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " lstat name varName\"", (char *)NULL);
			return TCL_ERROR;
		}
		if (__stat(fileName, &statBuf) == -1) {
			Tcl_AppendResult(interp, "couldn't lstat \"", argv[2], "\": ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		return StoreStatData(interp, argv[3], &statBuf);
	} else if (c == 'm' && !_strncmp(argv[1], "mtime", length)) {
		if (argc != 3) {
			argv[1] = "mtime";
			goto not3Args;
		}
		if (__stat(fileName, &statBuf) == -1) {
			goto badStat;
		}
		_sprintf(interp->result, "%ld", statBuf.st_mtime);
		return TCL_OK;
	} else if (c == 'o' && !_strncmp(argv[1], "owned", length)) {
		if (argc != 3) {
			argv[1] = "owned";
			goto not3Args;
		}
		statOp = 0;
#ifdef S_IFLNK
		// This option is only included if symbolic links exist on this system (in which case S_IFLNK should be defined).
	} else if (c == 'r' && !_strncmp(argv[1], "readlink", length) && length >= 5) {
		if (argc != 3) {
			argv[1] = "readlink";
			goto not3Args;
		}
		char linkValue[MAXPATHLEN+1];
		int linkLength = readlink(fileName, linkValue, sizeof(linkValue) - 1);
		if (linkLength == -1) {
			Tcl_AppendResult(interp, "couldn't readlink \"", argv[2], "\": ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		linkValue[linkLength] = 0;
		Tcl_SetResult(interp, linkValue, TCL_VOLATILE);
		return TCL_OK;
#endif
	} else if (c == 's' && !_strncmp(argv[1], "size", length) && length >= 2) {
		if (argc != 3) {
			argv[1] = "size";
			goto not3Args;
		}
		if (__stat(fileName, &statBuf) == -1) {
			goto badStat;
		}
		_sprintf(interp->result, "%ld", statBuf.st_size);
		return TCL_OK;
	} else if (c == 's' && !_strncmp(argv[1], "stat", length) && length >= 2) {
		if (argc != 4) {
			Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " stat name varName\"", (char *)NULL);
			return TCL_ERROR;
		}
		if (__stat(fileName, &statBuf) == -1) {
badStat:
			Tcl_AppendResult(interp, "couldn't stat \"", argv[2], "\": ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		return StoreStatData(interp, argv[3], &statBuf);
	} else if (c == 't' && !_strncmp(argv[1], "type", length) && length >= 2) {
		if (argc != 3) {
			argv[1] = "type";
			goto not3Args;
		}
		if (stat(fileName, &statBuf) == -1) {
			goto badStat;
		}
		interp->result = GetFileType((int)statBuf.st_mode);
		return TCL_OK;
	} else {
		Tcl_AppendResult(interp, "bad option \"", argv[1], "\": should be atime, dirname, executable, exists, ", "extension, isdirectory, isfile, lstat, mtime, owned, ", "readable, ",
#ifdef S_IFLNK
			"readlink, ",
#endif
			"root, size, stat, tail, type, ", "or writable", (char *)NULL);
		return TCL_ERROR;
	}
	if (__stat(fileName, &statBuf) == -1) {
		interp->result = "0";
		return TCL_OK;
	}
	switch (statOp) {
	case 0:
		//	mode = (geteuid() == statBuf.st_uid);
		break;
	case 1:
		mode = S_ISREG(statBuf.st_mode);
		break;
	case 2:
		mode = S_ISDIR(statBuf.st_mode);
		break;
	}
	if (mode) {
		interp->result = "1";
	} else {
		interp->result = "0";
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* StoreStatData --
*	This is a utility procedure that breaks out the fields of a "stat" structure and stores them in textual form into the elements of an associative array.
*
* Results:
*	Returns a standard Tcl return value.  If an error occurs then a message is left in interp->result.
*
* Side effects:
*	Elements of the associative array given by "varName" are modified.
*
*----------------------------------------------------------------------
*/
__device__ static int StoreStatData(Tcl_Interp *interp, char *varName, struct stat *statPtr)
{
	char string[30];
	_sprintf(string, "%d", (int)statPtr->st_dev);
	if (Tcl_SetVar2(interp, varName, "dev", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%d", (int)statPtr->st_ino);
	if (Tcl_SetVar2(interp, varName, "ino", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%d", statPtr->st_mode);
	if (Tcl_SetVar2(interp, varName, "mode", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%d", statPtr->st_nlink);
	if (Tcl_SetVar2(interp, varName, "nlink", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%d", statPtr->st_uid);
	if (Tcl_SetVar2(interp, varName, "uid", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%d", statPtr->st_gid);
	if (Tcl_SetVar2(interp, varName, "gid", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%ld", statPtr->st_size);
	if (Tcl_SetVar2(interp, varName, "size", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%ld", statPtr->st_atime);
	if (Tcl_SetVar2(interp, varName, "atime", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%ld", statPtr->st_mtime);
	if (Tcl_SetVar2(interp, varName, "mtime", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	_sprintf(string, "%ld", statPtr->st_ctime);
	if (Tcl_SetVar2(interp, varName, "ctime", string, TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	if (Tcl_SetVar2(interp, varName, "type", GetFileType((int)statPtr->st_mode), TCL_LEAVE_ERR_MSG) == NULL) {
		return TCL_ERROR;
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* GetFileType --
*	Given a mode word, returns a string identifying the type of a file.
*
* Results:
*	A static text string giving the file type from mode.
*
* Side effects:
*	None.
*
*----------------------------------------------------------------------
*/
__device__ static char *GetFileType(int mode)
{
	if (S_ISREG(mode)) return "file";
	else if (S_ISDIR(mode)) return "directory";
	//else if (S_ISCHR(mode)) return "characterSpecial";
	//else if (S_ISBLK(mode)) return "blockSpecial";
	//else if (S_ISFIFO(mode)) return "fifo";
	//else if (S_ISLNK(mode)) return "link";
	//else if (S_ISSOCK(mode)) return "socket";
	return "unknown";
}

/*
*----------------------------------------------------------------------
*
* Tcl_FlushCmd --
*	This procedure is invoked to process the "flush" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_FlushCmd(ClientData notUsed, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId\"", (char *)NULL);
		return TCL_ERROR;
	}
	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[1], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	if (!filePtr->writable) {
		Tcl_AppendResult(interp, "\"", argv[1], "\" wasn't opened for writing", (char *)NULL);
		return TCL_ERROR;
	}
	FILE *f = filePtr->f2;
	if (f == NULL) {
		f = filePtr->f;
	}
	if (_fflush(f) == EOF) {
		Tcl_AppendResult(interp, "error flushing \"", argv[1], "\": ", Tcl_OSError(interp), (char *)NULL);
		clearerr(f);
		return TCL_ERROR;
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_GetsCmd --
*	This procedure is invoked to process the "gets" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_GetsCmd(ClientData notUsed, Tcl_Interp *interp, int argc, char **argv)
{
#define BUF_SIZE 200
	if (argc != 2 && argc != 3) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId ?varName?\"", (char *)NULL);
		return TCL_ERROR;
	}
	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[1], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	if (!filePtr->readable) {
		Tcl_AppendResult(interp, "\"", argv[1], "\" wasn't opened for reading", (char *)NULL);
		return TCL_ERROR;
	}

	// We can't predict how large a line will be, so read it in pieces, appending to the current result or to a variable.
	int totalCount = 0;
	bool done = false;
	int flags = 0;
	register FILE *f = filePtr->f;
	while (!done) {
		char buffer[BUF_SIZE+1];
		register int c, count;
		register char *p;
		for (p = buffer, count = 0; count < BUF_SIZE-1; count++, p++) {
			c = _getc(f);
			if (c == EOF) {
				if (_ferror(filePtr->f)) {
					Tcl_ResetResult(interp);
					Tcl_AppendResult(interp, "error reading \"", argv[1], "\": ", Tcl_OSError(interp), (char *)NULL);
					clearerr(filePtr->f);
					return TCL_ERROR;
				} else if (_feof(filePtr->f)) {
					if (totalCount == 0 && count == 0) {
						totalCount = -1;
					}
					done = 1;
					break;
				}
			}
			if (c == '\n') {
				done = 1;
				break;
			}
			*p = c;
		}
		*p = 0;
		if (argc == 2) {
			Tcl_AppendResult(interp, buffer, (char *)NULL);
		} else {
			if (Tcl_SetVar(interp, argv[2], buffer, flags|TCL_LEAVE_ERR_MSG) == NULL) {
				return TCL_ERROR;
			}
			flags = TCL_APPEND_VALUE;
		}
		totalCount += count;
	}
	if (argc == 3) {
		_sprintf(interp->result, "%d", totalCount);
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_OpenCmd --
*	This procedure is invoked to process the "open" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_OpenCmd(ClientData notUsed, Tcl_Interp *interp, int argc, char **argv)
{
	Interp *iPtr = (Interp *)interp;
	char *access;
	if (argc == 2) {
		access = "r";
	} else if (argc == 3) {
		access = argv[2];
	} else {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " filename ?access?\"", (char *)NULL);
		return TCL_ERROR;
	}

	register OpenFile_ *filePtr = (OpenFile_ *)_allocFast(sizeof(OpenFile_));
	filePtr->f = NULL;
	filePtr->f2 = NULL;
	filePtr->readable = 0;
	filePtr->writable = 0;
	filePtr->numPids = 0;
	filePtr->pidPtr = NULL;
	filePtr->errorId = NULL;

	// Verify the requested form of access.
	int pipeline = 0;
	if (argv[1][0] == '|') {
#ifndef NO_FORK
		pipeline = 1;
#else
		Tcl_AppendResult(interp, "open with pipeline not supported in this version of Tcl", (char *)NULL);
		return TCL_ERROR;
#endif
	}
	switch (access[0]) {
	case 'r':
		filePtr->readable = 1;
		break;
	case 'w':
		filePtr->writable = 1;
		break;
	case 'a':
		filePtr->writable = 1;
		break;
	default:
badAccess:
		Tcl_AppendResult(interp, "illegal access mode \"", access, "\"", (char *)NULL);
		goto error;
	}
	if (access[1] == '+') {
		filePtr->readable = filePtr->writable = 1;
		if (access[2] != 0) {
			goto badAccess;
		}
	} else if (access[1] != 0) {
		goto badAccess;
	}

	// Before we open any files, make sure the file table is allocated so that stdin, etc. are sorted out
	TclMakeFileTable(iPtr, 0);

	// Open the file or create a process pipeline.
	if (!pipeline) {
		char *fileName = argv[1];
		if (fileName[0] == '~') {
			fileName = Tcl_TildeSubst(interp, fileName);
			if (fileName == NULL) {
				goto error;
			}
		}
		filePtr->f = fopen(fileName, access);
		if (filePtr->f == NULL) {
			Tcl_AppendResult(interp, "couldn't open \"", argv[1], "\": ", Tcl_OSError(interp), (char *)NULL);
			goto error;
		}
#ifdef DEBUG_FDS
		syslog(LOG_INFO, "Opened %s to give fd %d", fileName, fileno(filePtr->f));
#endif
	}
	else {
		int cmdArgc;
		char **cmdArgv;
		if (Tcl_SplitList(interp, argv[1]+1, &cmdArgc, &cmdArgv) != TCL_OK) {
			goto error;
		}
		int *inPipePtr = (filePtr->writable ? &inPipe : NULL);
		int *outPipePtr = (filePtr->readable ? &outPipe : NULL);
		int inPipe = -1, int outPipe = -1;
		filePtr->numPids = Tcl_CreatePipeline(interp, cmdArgc, cmdArgv, &filePtr->pidPtr, inPipePtr, outPipePtr, &filePtr->errorId);
		_freeFast((char *)cmdArgv);
		if (filePtr->numPids < 0) {
			goto error;
		}
		//if (filePtr->readable) {
		//	if (outPipe == -1) {
		//		if (inPipe != -1) {
		//			_close(inPipe);
		//		}
		//		Tcl_AppendResult(interp, "can't read output from command:", " standard output was redirected", (char *)NULL);
		//		goto error;
		//	}
		//	filePtr->f = _fdopen(outPipe, "r");
		//}
		//if (filePtr->writable) {
		//	if (inPipe == -1) {
		//		Tcl_AppendResult(interp, "can't write input to command:", " standard input was redirected", (char *)NULL);
		//		goto error;
		//	}
		//	if (filePtr->f != NULL) {
		//		filePtr->f2 = _fdopen(inPipe, "w");
		//	} else {
		//		filePtr->f = _fdopen(inPipe, "w");
		//	}
		//}
	}

	// Enter this new OpenFile_ structure in the table for the interpreter.  May have to expand the table to do this.
	int fd = _fileno(filePtr->f);
	TclMakeFileTable(iPtr, fd);
	if (iPtr->filePtrArray[fd] != NULL) {
		_panic("Tcl_OpenCmd found file already open");
	}
	iPtr->filePtrArray[fd] = filePtr;
	_sprintf(interp->result, "file%d", fd);
	return TCL_OK;

error:
	if (filePtr->f != NULL) {
		_fclose(filePtr->f);
	}
	if (filePtr->f2 != NULL) {
		_fclose(filePtr->f2);
	}
#ifndef NO_FORK
	if (filePtr->numPids > 0) {
		Tcl_DetachPids(filePtr->numPids, filePtr->pidPtr);
		_freeFast((char *)filePtr->pidPtr);
	}
#endif
	if (filePtr->errorId) {
		fclose(filePtr->errorId);
	}
	_freeFast((char *)filePtr);
	return TCL_ERROR;
}

/*
*----------------------------------------------------------------------
*
* Tcl_PwdCmd --
*	This procedure is invoked to process the "pwd" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
#define MAXPATHLEN 1024
__device__ int Tcl_PwdCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 1) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], "\"", (char *)NULL);
		return TCL_ERROR;
	}
	if (currentDir == NULL) {
		char buffer[MAXPATHLEN+1];
		if (__getcwd(buffer, MAXPATHLEN) == NULL) {
			Tcl_AppendResult(interp, "error getting working directory name: ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		currentDir = (char *)_allocFast((unsigned)(strlen(buffer) + 1));
		_strcpy(currentDir, buffer);
	}
	interp->result = currentDir;
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_PutsCmd --
*	This procedure is invoked to process the "puts" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_PutsCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	int i = 1;
	bool newline = true;
	if (argc >= 2 && !_strcmp(argv[1], "-nonewline")) {
		newline = false;
		i++;
	}
	if (i < (argc-3) || i >= argc) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], "\" ?-nonewline? ?fileId? string", (char *)NULL);
		return TCL_ERROR;
	}

	// The code below provides backwards compatibility with an old form of the command that is no longer recommended or documented.
	if (i == (argc-3)) {
		if (_strncmp(argv[i+2], "nonewline", _strlen(argv[i+2]))) {
			Tcl_AppendResult(interp, "bad argument \"", argv[i+2], "\": should be \"nonewline\"", (char *)NULL);
			return TCL_ERROR;
		}
		newline = 0;
	}
	char *fileId;
	if (i == (argc-1)) {
		fileId = "stdout";
	} else {
		fileId = argv[i];
		i++;
	}

	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, fileId, &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	if (!filePtr->writable) {
		Tcl_AppendResult(interp, "\"", fileId, "\" wasn't opened for writing", (char *)NULL);
		return TCL_ERROR;
	}
	FILE *f = filePtr->f2;
	if (f == NULL) {
		f = filePtr->f;
	}

	fputs(argv[i], f);
	if (newline) {
		fputc('\n', f);
	}
	if (ferror(f)) {
		Tcl_AppendResult(interp, "error writing \"", fileId, "\": ", Tcl_OSError(interp), (char *)NULL);
		clearerr(f);
		return TCL_ERROR;
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_ReadCmd --
*	This procedure is invoked to process the "read" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_ReadCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
#define READ_BUF_SIZE 4096
	if (argc != 2 && argc != 3) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId ?numBytes?\" or \"", argv[0], " ?-nonewline? fileId\"", (char *)NULL);
		return TCL_ERROR;
	}
	int i = 1;
	bool newline = true;
	if (argc == 3 && !_strcmp(argv[1], "-nonewline")) {
		newline = false;
		i++;
	}

	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[i], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	if (!filePtr->readable) {
		Tcl_AppendResult(interp, "\"", argv[i], "\" wasn't opened for reading", (char *)NULL);
		return TCL_ERROR;
	}

	// Compute how many bytes to read, and see whether the final newline should be dropped.
	int bytesLeft;
	if (argc >= (i + 2) && _isdigit(argv[i+1][0])) {
		if (Tcl_GetInt(interp, argv[i+1], &bytesLeft) != TCL_OK) {
			return TCL_ERROR;
		}
	} else {
		bytesLeft = 1<<30;

		// The code below provides backward compatibility for an archaic earlier version of this command.
		if (argc >= (i + 2)) {
			if (!_strncmp(argv[i+1], "nonewline", _strlen(argv[i+1]))) {
				newline = false;
			} else {
				Tcl_AppendResult(interp, "bad argument \"", argv[i+1], "\": should be \"nonewline\"", (char *)NULL);
				return TCL_ERROR;
			}
		}
	}

	// Read the file in one or more chunks.
	int bytesRead = 0;
	while (bytesLeft > 0) {
		int count = READ_BUF_SIZE;
		if (bytesLeft < READ_BUF_SIZE) {
			count = bytesLeft;
		}
		char buffer[READ_BUF_SIZE+1];
		count = fread(buffer, 1, count, filePtr->f);
		if (ferror(filePtr->f)) {
			Tcl_ResetResult(interp);
			Tcl_AppendResult(interp, "error reading \"", argv[i], "\": ", Tcl_OSError(interp), (char *)NULL);
			clearerr(filePtr->f);
			return TCL_ERROR;
		}
		if (count == 0) {
			break;
		}
		buffer[count] = 0;
		Tcl_AppendResult(interp, buffer, (char *)NULL);
		bytesLeft -= count;
		bytesRead += count;
	}
	if (!newline && bytesRead > 0 && interp->result[bytesRead-1] == '\n') {
		interp->result[bytesRead-1] = 0;
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_SeekCmd --
*	This procedure is invoked to process the "seek" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_SeekCmd(ClientData notUsed, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 3 && argc != 4) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId offset ?origin?\"", (char *)NULL);
		return TCL_ERROR;
	}
	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[1], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	int offset;
	if (Tcl_GetInt(interp, argv[2], &offset) != TCL_OK) {
		return TCL_ERROR;
	}
	int mode = SEEK_SET;
	if (argc == 4) {
		int length = _strlen(argv[3]);
		char c = argv[3][0];
		if (c == 's' && !_strncmp(argv[3], "start", length)) {
			mode = SEEK_SET;
		} else if (c == 'c' && !_strncmp(argv[3], "current", length)) {
			mode = SEEK_CUR;
		} else if (c == 'e' && !_strncmp(argv[3], "end", length)) {
			mode = SEEK_END;
		} else {
			Tcl_AppendResult(interp, "bad origin \"", argv[3], "\": should be start, current, or end", (char *)NULL);
			return TCL_ERROR;
		}
	}
	if (fseek(filePtr->f, (long)offset, mode) == -1) {
		Tcl_AppendResult(interp, "error during seek: ", Tcl_OSError(interp), (char *)NULL);
		clearerr(filePtr->f);
		return TCL_ERROR;
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_SourceCmd --
*	This procedure is invoked to process the "source" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_SourceCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileName\"", (char *)NULL);
		return TCL_ERROR;
	}
	return Tcl_EvalFile(interp, argv[1]);
}

/*
*----------------------------------------------------------------------
*
* Tcl_TellCmd --
*	This procedure is invoked to process the "tell" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_TellCmd(ClientData notUsed, Tcl_Interp *interp, int argc, char **argv)
{
	if (argc != 2) {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " fileId\"", (char *)NULL);
		return TCL_ERROR;
	}
	OpenFile_ *filePtr;
	if (TclGetOpenFile(interp, argv[1], &filePtr) != TCL_OK) {
		return TCL_ERROR;
	}
	_sprintf(interp->result, "%ld", ftell(filePtr->f));
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_TimeCmd --
*	This procedure is invoked to process the "time" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_TimeCmd(ClientData dummy, Tcl_Interp *interp, int argc, char **argv)
{
	int count;
	if (argc == 2) {
		count = 1;
	} else if (argc == 3) {
		if (Tcl_GetInt(interp, argv[2], &count) != TCL_OK) {
			return TCL_ERROR;
		}
	} else {
		Tcl_AppendResult(interp, "wrong # args: should be \"", argv[0], " command ?count?\"", (char *)NULL);
		return TCL_ERROR;
	}
	double timePer;
	clock_t start = clock();
	for (int i = count; i > 0; i--) {
		int result = Tcl_Eval(interp, argv[1], 0, (char **)NULL);
		if (result != TCL_OK) {
			if (result == TCL_ERROR) {
				char msg[60];
				_sprintf(msg, "\n    (\"time\" body line %d)", interp->errorLine);
				Tcl_AddErrorInfo(interp, msg);
			}
			return result;
		}
	}
	clock_t stop = clock();
	timePer = (((double)(stop - start))*1000000.0)/CLOCKS_PER_SEC;
	Tcl_ResetResult(interp);
	_sprintf(interp->result, "%.0f microseconds per iteration", timePer/count);
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* CleanupChildren --
*	This is a utility procedure used to wait for child processes to exit, record information about abnormal exits, and then
*	collect any stderr output generated by them.
*
* Results:
*	The return value is a standard Tcl result.  If anything at weird happened with the child processes, TCL_ERROR is returned
*	and a message is left in interp->result.
*
* Side effects:
*	If the last character of interp->result is a newline, then it is removed.  File errorId gets closed, and pidPtr is freed
*	back to the storage allocator.
*
*----------------------------------------------------------------------
*/
__device__ static int CleanupChildren(Tcl_Interp *interp, int numPids, int *pidPtr, FILE *errorId)
{
	//	int result = TCL_OK;
	//	int i, pid;
	//#define WAIT_STATUS_TYPE int
	//	WAIT_STATUS_TYPE waitStatus;
	//	for (i = 0; i < numPids; i++) {
	//		pid = Tcl_WaitPids(1, &pidPtr[i], (int *) &waitStatus);
	//		if (pid == -1) {
	//			// This can happen if the process was already reaped, so just ignore it
	//#if 0
	//			Tcl_AppendResult(interp, "error waiting for process to exit: ", Tcl_OSError(interp), (char *)NULL);
	//#endif
	//			continue;
	//		}
	//
	//		// Create error messages for unusual process exits.  An extra newline gets appended to each error message, but
	//		// it gets removed below (in the same fashion that an extra newline in the command's output is removed).
	//		if (!WIFEXITED(waitStatus) || WEXITSTATUS(waitStatus) != 0) {
	//			char msg1[20], msg2[20];
	//			result = TCL_ERROR;
	//			sprintf(msg1, "%d", pid);
	//			if (WIFEXITED(waitStatus)) {
	//				sprintf(msg2, "%d", WEXITSTATUS(waitStatus));
	//				Tcl_SetErrorCode(interp, "CHILDSTATUS", msg1, msg2, (char *)NULL);
	//			} else if (WIFSIGNALED(waitStatus)) {
	//				char *p;
	//				p = Tcl_SignalMsg((int) (WTERMSIG(waitStatus)));
	//				Tcl_SetErrorCode(interp, "CHILDKILLED", msg1, Tcl_SignalId((int)(WTERMSIG(waitStatus))), p, (char *)NULL);
	//				Tcl_AppendResult(interp, "child killed: ", p, "\n", (char *)NULL);
	//			} else if (WIFSTOPPED(waitStatus)) {
	//				char *p;
	//				p = Tcl_SignalMsg((int) (WSTOPSIG(waitStatus)));
	//				Tcl_SetErrorCode(interp, "CHILDSUSP", msg1, Tcl_SignalId((int)(WSTOPSIG(waitStatus))), p, (char *)NULL);
	//				Tcl_AppendResult(interp, "child suspended: ", p, "\n", (char *)NULL);
	//			} else {
	//				Tcl_AppendResult(interp, "child wait status didn't make sense\n", (char *)NULL);
	//			}
	//		}
	//	}
	//	_freeFast((char *)pidPtr);
	//
	//	// Read the standard error file.  If there's anything there, then return an error and add the file's contents to the result string.
	//	if (errorId >= 0) {
	//		while (true) {
	//#define BUFFER_SIZE 1000
	//			char buffer[BUFFER_SIZE+1];
	//			int count;
	//			count = read(errorId, buffer, BUFFER_SIZE);
	//			if (count == 0) {
	//				break;
	//			}
	//			if (count < 0) {
	//				Tcl_AppendResult(interp, "error reading stderr output file: ", Tcl_OSError(interp), (char *)NULL);
	//				break;
	//			}
	//			buffer[count] = 0;
	//			Tcl_AppendResult(interp, buffer, (char *)NULL);
	//		}
	//		fclose(errorId);
	//	}
	//
	//	// If the last character of interp->result is a newline, then remove the newline character (the newline would just confuse things).
	//	int length = strlen(interp->result);
	//	if (length > 0 && interp->result[length-1] == '\n') {
	//		interp->result[length-1] = '\0';
	//	}
	//	return result;
	return 0;
}

/*
*-----------------------------------------------------------------------------
*
* Tcl_PidCmd --
*     Implements the pid TCL command:
*         pid
*
* Results:
*      Standard TCL result.
*-----------------------------------------------------------------------------
*/
#define GetCurrentProcessId 1
__device__ int Tcl_PidCmd(ClientData clientData, Tcl_Interp *interp, int argc, char **argv)
{
	char buf[10];
	if (argc != 1) {
		Tcl_AppendResult (interp, "bad # args: ", argv[0], (char *)NULL);
		return TCL_ERROR;
	}
	_sprintf(buf, "%d", GetCurrentProcessId);
	Tcl_AppendResult(interp, buf, (char *)NULL);
	return TCL_OK;
}

#endif