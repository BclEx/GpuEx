﻿// status.c
#include "Runtime.h"

__device__ static _WSD struct Status
{
	int NowValue[10]; // Current value
	int MaxValue[10]; // Maximum value
} g_status = { {0,}, {0,} };
#ifndef OMIT_WSD
#define _status_Init
#define _stat g_status
#else
#define _status_Init Status *x = &_GLOBAL(Status, g_status)
#define _stat x[0]
#endif

__device__ int _status_value(STATUS op)
{
	_status_Init;
	_assert(op < _lengthof(_stat.NowValue));
	return _stat.NowValue[op];
}

__device__ void _status_add(STATUS op, int n)
{
	_status_Init;
	_assert(op < _lengthof(_stat.NowValue));
	_stat.NowValue[op] += n;
	if (_stat.NowValue[op] > _stat.MaxValue[op])
		_stat.MaxValue[op] = _stat.NowValue[op];
}

__device__ void _status_set(STATUS op, int x)
{
	_status_Init;
	_assert(op < _lengthof(_stat.NowValue));
	_stat.NowValue[op] = x;
	if (_stat.NowValue[op] > _stat.MaxValue[op])
		_stat.MaxValue[op] = _stat.NowValue[op];
}

__device__ bool _status(STATUS op, int *current, int *highwater, bool resetFlag)
{
	_status_Init;
	if (op >= _lengthof(_stat.NowValue))
		return false;
	*current = _stat.NowValue[op];
	*highwater = _stat.MaxValue[op];
	if (resetFlag)
		_stat.MaxValue[op] = _stat.NowValue[op];
	return true;
}
