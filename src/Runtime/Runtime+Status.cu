﻿// status.c
#include "Runtime.h"

__device__ static _WSD struct Stat
{
	int NowValue[10];	// Current value
	int MaxValue[10];	// Maximum value
} g_stat = { {0,}, {0,} };
#ifndef OMIT_WSD
#define _stat_Init
#define _stat g_stat
#else
#define _stat_Init Stat *x = &_GLOBAL(Stat, g_stat)
#define _stat x[0]
#endif

__device__ int _status_value(STATUS op)
{
	_stat_Init;
	_assert(op < _lengthof(_stat.NowValue));
	return _stat.NowValue[op];
}

__device__ void _status_add(STATUS op, int n)
{
	_stat_Init;
	_assert(op < _lengthof(_stat.NowValue));
	_stat.NowValue[op] += n;
	if (_stat.NowValue[op] > _stat.MaxValue[op])
		_stat.MaxValue[op] = _stat.NowValue[op];
}

__device__ void _status_set(STATUS op, int x)
{
	_stat_Init;
	_assert(op < _lengthof(_stat.NowValue));
	_stat.NowValue[op] = x;
	if (_stat.NowValue[op] > _stat.MaxValue[op])
		_stat.MaxValue[op] = _stat.NowValue[op];
}

__device__ bool _status(STATUS op, int *current, int *highwater, bool resetFlag)
{
	_stat_Init;
	if (op >= _lengthof(_stat.NowValue))
		return false;
	*current = _stat.NowValue[op];
	*highwater = _stat.MaxValue[op];
	if (resetFlag)
		_stat.MaxValue[op] = _stat.NowValue[op];
	return true;
}
