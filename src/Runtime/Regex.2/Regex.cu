#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <setjmp.h>
#include "Runtime.h"
#include "Regex.h"


#ifdef _DEBUG
#include <stdio.h>

static const char_t *g_nnames[] =
{
	_L("NONE"),_L("OP_GREEDY"),	_L("OP_OR"),
	_L("OP_EXPR"),_L("OP_NOCAPEXPR"),_L("OP_DOT"),	_L("OP_CLASS"),
	_L("OP_CCLASS"),_L("OP_NCLASS"),_L("OP_RANGE"),_L("OP_CHAR"),
	_L("OP_EOL"),_L("OP_BOL"),_L("OP_WB")
};

#endif
#define OP_GREEDY		(MAX_CHAR+1) // * + ? {n}
#define OP_OR			(MAX_CHAR+2)
#define OP_EXPR			(MAX_CHAR+3) //parentesis ()
#define OP_NOCAPEXPR	(MAX_CHAR+4) //parentesis (?:)
#define OP_DOT			(MAX_CHAR+5)
#define OP_CLASS		(MAX_CHAR+6)
#define OP_CCLASS		(MAX_CHAR+7)
#define OP_NCLASS		(MAX_CHAR+8) //negates class the [^
#define OP_RANGE		(MAX_CHAR+9)
#define OP_CHAR			(MAX_CHAR+10)
#define OP_EOL			(MAX_CHAR+11)
#define OP_BOL			(MAX_CHAR+12)
#define OP_WB			(MAX_CHAR+13)

#define REG_SYMBOL_ANY_CHAR ('.')
#define REG_SYMBOL_GREEDY_ONE_OR_MORE ('+')
#define REG_SYMBOL_GREEDY_ZERO_OR_MORE ('*')
#define REG_SYMBOL_GREEDY_ZERO_OR_ONE ('?')
#define REG_SYMBOL_BRANCH ('|')
#define REG_SYMBOL_END_OF_STRING ('$')
#define REG_SYMBOL_BEGINNING_OF_STRING ('^')
#define REG_SYMBOL_ESCAPE_CHAR ('\\')


typedef int NodeType;

typedef struct tagNode
{
	NodeType type;
	int left;
	int right;
	int next;
} node_t;

struct regex_t
{
	const char_t *_eol;
	const char_t *_bol;
	const char_t *_p;
	int _first;
	int _op;
	node_t *_nodes;
	int _nallocated;
	int _nsize;
	int _nsubexpr;
	regmatch_t *_matches;
	int _currsubexp;
	void *_jmpbuf;
	const char_t **_error;
};

static int reg_list(regex_t *exp);

static int reg_newnode(regex_t *exp, NodeType type)
{
	node_t n;
	int newid;
	n.type = type;
	n.next = n.right = n.left = -1;
	if(type == OP_EXPR)
		n.right = exp->_nsubexpr++;
	if(exp->_nallocated < (exp->_nsize + 1)) {
		int oldsize = exp->_nallocated;
		exp->_nallocated *= 2;
		exp->_nodes = (node_t *)realloc(exp->_nodes, exp->_nallocated * sizeof(node_t));
	}
	exp->_nodes[exp->_nsize++] = n;
	newid = exp->_nsize - 1;
	return (int)newid;
}

static void reg_error(regex_t *exp,const char_t *error)
{
	if(exp->_error) *exp->_error = error;
	longjmp(*((jmp_buf*)exp->_jmpbuf),-1);
}

static void reg_expect(regex_t *exp, int n){
	if((*exp->_p) != n)
		reg_error(exp, _L("expected paren"));
	exp->_p++;
}

static char_t reg_escapechar(regex_t *exp)
{
	if(*exp->_p == REG_SYMBOL_ESCAPE_CHAR){
		exp->_p++;
		switch(*exp->_p) {
		case 'v': exp->_p++; return '\v';
		case 'n': exp->_p++; return '\n';
		case 't': exp->_p++; return '\t';
		case 'r': exp->_p++; return '\r';
		case 'f': exp->_p++; return '\f';
		default: return (*exp->_p++);
		}
	} else if(!_isprint(*exp->_p)) reg_error(exp,_L("letter expected"));
	return (*exp->_p++);
}

static int reg_charclass(regex_t *exp,int classid)
{
	int n = reg_newnode(exp,OP_CCLASS);
	exp->_nodes[n].left = classid;
	return n;
}

static int reg_charnode(regex_t *exp,bool isclass)
{
	char_t t;
	if(*exp->_p == REG_SYMBOL_ESCAPE_CHAR) {
		exp->_p++;
		switch(*exp->_p) {
		case 'n': exp->_p++; return reg_newnode(exp,'\n');
		case 't': exp->_p++; return reg_newnode(exp,'\t');
		case 'r': exp->_p++; return reg_newnode(exp,'\r');
		case 'f': exp->_p++; return reg_newnode(exp,'\f');
		case 'v': exp->_p++; return reg_newnode(exp,'\v');
		case 'a': case 'A': case 'w': case 'W': case 's': case 'S':
		case 'd': case 'D': case 'x': case 'X': case 'c': case 'C':
		case 'p': case 'P': case 'l': case 'u':
			{
				t = *exp->_p; exp->_p++;
				return reg_charclass(exp,t);
			}
		case 'b':
		case 'B':
			if(!isclass) {
				int node = reg_newnode(exp,OP_WB);
				exp->_nodes[node].left = *exp->_p;
				exp->_p++;
				return node;
			} //else default
		default:
			t = *exp->_p; exp->_p++;
			return reg_newnode(exp,t);
		}
	}
	else if(!_isprint(*exp->_p)) {

		reg_error(exp,_L("letter expected"));
	}
	t = *exp->_p; exp->_p++;
	return reg_newnode(exp,t);
}
static int reg_class(regex_t *exp)
{
	int ret = -1;
	int first = -1,chain;
	if(*exp->_p == REG_SYMBOL_BEGINNING_OF_STRING){
		ret = reg_newnode(exp,OP_NCLASS);
		exp->_p++;
	}else ret = reg_newnode(exp,OP_CLASS);

	if(*exp->_p == ']') reg_error(exp,_L("empty class"));
	chain = ret;
	while(*exp->_p != ']' && exp->_p != exp->_eol) {
		if(*exp->_p == '-' && first != -1){
			int r,t;
			if(*exp->_p++ == ']') reg_error(exp,_L("unfinished range"));
			r = reg_newnode(exp,OP_RANGE);
			if(first>*exp->_p) reg_error(exp,_L("invalid range"));
			if(exp->_nodes[first].type == OP_CCLASS) reg_error(exp,_L("cannot use character classes in ranges"));
			exp->_nodes[r].left = exp->_nodes[first].type;
			t = reg_escapechar(exp);
			exp->_nodes[r].right = t;
			exp->_nodes[chain].next = r;
			chain = r;
			first = -1;
		}
		else{
			if(first!=-1){
				int c = first;
				exp->_nodes[chain].next = c;
				chain = c;
				first = reg_charnode(exp,true);
			}
			else{
				first = reg_charnode(exp,true);
			}
		}
	}
	if(first!=-1){
		int c = first;
		exp->_nodes[chain].next = c;
		chain = c;
		first = -1;
	}
	/* hack? */
	exp->_nodes[ret].left = exp->_nodes[ret].next;
	exp->_nodes[ret].next = -1;
	return ret;
}

static int reg_parsenumber(regex_t *exp)
{
	int ret = *exp->_p-'0';
	int positions = 10;
	exp->_p++;
	while(isdigit(*exp->_p)) {
		ret = ret*10+(*exp->_p++-'0');
		if(positions==1000000000) reg_error(exp,_L("overflow in numeric constant"));
		positions *= 10;
	};
	return ret;
}

static int reg_element(regex_t *exp)
{
	int ret = -1;
	switch(*exp->_p)
	{
	case '(': {
		int expr,newn;
		exp->_p++;


		if(*exp->_p =='?') {
			exp->_p++;
			reg_expect(exp,':');
			expr = reg_newnode(exp,OP_NOCAPEXPR);
		}
		else
			expr = reg_newnode(exp,OP_EXPR);
		newn = reg_list(exp);
		exp->_nodes[expr].left = newn;
		ret = expr;
		reg_expect(exp,')');
			  }
			  break;
	case '[':
		exp->_p++;
		ret = reg_class(exp);
		reg_expect(exp,']');
		break;
	case REG_SYMBOL_END_OF_STRING: exp->_p++; ret = reg_newnode(exp,OP_EOL);break;
	case REG_SYMBOL_ANY_CHAR: exp->_p++; ret = reg_newnode(exp,OP_DOT);break;
	default:
		ret = reg_charnode(exp,false);
		break;
	}

	{
		int op;
		bool isgreedy = false;
		unsigned short p0 = 0, p1 = 0;
		switch(*exp->_p){
		case REG_SYMBOL_GREEDY_ZERO_OR_MORE: p0 = 0; p1 = 0xFFFF; exp->_p++; isgreedy = true; break;
		case REG_SYMBOL_GREEDY_ONE_OR_MORE: p0 = 1; p1 = 0xFFFF; exp->_p++; isgreedy = true; break;
		case REG_SYMBOL_GREEDY_ZERO_OR_ONE: p0 = 0; p1 = 1; exp->_p++; isgreedy = true; break;
		case '{':
			exp->_p++;
			if(!isdigit(*exp->_p)) reg_error(exp,_L("number expected"));
			p0 = (unsigned short)reg_parsenumber(exp);
			/*******************************/
			switch(*exp->_p) {
			case '}':
				p1 = p0; exp->_p++;
				break;
			case ',':
				exp->_p++;
				p1 = 0xFFFF;
				if(isdigit(*exp->_p)){
					p1 = (unsigned short)reg_parsenumber(exp);
				}
				reg_expect(exp,'}');
				break;
			default:
				reg_error(exp,_L(", or } expected"));
			}
			/*******************************/
			isgreedy = true;
			break;

		}
		if(isgreedy) {
			int nnode = reg_newnode(exp,OP_GREEDY);
			op = OP_GREEDY;
			exp->_nodes[nnode].left = ret;
			exp->_nodes[nnode].right = ((p0)<<16)|p1;
			ret = nnode;
		}
	}
	if((*exp->_p != REG_SYMBOL_BRANCH) && (*exp->_p != ')') && (*exp->_p != REG_SYMBOL_GREEDY_ZERO_OR_MORE) && (*exp->_p != REG_SYMBOL_GREEDY_ONE_OR_MORE) && (*exp->_p != '\0')) {
		int nnode = reg_element(exp);
		exp->_nodes[ret].next = nnode;
	}

	return ret;
}

static int reg_list(regex_t *exp)
{
	int ret=-1,e;
	if(*exp->_p == REG_SYMBOL_BEGINNING_OF_STRING) {
		exp->_p++;
		ret = reg_newnode(exp,OP_BOL);
	}
	e = reg_element(exp);
	if(ret != -1) {
		exp->_nodes[ret].next = e;
	}
	else ret = e;

	if(*exp->_p == REG_SYMBOL_BRANCH) {
		int temp,tright;
		exp->_p++;
		temp = reg_newnode(exp,OP_OR);
		exp->_nodes[temp].left = ret;
		tright = reg_list(exp);
		exp->_nodes[temp].right = tright;
		ret = temp;
	}
	return ret;
}

static bool reg_matchcclass(int cclass,char_t c)
{
	switch(cclass) {
	case 'a': return isalpha(c)?true:false;
	case 'A': return !isalpha(c)?true:false;
	case 'w': return (isalnum(c) || c == '_')?true:false;
	case 'W': return (!isalnum(c) && c != '_')?true:false;
	case 's': return isspace(c)?true:false;
	case 'S': return !isspace(c)?true:false;
	case 'd': return isdigit(c)?true:false;
	case 'D': return !isdigit(c)?true:false;
	case 'x': return isxdigit(c)?true:false;
	case 'X': return !isxdigit(c)?true:false;
	case 'c': return iscntrl(c)?true:false;
	case 'C': return !iscntrl(c)?true:false;
	case 'p': return ispunct(c)?true:false;
	case 'P': return !ispunct(c)?true:false;
	case 'l': return islower(c)?true:false;
	case 'u': return isupper(c)?true:false;
	}
	return false; /*cannot happen*/
}

static bool reg_matchclass(regex_t* exp,node_t *node,char_t c)
{
	do {
		switch(node->type) {
		case OP_RANGE:
			if(c >= node->left && c <= node->right) return true;
			break;
		case OP_CCLASS:
			if(reg_matchcclass(node->left,c)) return true;
			break;
		default:
			if(c == node->type)return true;
		}
	} while((node->next != -1) && (node = &exp->_nodes[node->next]));
	return false;
}

static const char_t *reg_matchnode(regex_t* exp,node_t *node,const char_t *str,node_t *next)
{

	NodeType type = node->type;
	switch(type) {
	case OP_GREEDY: {
		//node_t *greedystop = (node->next != -1) ? &exp->_nodes[node->next] : NULL;
		node_t *greedystop = NULL;
		int p0 = (node->right >> 16)&0x0000FFFF, p1 = node->right&0x0000FFFF, nmaches = 0;
		const char_t *s=str, *good = str;

		if(node->next != -1) {
			greedystop = &exp->_nodes[node->next];
		}
		else {
			greedystop = next;
		}

		while((nmaches == 0xFFFF || nmaches < p1)) {

			const char_t *stop;
			if(!(s = reg_matchnode(exp,&exp->_nodes[node->left],s,greedystop)))
				break;
			nmaches++;
			good=s;
			if(greedystop) {
				//checks that 0 matches satisfy the expression(if so skips)
				//if not would always stop(for instance if is a '?')
				if(greedystop->type != OP_GREEDY ||
					(greedystop->type == OP_GREEDY && ((greedystop->right >> 16)&0x0000FFFF) != 0))
				{
					node_t *gnext = NULL;
					if(greedystop->next != -1) {
						gnext = &exp->_nodes[greedystop->next];
					}else if(next && next->next != -1){
						gnext = &exp->_nodes[next->next];
					}
					stop = reg_matchnode(exp,greedystop,s,gnext);
					if(stop) {
						//if satisfied stop it
						if(p0 == p1 && p0 == nmaches) break;
						else if(nmaches >= p0 && p1 == 0xFFFF) break;
						else if(nmaches >= p0 && nmaches <= p1) break;
					}
				}
			}

			if(s >= exp->_eol)
				break;
		}
		if(p0 == p1 && p0 == nmaches) return good;
		else if(nmaches >= p0 && p1 == 0xFFFF) return good;
		else if(nmaches >= p0 && nmaches <= p1) return good;
		return NULL;
					}
	case OP_OR: {
		const char_t *asd = str;
		node_t *temp=&exp->_nodes[node->left];
		while( (asd = reg_matchnode(exp,temp,asd,NULL)) ) {
			if(temp->next != -1)
				temp = &exp->_nodes[temp->next];
			else
				return asd;
		}
		asd = str;
		temp = &exp->_nodes[node->right];
		while( (asd = reg_matchnode(exp,temp,asd,NULL)) ) {
			if(temp->next != -1)
				temp = &exp->_nodes[temp->next];
			else
				return asd;
		}
		return NULL;
		break;
				}
	case OP_EXPR:
	case OP_NOCAPEXPR:{
		node_t *n = &exp->_nodes[node->left];
		const char_t *cur = str;
		int capture = -1;
		if(node->type != OP_NOCAPEXPR && node->right == exp->_currsubexp) {
			capture = exp->_currsubexp;
			exp->_matches[capture].begin = cur;
			exp->_currsubexp++;
		}

		do {
			node_t *subnext = NULL;
			if(n->next != -1) {
				subnext = &exp->_nodes[n->next];
			}else {
				subnext = next;
			}
			if(!(cur = reg_matchnode(exp,n,cur,subnext))) {
				if(capture != -1){
					exp->_matches[capture].begin = 0;
					exp->_matches[capture].len = 0;
				}
				return NULL;
			}
		} while((n->next != -1) && (n = &exp->_nodes[n->next]));

		if(capture != -1)
			exp->_matches[capture].len = cur - exp->_matches[capture].begin;
		return cur;
					  }
	case OP_WB:
		if(str == exp->_bol && !isspace(*str)
			|| (str == exp->_eol && !isspace(*(str-1)))
			|| (!isspace(*str) && isspace(*(str+1)))
			|| (isspace(*str) && !isspace(*(str+1))) ) {
				return (node->left == 'b')?str:NULL;
		}
		return (node->left == 'b')?NULL:str;
	case OP_BOL:
		if(str == exp->_bol) return str;
		return NULL;
	case OP_EOL:
		if(str == exp->_eol) return str;
		return NULL;
	case OP_DOT:{
		*str++;
				}
				return str;
	case OP_NCLASS:
	case OP_CLASS:
		if(reg_matchclass(exp,&exp->_nodes[node->left],*str)?(type == OP_CLASS?true:false):(type == OP_NCLASS?true:false)) {
			*str++;
			return str;
		}
		return NULL;
	case OP_CCLASS:
		if(reg_matchcclass(node->left,*str)) {
			*str++;
			return str;
		}
		return NULL;
	default: /* char */
		if(*str != node->type) return NULL;
		*str++;
		return str;
	}
	return NULL;
}

/* public api */
regex_t *reg_compile(const char_t *pattern,const char_t **error)
{
	regex_t *exp = (regex_t *)malloc(sizeof(regex_t));
	exp->_eol = exp->_bol = NULL;
	exp->_p = pattern;
	exp->_nallocated = (int)_strlen30(pattern) * sizeof(char_t);
	exp->_nodes = (node_t *)malloc(exp->_nallocated * sizeof(node_t));
	exp->_nsize = 0;
	exp->_matches = 0;
	exp->_nsubexpr = 0;
	exp->_first = reg_newnode(exp,OP_EXPR);
	exp->_error = error;
	exp->_jmpbuf = malloc(sizeof(jmp_buf));
	if(setjmp(*((jmp_buf*)exp->_jmpbuf)) == 0) {
		int res = reg_list(exp);
		exp->_nodes[exp->_first].left = res;
		if(*exp->_p!='\0')
			reg_error(exp,_L("unexpected character"));
#ifdef _DEBUG
		{
			int nsize,i;
			node_t *t;
			nsize = exp->_nsize;
			t = &exp->_nodes[0];
			_printf(_L("\n"));
			for(i = 0;i < nsize; i++) {
				if(exp->_nodes[i].type>MAX_CHAR)
					_printf(_L("[%02d] %10s "),i,g_nnames[exp->_nodes[i].type-MAX_CHAR]);
				else
					_printf(_L("[%02d] %10c "),i,exp->_nodes[i].type);
				_printf(_L("left %02d right %02d next %02d\n"),exp->_nodes[i].left,exp->_nodes[i].right,exp->_nodes[i].next);
			}
			_printf(_L("\n"));
		}
#endif
		exp->_matches = (regmatch_t *) malloc(exp->_nsubexpr * sizeof(regmatch_t));
		memset(exp->_matches,0,exp->_nsubexpr * sizeof(regmatch_t));
	}
	else{
		reg_free(exp);
		return NULL;
	}
	return exp;
}

void reg_free(regex_t *exp)
{
	if(exp)	{
		if(exp->_nodes) free(exp->_nodes);
		if(exp->_jmpbuf) free(exp->_jmpbuf);
		if(exp->_matches) free(exp->_matches);
		free(exp);
	}
}

bool reg_match(regex_t* exp,const char_t* text)
{
	const char_t* res = NULL;
	exp->_bol = text;
	exp->_eol = text + _strlen30(text);
	exp->_currsubexp = 0;
	res = reg_matchnode(exp,exp->_nodes,text,NULL);

#ifdef _DEBUG
	_printf("DEBUG reg_match: res = '%s'\n", res);
	_printf("DEBUG reg_match: exp->_eol = '%s'\n", exp->_eol);
#endif

	// Fail match if reg_matchnode returns nothing
	if (!res) {
		return false;
	}

	return true;
}

bool reg_searchrange(regex_t* exp,const char_t* text_begin,const char_t* text_end,const char_t** out_begin, const char_t** out_end)
{
	const char_t *cur = NULL;
	int node = exp->_first;
	if(text_begin >= text_end) return false;
	exp->_bol = text_begin;
	exp->_eol = text_end;
	do {
		cur = text_begin;
		while(node != -1) {
			exp->_currsubexp = 0;
			cur = reg_matchnode(exp,&exp->_nodes[node],cur,NULL);
			if(!cur)
				break;
			node = exp->_nodes[node].next;
		}
		*text_begin++;
	} while(cur == NULL && text_begin != text_end);

	if(cur == NULL)
		return false;

	--text_begin;

	if(out_begin) *out_begin = text_begin;
	if(out_end) *out_end = cur;
	return true;
}

bool reg_search(regex_t* exp,const char_t* text, const char_t** out_begin, const char_t** out_end)
{
	return reg_searchrange(exp,text,text + _strlen30(text),out_begin,out_end);
}

int reg_getsubexpcount(regex_t* exp)
{
	return exp->_nsubexpr;
}

bool reg_getsubexp(regex_t* exp, int n, regmatch_t *subexp)
{
	if( n<0 || n >= exp->_nsubexpr) return false;
	*subexp = exp->_matches[n];
	return true;
}

