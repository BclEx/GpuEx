#include <windows.h>
#include <process.h>
#include <stdio.h>
#include "RuntimeSentinel.h"

#if OS_WIN
#pragma region OS_WIN

__device__ void RuntimeSentinel::Send(void *msg, int msgLength)
{
	RuntimeSentinelMap *map = GetMap();
	int id = InterlockedAdd((LONG *)&map->AddId, 1);
	RuntimeSentinelCommand *cmd = &map->Commands[(id-1)%_lengthof(map->Commands)];
	while (InterlockedCompareExchange((LONG *)&cmd->Status, 1, 0) != 0) { _sleep(10); }
	cmd->Length = msgLength;
	RuntimeSentinelMessage *msg2 = (RuntimeSentinelMessage *)msg;
	if (msg2->Prepare)
		msg2->Prepare(msg, cmd->Data, sizeof(cmd->Data));
	memcpy(cmd->Data, msg, msgLength);
	cmd->Status = 2;
	while (InterlockedCompareExchange((LONG *)&cmd->Status, 5, 4) != 4) { _sleep(10); }
	memcpy(msg, cmd->Data, msgLength);
	cmd->Status = 0;
}

#pragma endregion
#endif