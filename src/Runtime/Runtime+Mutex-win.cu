#define WIN32_LEAN_AND_MEAN
#include "Runtime.h"
#include <Windows.h>

#ifdef MUTEX_WIN
#pragma region MUTEX_WIN

struct _mutex_obj
{
	CRITICAL_SECTION Mutex;		// Mutex controlling the lock
	MUTEX Id;						// Mutex type
#ifdef _DEBUG
	volatile int Refs;			// Number of enterances
	volatile DWORD Owner;		// Thread holding this mutex
	bool Trace;					// True to trace changes
#endif
};
#ifdef _DEBUG
#define MUTEX_INIT { 0, 0, 0L, (DWORD)0, 0 }
#else
#define MUTEX_INIT { 0, 0 }
#endif
static _mutex_obj g_mutex_Statics[6] = { MUTEX_INIT, MUTEX_INIT, MUTEX_INIT, MUTEX_INIT, MUTEX_INIT, MUTEX_INIT };
#undef MUTEX_INIT
static bool g_mutex_IsInit = false;
static long g_mutex_Lock = 0;

#ifdef _DEBUG
bool _mutex_held(MutexEx p) { return (!p || (p->Refs != 0 && p->Owner == GetCurrentThreadId())); }
bool _mutex_notheld(MutexEx p, DWORD tid) { return (!p || p->Refs == 0 || p->Owner != tid); }
bool _mutex_notheld(MutexEx p) { DWORD tid = GetCurrentThreadId(); return (!p || _mutex_notheld(p, tid)); }
#endif

int _mutex_init()
{ 
	// The first to increment to 1 does actual initialization
	if (InterlockedCompareExchange(&g_mutex_Lock, 1, 0) == 0)
	{
		for (int i = 0; i < _lengthof(g_mutex_Statics); i++)
		{
#if OS_WINRT
			InitializeCriticalSectionEx(&g_mutex_Statics[i].Mutex, 0, 0);
#else
			InitializeCriticalSection(&g_mutex_Statics[i].Mutex);
#endif
		}
		g_mutex_IsInit = true;
	}
	else while (!g_mutex_IsInit) // Someone else is in the process of initing the static mutexes
		Sleep(1);
	return 0; 
}

void _mutex_shutdown()
{
	// The first to decrement to 0 does actual shutdown (which should be the last to shutdown.)
	if (InterlockedCompareExchange(&g_mutex_Lock, 0, 1) == 1)
	{
		if (g_mutex_IsInit)
		{
			for (int i =0 ; i < _lengthof(g_mutex_Statics); i++)
				DeleteCriticalSection(&g_mutex_Statics[i].Mutex);
			g_mutex_IsInit = false;
		}
	}
}

MutexEx _mutex_alloc(MUTEX id)
{
	_mutex_obj *p;
	switch (id)
	{
	case MUTEX_FAST:
	case MUTEX_RECURSIVE: {
		p = (_mutex_obj *)_allocZero(sizeof(*p));
		if (p)
		{  
#ifdef _DEBUG
			p->Id = id;
#endif
#if OS_WINRT
			InitializeCriticalSectionEx(&p->Mutex, 0, 0);
#else
			InitializeCriticalSection(&p->Mutex);
#endif
		}
		break; }
	default: {
		_assert(g_mutex_IsInit);
		_assert(id-2 >= 0);
		_assert(id-2 < _lengthof(g_mutex_Statics));
		p = &g_mutex_Statics[id-2];
#ifdef _DEBUG
		p->Id = id;
#endif
		break; }
	}
	return p;
}

void _mutex_free(MutexEx p)
{
	if (!p) return;
	_assert(p);
	_assert(p->Refs == 0 && p->Owner == 0);
	_assert(p->Id == MUTEX_FAST || p->Id == MUTEX_RECURSIVE);
	DeleteCriticalSection(&p->Mutex);
	_free(p);
}

void _mutex_enter(MutexEx p)
{
	if (!p) return;
#ifdef _DEBUG
	DWORD tid = GetCurrentThreadId(); 
	_assert(p->Id == MUTEX_RECURSIVE || _mutex_notheld(p, tid));
#endif
	EnterCriticalSection(&p->Mutex);
#ifdef _DEBUG
	_assert(p->Refs > 0 || p->Owner == 0);
	p->Owner = tid; 
	p->Refs++;
	if (p->Trace)
		printf("enter mutex %p (%d) with nRef=%d\n", p, p->Trace, p->Refs);
#endif
}

bool _mutex_tryenter(MutexEx p)
{
	if (!p) return true;
#ifndef NDEBUG
	DWORD tid = GetCurrentThreadId(); 
#endif
	bool rc = false;
	_assert(p->Id == MUTEX_RECURSIVE || _mutex_notheld(p, tid));
	// The sqlite3_mutex_try() routine is very rarely used, and when it is used it is merely an optimization.  So it is OK for it to always fail.  
	//
	// The TryEnterCriticalSection() interface is only available on WinNT. And some windows compilers complain if you try to use it without
	// first doing some #defines that prevent SQLite from building on Win98. For that reason, we will omit this optimization for now.  See ticket #2685.
	if (TryEnterCriticalSection(&p->Mutex))
	{
		p->Owner = tid;
		p->Refs++;
		rc = true;
	}
#ifdef _DEBUG
	if (rc && p->Trace)
		printf("try mutex %p (%d) with nRef=%d\n", p, p->Trace, p->Refs);
#endif
	return rc;
}

void _mutex_leave(MutexEx p)
{
	if (!p) return;
#ifndef NDEBUG
	DWORD tid = GetCurrentThreadId();
	_assert(p->Refs > 0);
	_assert(p->Owner == tid);
	p->Refs--;
	if (p->Refs == 0) p->Owner = 0;
	_assert(p->Refs == 0 || p->Id == MUTEX_RECURSIVE);
#endif
	LeaveCriticalSection(&p->Mutex);
#ifdef _DEBUG
	if (p->Trace)
		printf("leave mutex %p (%d) with nRef=%d\n", p, p->Trace, p->Refs);
#endif
}

#pragma endregion
#endif