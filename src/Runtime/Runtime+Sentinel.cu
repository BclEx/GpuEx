#include <windows.h>
#include <process.h>
#include <assert.h>
#ifdef __device__
#undef __device__
#define __device__
#endif
#define RUNTIME_NAME RuntimeS
#include "Runtime.h"

#if OS_MAP
#pragma region OS_MAP
#include "Runtime+Alloc.cu"
#include "Runtime+BenignAlloc.cu"
#include "Runtime+Mem0.cu"
#include "Runtime+Mem1.cu"
#include "Runtime+TagBase.cu"

static RuntimeSentinelContext _ctx;

static bool Executor(void *tag, RuntimeSentinelMessage *data, int length)
{
	switch (data->OP)
	{
	case 1: {
		Messages::Stdio_fprintf *msg = (Messages::Stdio_fprintf *)data;
		msg->RC = fprintf(msg->File, msg->Format);
		return true; }
	case 2: {
		Messages::Stdio_fopen *msg = (Messages::Stdio_fopen *)data;
		msg->RC = fopen(msg->Filename, msg->Mode);
		return true; }
	case 3: {
		Messages::Stdio_fflush *msg = (Messages::Stdio_fflush *)data;
		msg->RC = fflush(msg->File);
		return true; }
	case 4: {
		Messages::Stdio_fclose *msg = (Messages::Stdio_fclose *)data;
		msg->RC = fclose(msg->File);
		return true; }
	case 5: {
		Messages::Stdio_fputc *msg = (Messages::Stdio_fputc *)data;
		msg->RC = fputc(msg->Ch, msg->File);
		return true; }
	case 6: {
		Messages::Stdio_fputs *msg = (Messages::Stdio_fputs *)data;
		msg->RC = fputs(msg->Str, msg->File);
		return true; }
	}
	return false;
}

static unsigned int __stdcall SentinelThread(void *data) 
{
	RuntimeSentinelContext *ctx = &_ctx; //(RuntimeSentinelContext *)data;
	RuntimeSentinelMap *map = ctx->Map;
	while (map)
	{
		long id = map->GetId;
		RuntimeSentinelCommand *cmd = (RuntimeSentinelCommand *)&map->Data[id%sizeof(map->Data)];
		volatile long *status = (volatile long *)&cmd->Status;
		unsigned int s_;
		while ((s_ = InterlockedCompareExchange((long *)status, 3, 2)) != 2) { /*printf("[%d ]", s_);*/ Sleep(50); } //
		if (cmd->Magic != SENTINEL_MAGIC)
		{
			printf("Bad Sentinel Magic");
			exit(1);
		}
		//map->Dump();
		//cmd->Dump();
		RuntimeSentinelMessage *msg = (RuntimeSentinelMessage *)cmd->Data;
		for (RuntimeSentinelExecutor *exec = _ctx.List; exec && exec->Executor && !exec->Executor(exec->Tag, msg, cmd->Length); exec = exec->Next) { }
		//printf(".");
		*status = (!msg->Async ? 4 : 0);
		map->GetId += SENTINEL_SIZE;
	}
	return 0;
}

static HANDLE _thread;
static RuntimeSentinelExecutor _baseExecutor;
void RuntimeSentinel::Initialize(RuntimeSentinelExecutor *executor)
{
#ifdef _GPU
	cudaErrorCheck(hipHostAlloc(&_ctx.Map, sizeof(RuntimeSentinelMap), hipHostMallocPortable));
	RuntimeSentinelContext *d_map;
	cudaErrorCheck(hipHostGetDevicePointer(&d_map, _ctx.Map, 0));
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(_runtimeSentinelMap), &d_map, sizeof(_ctx.Map)));
#else
	_ctx.Map = _runtimeSentinelMap = (RuntimeSentinelMap *)malloc(sizeof(RuntimeSentinelMap));
#endif
	memset(_ctx.Map, 0, sizeof(RuntimeSentinelMap));
	_baseExecutor.Name = "base";
	_baseExecutor.Executor = Executor;
	_baseExecutor.Tag = nullptr;
	RegisterExecutor(&_baseExecutor, true);
	if (executor)
		RegisterExecutor(executor, true);
	_thread = (HANDLE)_beginthreadex(0, 0, SentinelThread, nullptr, 0, 0);
}

void RuntimeSentinel::Shutdown()
{
	CloseHandle(_thread); _thread = nullptr;
#ifdef _GPU
	cudaErrorCheck(hipHostFree(_ctx.Map));
#else
	free(_ctx.Map);
#endif
	_ctx.Map = nullptr;
}

RuntimeSentinelExecutor *RuntimeSentinel::FindExecutor(const char *name)
{
	RuntimeSentinelExecutor *exec = nullptr;
	for (exec = _ctx.List; exec && name && strcmp(name, exec->Name); exec = exec->Next) { }
	return exec;
}

static void UnlinkExecutor(RuntimeSentinelExecutor *exec)
{
	if (!exec) { }
	else if (_ctx.List == exec)
		_ctx.List = exec->Next;
	else if (_ctx.List)
	{
		RuntimeSentinelExecutor *p = _ctx.List;
		while (p->Next && p->Next != exec)
			p = p->Next;
		if (p->Next == exec)
			p->Next = exec->Next;
	}
}

void RuntimeSentinel::RegisterExecutor(RuntimeSentinelExecutor *exec, bool default_)
{
	UnlinkExecutor(exec);
	if (default_ || !_ctx.List)
	{
		exec->Next = _ctx.List;
		_ctx.List = exec;
	}
	else
	{
		exec->Next = _ctx.List->Next;
		_ctx.List->Next = exec;
	}
	assert(_ctx.List != nullptr);
}

void RuntimeSentinel::UnregisterExecutor(RuntimeSentinelExecutor *exec)
{
	UnlinkExecutor(exec);
}

#pragma endregion
#endif