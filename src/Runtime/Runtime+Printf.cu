#include "hip/hip_runtime.h"
#include "Runtime.h"

#ifndef PRINT_BUF_SIZE
#define PRINT_BUF_SIZE 70
#endif
#define BUFSIZE PRINT_BUF_SIZE  // Size of the output buffer

enum TYPE : unsigned char
{
	TYPE_RADIX = 1,			// Integer types.  %d, %x, %o, and so forth
	TYPE_FLOAT = 2,			// Floating point.  %f
	TYPE_EXP = 3,			// Exponentional notation. %e and %E
	TYPE_GENERIC = 4,		// Floating or exponential, depending on exponent. %g
	TYPE_SIZE = 5,			// Return number of characters processed so far. %n
	TYPE_STRING = 6,		// Strings. %s
	TYPE_DYNSTRING = 7,		// Dynamically allocated strings. %z
	TYPE_PERCENT = 8,		// Percent symbol. %%
	TYPE_CHARX = 9,			// Characters. %c
	// The rest are extensions, not normally found in printf()
	TYPE_SQLESCAPE = 10,	// Strings with '\'' doubled.  %q
	TYPE_SQLESCAPE2 = 11,	// Strings with '\'' doubled and enclosed in '', NULL pointers replaced by SQL NULL.  %Q
	TYPE_TOKEN = 12,		// a pointer to a Token structure
	TYPE_SRCLIST = 13,		// a pointer to a SrcList
	TYPE_POINTER = 14,		// The %p conversion
	TYPE_SQLESCAPE3 = 15,	// %w -> Strings with '\"' doubled
	TYPE_ORDINAL = 16,		// %r -> 1st, 2nd, 3rd, 4th, etc.  English only
	//
	TYPE_INVALID = 0,		// Any unrecognized conversion type
};

enum FLAG : unsigned char
{
	FLAG_SIGNED = 1,	// True if the value to convert is signed
	FLAG_INTERN = 2,	// True if for internal use only
	FLAG_STRING = 4,	// Allow infinity precision
};

// Each builtin conversion character (ex: the 'd' in "%d") is described by an instance of the following structure
struct Info
{   // Information about each format field
	char Fmttype; // The format field code letter
	unsigned char Base; // The base for radix conversion
	FLAG Flags; // One or more of FLAG_ constants below
	TYPE Type; // Conversion paradigm
	unsigned char Charset; // Offset into aDigits[] of the digits string
	unsigned char Prefix; // Offset into aPrefix[] of the prefix string
};

// The following table is searched linearly, so it is good to put the most frequently used conversion types first.
__device__ static const char _digits[] = "0123456789ABCDEF0123456789abcdef";
__device__ static const char _prefix[] = "-x0\000X0";
__device__ static const Info _info[] = {
	{ 'd', 10, (FLAG)1, TYPE_RADIX,      0,  0 },
	{ 's',  0, (FLAG)4, TYPE_STRING,     0,  0 },
	{ 'g',  0, (FLAG)1, TYPE_GENERIC,    30, 0 },
	{ 'z',  0, (FLAG)4, TYPE_DYNSTRING,  0,  0 },
	{ 'q',  0, (FLAG)4, TYPE_SQLESCAPE,  0,  0 },
	{ 'Q',  0, (FLAG)4, TYPE_SQLESCAPE2, 0,  0 },
	{ 'w',  0, (FLAG)4, TYPE_SQLESCAPE3, 0,  0 },
	{ 'c',  0, (FLAG)0, TYPE_CHARX,      0,  0 },
	{ 'o',  8, (FLAG)0, TYPE_RADIX,      0,  2 },
	{ 'u', 10, (FLAG)0, TYPE_RADIX,      0,  0 },
	{ 'x', 16, (FLAG)0, TYPE_RADIX,      16, 1 },
	{ 'X', 16, (FLAG)0, TYPE_RADIX,      0,  4 },
#ifndef OMIT_FLOATING_POINT
	{ 'f',  0, (FLAG)1, TYPE_FLOAT,      0,  0 },
	{ 'e',  0, (FLAG)1, TYPE_EXP,        30, 0 },
	{ 'E',  0, (FLAG)1, TYPE_EXP,        14, 0 },
	{ 'G',  0, (FLAG)1, TYPE_GENERIC,    14, 0 },
#endif
	{ 'i', 10, (FLAG)1, TYPE_RADIX,      0,  0 },
	{ 'n',  0, (FLAG)0, TYPE_SIZE,       0,  0 },
	{ '%',  0, (FLAG)0, TYPE_PERCENT,    0,  0 },
	{ 'p', 16, (FLAG)0, TYPE_POINTER,    0,  1 },
	// All the rest have the FLAG_INTERN bit set and are thus for internal use only
	{ 'T',  0, (FLAG)2, TYPE_TOKEN,      0,  0 },
	{ 'S',  0, (FLAG)2, TYPE_SRCLIST,    0,  0 },
	{ 'r', 10, (FLAG)3, TYPE_ORDINAL,    0,  0 },
};

#ifndef OMIT_FLOATING_POINT
__device__ static char GetDigit(double64 *val, int *cnt)
{
	if ((*cnt) <= 0) return '0';
	(*cnt)--;
	int digit = (int)*val;
	double64 d = digit;
	digit += '0';
	*val = (*val - d)*10.0;
	return (char)digit;
}
#endif

__constant__ static const char _spaces[] = "                             ";
__device__ void TextBuilder::AppendSpace(int length)
{
	while (length >= (int)sizeof(_spaces)-1)
	{
		Append(_spaces, sizeof(_spaces)-1);
		length -= sizeof(_spaces)-1;
	}
	if (length > 0)
		Append(_spaces, length);
}

__constant__ static const char _ord[] = "thstndrd";
__device__ void TextBuilder::AppendFormat_(bool useExtended, const char *fmt, va_list &args) //: was: vxprintf
{
	char buf[BUFSIZE]; // Conversion buffer
	char *bufpt = nullptr; // Pointer to the conversion buffer
	int c; // Next character in the format string
	bool flag_leftjustify = false; // True if "-" flag is present
	int width = 0; // Width of the current field
	int length = 0; // Length of the field
	for (; (c = (*fmt)) != 0; ++fmt)
	{
		if (c != '%')
		{
			bufpt = (char *)fmt;
			int amt = 1;
			while ((c = (*++fmt)) != '%' && c != 0) amt++;
			Append(bufpt, amt);
			if (c == 0) break;
		}
		if ((c = (*++fmt)) == 0)
		{
			Append("%", 1);
			break;
		}
		// Find out what flags are present
		flag_leftjustify = false; // True if "-" flag is present
		bool flag_plussign = false; // True if "+" flag is present
		bool flag_blanksign = false; // True if " " flag is present
		bool flag_alternateform = false; // True if "#" flag is present
		bool flag_altform2 = false; // True if "!" flag is present
		bool flag_zeropad = false; // True if field width constant starts with zero
		bool done = false; // Loop termination flag
		do
		{
			switch (c)
			{
			case '-': flag_leftjustify = true; break;
			case '+': flag_plussign = true; break;
			case ' ': flag_blanksign = true; break;
			case '#': flag_alternateform = true; break;
			case '!': flag_altform2 = true; break;
			case '0': flag_zeropad = true; break;
			default: done = true; break;
			}
		} while (!done && (c = (*++fmt)) != 0);
		// Get the field width
		width = 0; // Width of the current field
		if (c == '*')
		{
			width = va_arg(args, int);
			if (width < 0)
			{
				flag_leftjustify = true;
				width = -width;
			}
			c = *++fmt;
		}
		else
		{
			while (c >= '0' && c <= '9')
			{
				width = width*10 + c - '0';
				c = *++fmt;
			}
		}
		// Get the precision
		int precision; // Precision of the current field
		if (c == '.')
		{
			precision = 0;
			c = *++fmt;
			if (c == '*')
			{
				precision = va_arg(args, int);
				if (precision < 0) precision = -precision;
				c = *++fmt;
			}
			else
			{
				while (c >= '0' && c <= '9')
				{
					precision = precision*10 + c - '0';
					c = *++fmt;
				}
			}
		}
		else
			precision = -1;
		// Get the conversion type modifier
		bool flag_long; // True if "l" flag is present
		bool flag_longlong; // True if the "ll" flag is present
		if (c == 'l')
		{
			flag_long = true;
			c = *++fmt;
			if (c == 'l')
			{
				flag_longlong = true;
				c = *++fmt;
			}
			else
				flag_longlong = false;
		}
		else
			flag_long = flag_longlong = false;
		// Fetch the info entry for the field
		const Info *info = &_info[0]; // Pointer to the appropriate info structure
		TYPE type = TYPE_INVALID; // Conversion paradigm
		int i;
		for (i = 0; i < _lengthof(_info); i++)
		{
			if (c == _info[i].Fmttype)
			{
				info = &_info[i];
				if (useExtended || (info->Flags & FLAG_INTERN) == 0) type = info->Type;
				else return;
				break;
			}
		}

		char prefix; // Prefix character.  "+" or "-" or " " or '\0'.
		unsigned long long longvalue; // Value for integer types
		double64 realvalue; // Value for real types
#ifndef OMIT_FLOATING_POINT
		int exp, e2; // exponent of real numbers
		int nsd; // Number of significant digits returned
		double rounder; // Used for rounding floating point values
		bool flag_dp; // True if decimal point should be shown
		bool flag_rtz; // True if trailing zeros should be removed
#endif

		// At this point, variables are initialized as follows:
		//   flag_alternateform          TRUE if a '#' is present.
		//   flag_altform2               TRUE if a '!' is present.
		//   flag_plussign               TRUE if a '+' is present.
		//   flag_leftjustify            TRUE if a '-' is present or if the field width was negative.
		//   flag_zeropad                TRUE if the width began with 0.
		//   flag_long                   TRUE if the letter 'l' (ell) prefixed the conversion character.
		//   flag_longlong               TRUE if the letter 'll' (ell ell) prefixed the conversion character.
		//   flag_blanksign              TRUE if a ' ' is present.
		//   width                       The specified field width.  This is always non-negative.  Zero is the default.
		//   precision                   The specified precision.  The default is -1.
		//   type                        The class of the conversion.
		//   info                        Pointer to the appropriate info struct.
		char *extra = nullptr; // Malloced memory used by some conversion
		char *out_; // Rendering buffer
		int outLength; // Size of the rendering buffer
		switch (type)
		{
		case TYPE_POINTER:
			flag_longlong = (sizeof(char *) == sizeof(long long));
			flag_long = (sizeof(char *) == sizeof(long int));
			// Fall through into the next case
		case TYPE_ORDINAL:
		case TYPE_RADIX:
			if (info->Flags & FLAG_SIGNED)
			{
				long long v;
				if (flag_longlong) v = va_arg(args, long long);
				else if (flag_long) v = va_arg(args, long int);
				else v = va_arg(args, int);
				if (v < 0)
				{
					longvalue = (v == SMALLEST_INT64 ? ((unsigned long long)1)<<63 : -v);
					prefix = '-';
				}
				else
				{
					longvalue = v;
					if (flag_plussign) prefix = '+';
					else if (flag_blanksign) prefix = ' ';
					else prefix = '\0';
				}
			}
			else
			{
				if (flag_longlong) longvalue = va_arg(args, unsigned long long);
				else if (flag_long) longvalue = va_arg(args, unsigned long int);
				else longvalue = va_arg(args, unsigned int);
				prefix = 0;
			}
			if (longvalue == 0) flag_alternateform = false;
			if (flag_zeropad && precision < width - (prefix != '\0'))
				precision = width-(prefix!=0);
			if (precision < BUFSIZE-10)
			{
				outLength = BUFSIZE;
				out_ = buf;
			}
			else
			{
				outLength = precision + 10;
				out_ = extra = (char *)_alloc(outLength);
				if (!out_)
				{
					AllocFailed = true;
					return;
				}
			}
			bufpt = &out_[outLength-1];
			if (type == TYPE_ORDINAL)
			{
				int x = (int)(longvalue % 10);
				if (x >= 4 || (longvalue/10)%10 == 1) x = 0;
				*(--bufpt) = _ord[x*2+1];
				*(--bufpt) = _ord[x*2];
			}
			{
				register const char *cset = &_digits[info->Charset]; // Use registers for speed
				register int base = info->Base;
				do // Convert to ascii
				{                                           
					*(--bufpt) = cset[longvalue % base];
					longvalue = longvalue / base;
				} while (longvalue > 0);
			}
			length = (int)(&out_[outLength-1]-bufpt);
			for (i = precision - length; i > 0; i--) *(--bufpt) = '0'; // Zero pad
			if (prefix) *(--bufpt) = prefix; // Add sign
			if (flag_alternateform && info->Prefix) // Add "0" or "0x"
			{
				char x;
				const char *pre = &_prefix[info->Prefix];
				for (; (x = (*pre)) != 0; pre++) *(--bufpt) = x;
			}
			length = (int)(&out_[outLength-1]-bufpt);
			break;
		case TYPE_FLOAT:
		case TYPE_EXP:
		case TYPE_GENERIC:
			realvalue = va_arg(args, double);
#ifdef OMIT_FLOATING_POINT
			length = 0;
#else
			if (precision < 0) precision = 6; // Set default precision
			if (realvalue < 0.0)
			{
				realvalue = -realvalue;
				prefix = '-';
			}
			else
			{
				if (flag_plussign) prefix = '+';
				else if (flag_blanksign) prefix = ' ';
				else prefix = 0;
			}
			if (type == TYPE_GENERIC && precision > 0) precision--;
#if 0
			// Rounding works like BSD when the constant 0.4999 is used.  Wierd!
			for (i = precision, rounder = 0.4999; i > 0; i--, rounder *= 0.1);
#else
			// It makes more sense to use 0.5
			for (i = precision, rounder = 0.5; i > 0; i--, rounder *= 0.1) { }
#endif
			if (type == TYPE_FLOAT) realvalue += rounder;
			// Normalize realvalue to within 10.0 > realvalue >= 1.0
			exp = 0;
			if (_isnan((double)realvalue))
			{
				bufpt = "NaN";
				length = 3;
				break;
			}
			if (realvalue > 0.0)
			{
				double64 scale = 1.0;
				while (realvalue >= 1e100*scale && exp <= 350) { scale *= 1e100;exp += 100; }
				while (realvalue >= 1e64*scale && exp <= 350) { scale *= 1e64; exp += 64; }
				while (realvalue >= 1e8*scale && exp <= 350) { scale *= 1e8; exp += 8; }
				while (realvalue >= 10.0*scale && exp <= 350) { scale *= 10.0; exp++; }
				realvalue /= scale;
				while (realvalue < 1e-8) { realvalue *= 1e8; exp -= 8; }
				while (realvalue < 1.0) { realvalue *= 10.0; exp--; }
				if (exp > 350)
				{
					if (prefix == '-') bufpt = "-Inf";
					else if (prefix == '+') bufpt = "+Inf";
					else bufpt = "Inf";
					length = _strlen(bufpt);
					break;
				}
			}
			bufpt = buf;
			// If the field type is etGENERIC, then convert to either etEXP or etFLOAT, as appropriate.
			if (type != TYPE_FLOAT)
			{
				realvalue += rounder;
				if (realvalue >= 10.0) { realvalue *= 0.1; exp++; }
			}
			if (type == TYPE_GENERIC)
			{
				flag_rtz = !flag_alternateform;
				if (exp < -4 || exp > precision) type = TYPE_EXP;
				else { precision = precision - exp; type = TYPE_FLOAT; }
			}
			else
				flag_rtz = flag_altform2;
			e2 = (type == TYPE_EXP ? 0 : exp);
			if (e2+precision+width > BUFSIZE - 15)
			{
				bufpt = extra = (char *)_alloc(e2+precision+width+15);
				if (!bufpt)
				{
					AllocFailed = true;
					return;
				}
			}
			out_ = bufpt;
			nsd = 16 + flag_altform2*10;
			flag_dp = (precision > 0) | flag_alternateform | flag_altform2;
			// The sign in front of the number
			if (prefix) *(bufpt++) = prefix;
			// Digits prior to the decimal point
			if (e2 < 0) *(bufpt++) = '0';
			else for (; e2 >= 0; e2--) *(bufpt++) = GetDigit(&realvalue, &nsd);
			// The decimal point
			if (flag_dp) *(bufpt++) = '.';
			// "0" digits after the decimal point but before the first significant digit of the number
			for (e2++; e2 < 0; precision--, e2++) { _assert(precision > 0); *(bufpt++) = '0'; }
			// Significant digits after the decimal point
			while ((precision--) > 0) *(bufpt++) = GetDigit(&realvalue, &nsd);
			// Remove trailing zeros and the "." if no digits follow the "."
			if (flag_rtz && flag_dp)
			{
				while (bufpt[-1] == '0') *(--bufpt) = 0;
				_assert(bufpt > out_);
				if (bufpt[-1] == '.')
				{
					if (flag_altform2) *(bufpt++) = '0';
					else *(--bufpt) = 0;
				}
			}
			// Add the "eNNN" suffix
			if (type == TYPE_EXP)
			{
				*(bufpt++) = _digits[info->Charset];
				if (exp < 0) { *(bufpt++) = '-'; exp = -exp; }
				else *(bufpt++) = '+';
				if (exp >= 100) { *(bufpt++) = (char)((exp/100)+'0'); exp %= 100; } // 100's digit
				*(bufpt++) = (char)(exp/10+'0'); // 10's digit
				*(bufpt++) = (char)(exp%10+'0'); // 1's digit
			}
			*bufpt = 0;

			// The converted number is in buf[] and zero terminated. Output it. Note that the number is in the usual order, not reversed as with integer conversions.
			length = (int)(bufpt-out_);
			bufpt = out_;

			// Special case:  Add leading zeros if the flag_zeropad flag is set and we are not left justified
			if (flag_zeropad && !flag_leftjustify && length < width)
			{
				int pad = width - length;
				for (i = width; i >= pad; i--) bufpt[i] = bufpt[i-pad];
				i = (prefix != '\0');
				while (pad--) bufpt[i++] = '0';
				length = width;
			}
#endif
			break;
		case TYPE_SIZE:
			*(va_arg(args, int*)) = Size;
			length = width = 0;
			break;
		case TYPE_PERCENT:
			buf[0] = '%';
			bufpt = buf;
			length = 1;
			break;
		case TYPE_CHARX:
			c = va_arg(args, int);
			buf[0] = (char)c;
			if (precision >= 0)
			{
				for (i = 1; i < precision; i++) buf[i] = (char)c;
				length = precision;
			}
			else length =1;
			bufpt = buf;
			break;
		case TYPE_STRING:
		case TYPE_DYNSTRING:
			bufpt = va_arg(args, char*);
			if (bufpt == 0) bufpt = "";
			else if (type == TYPE_DYNSTRING) extra = bufpt;
			if (precision >= 0) for (length = 0; length < precision && bufpt[length]; length++) { }
			else length = _strlen(bufpt);
			break;
		case TYPE_SQLESCAPE:
		case TYPE_SQLESCAPE2:
		case TYPE_SQLESCAPE3: {
			char q = (type == TYPE_SQLESCAPE3 ? '"' : '\''); // Quote character
			char *escarg = va_arg(args, char*);
			bool isnull = (escarg == 0);
			if (isnull) escarg = (type == TYPE_SQLESCAPE2 ? "NULL" : "(NULL)");
			int k = precision;
			int j, n;
			char ch;
			for (i = n = 0; k != 0 && (ch = escarg[i]) != 0; i++, k--)
				if (ch == q) n++;
			bool needQuote = (!isnull && type == TYPE_SQLESCAPE2);
			n += i + 1 + needQuote*2;
			if (n > BUFSIZE)
			{
				bufpt = extra = (char *)_alloc(n);
				if (!bufpt)
				{
					AllocFailed = true;
					return;
				}
			}
			else
				bufpt = buf;
			j = 0;
			if (needQuote) bufpt[j++] = q;
			k = i;
			for (i = 0; i < k; i++)
			{
				bufpt[j++] = ch = escarg[i];
				if (ch == q) bufpt[j++] = ch;
			}
			if (needQuote) bufpt[j++] = q;
			bufpt[j] = 0;
			length = j;
			// The precision in %q and %Q means how many input characters to consume, not the length of the output...
			// if (precision>=0 && precision<length) length = precision;
			break; }
		case TYPE_TOKEN: {
			TagBase_RuntimeStatics.AppendFormat[0](this, args);
			length = width = 0;
			break; }
		case TYPE_SRCLIST: {
			TagBase_RuntimeStatics.AppendFormat[1](this, args);
			length = width = 0;
			break; }
		default: {
			_assert(type == TYPE_INVALID);
			return; }
		}
		// The text of the conversion is pointed to by "bufpt" and is "length" characters long.  The field width is "width".  Do the output.
		if (!flag_leftjustify)
		{
			register int nspace = width-length;
			if (nspace > 0) AppendSpace(nspace);
		}
		if (length > 0) Append(bufpt, length);
		if (flag_leftjustify)
		{
			register int nspace = width-length;
			if (nspace > 0) AppendSpace(nspace);
		}
		if (extra) _free(extra);
	}
}

__device__ void TextBuilder::Append(const char *z, int length)
{
	_assert(z != nullptr || length == 0);
	if (Overflowed | AllocFailed)
	{
		ASSERTCOVERAGE(Overflowed);
		ASSERTCOVERAGE(AllocFailed);
		return;
	}
	_assert(Text != nullptr || Index == 0);
	if (length < 0)
		length = _strlen(z);
	if (length == 0 || _NEVER(z == nullptr))
		return;
	if (Index + length >= Size)
	{
		char *newText;
		if (!AllocType)
		{
			Overflowed = true;
			length = Size - Index - 1;
			if (length <= 0)
				return;
		}
		else
		{
			char *oldText = (Text == Base ? nullptr : Text);
			long long newSize = Index;
			newSize += length + 1;
			if (newSize > MaxSize)
			{
				Reset();
				Overflowed = true;
				return;
			}
			else
				Size = (int)newSize;
			if (AllocType == 1)
				newText = (char *)_tagrealloc(Tag, oldText, Size);
			else
				newText = (char *)_realloc(oldText, Size);
			if (newText)
			{
				if (!oldText && Index > 0) _memcpy(newText, Text, Index);
				Text = newText;
			}
			else
			{
				AllocFailed = true;
				Reset();
				return;
			}
		}
	}
	_assert(Text != nullptr);
	_memcpy(&Text[Index], z, length);
	Index += length;
}

__device__ char *TextBuilder::ToString()
{
	if (Text)
	{
		Text[Index] = 0;
		if (AllocType && Text == Base)
		{
			if (AllocType == 1)
				Text = (char *)_tagalloc(Tag, Index + 1);
			else
				Text = (char *)_alloc(Index + 1);
			if (Text)
				_memcpy(Text, Base, Index + 1);
			else
				AllocFailed = true;
		}
	}
	return Text;
}

__device__ void TextBuilder::Reset()
{
	if (Text != Base)
	{
		if (AllocType == 1)
			_tagfree(Tag, Text);
		else
			_free(Text);
	}
	Text = nullptr;
}

__device__ void TextBuilder::Init(TextBuilder *b, char *text, int capacity, int maxSize)
{
	b->Text = b->Base = text;
	b->Tag = nullptr;
	b->Index = 0;
	b->Size = capacity;
	b->MaxSize = maxSize;
	b->AllocType = 1;
	b->Overflowed = false;
	b->AllocFailed = false;
}

__device__ char *_vmtagprintf(TagBase *tag, const char *fmt, va_list *args)
{
	//if (!RuntimeInitialize()) return nullptr;
	_assert(tag != nullptr);
	char base[PRINT_BUF_SIZE];
	TextBuilder b;
	TextBuilder::Init(&b, base, sizeof(base), 1000000000); //? tag->Limit[LIMIT_LENGTH]);
	b.Tag = tag;
	b.AppendFormat_(true, fmt, *args);
	char *z = b.ToString();
	// if (b.AllocFailed) _tagallocfailed(tag);
	return z;
}

__device__ char *_vmprintf(const char *fmt, va_list *args)
{
	//if (!RuntimeInitialize()) return nullptr;
	char base[PRINT_BUF_SIZE];
	TextBuilder b;
	TextBuilder::Init(&b, base, sizeof(base), CORE_MAX_LENGTH);
	b.AllocType = 2;
	b.AppendFormat_(false, fmt, *args);
	return b.ToString();
}

__device__ char *__vsnprintf(const char *buf, size_t bufLen, const char *fmt, va_list *args)
{
	if (bufLen <= 0) return (char *)buf;
	TextBuilder b;
	TextBuilder::Init(&b, (char *)buf, (int)bufLen, 0);
	b.AllocType = 0;
	b.AppendFormat_(false, fmt, *args);
	return b.ToString();
}

