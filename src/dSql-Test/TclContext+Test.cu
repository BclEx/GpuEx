
#include <hip/hip_runtime.h>
#ifdef TCLSH
/*****************************************************************************
** All of the code that follows is used to build standalone TCL interpreters
** that are statically linked with SQLite.  Enable these by compiling
** with -DTCLSH=n where n can be 1 or 2.  An n of 1 generates a standard
** tclsh but with SQLite built in.  An n of 2 generates the SQLite space
** analysis program.
*/

#if defined(SQLITE_TEST) || defined(SQLITE_TCLMD5)
/*
* This code implements the MD5 message-digest algorithm.
* The algorithm is due to Ron Rivest.  This code was
* written by Colin Plumb in 1993, no copyright is claimed.
* This code is in the public domain; do with it what you wish.
*
* Equivalent code is available from RSA Data Security, Inc.
* This code has been tested against that, and is equivalent,
* except that you don't need to include two pages of legalese
* with every copy.
*
* To compute the message digest of a chunk of bytes, declare an
* MD5Context structure, pass it to MD5Init, call MD5Update as
* needed on buffers full of bytes, and then call MD5Final, which
* will fill a supplied 16-byte array with the digest.
*/

/*
* If compiled on a machine that doesn't have a 32-bit integer,
* you just set "uint32" to the appropriate datatype for an
* unsigned 32-bit integer.  For example:
*
*       cc -Duint32='unsigned long' md5.c
*
*/
#ifndef uint32
#  define uint32 unsigned int
#endif

struct MD5Context {
	int isInit;
	uint32 buf[4];
	uint32 bits[2];
	unsigned char in[64];
};
typedef struct MD5Context MD5Context;

/*
* Note: this code is harmless on little-endian machines.
*/
static void byteReverse (unsigned char *buf, unsigned longs){
	uint32 t;
	do {
		t = (uint32)((unsigned)buf[3]<<8 | buf[2]) << 16 |
			((unsigned)buf[1]<<8 | buf[0]);
		*(uint32 *)buf = t;
		buf += 4;
	} while (--longs);
}
/* The four core functions - F1 is optimized somewhat */

/* #define F1(x, y, z) (x & y | ~x & z) */
#define F1(x, y, z) (z ^ (x & (y ^ z)))
#define F2(x, y, z) F1(z, x, y)
#define F3(x, y, z) (x ^ y ^ z)
#define F4(x, y, z) (y ^ (x | ~z))

/* This is the central step in the MD5 algorithm. */
#define MD5STEP(f, w, x, y, z, data, s) \
	( w += f(x, y, z) + data,  w = w<<s | w>>(32-s),  w += x )

/*
* The core of the MD5 algorithm, this alters an existing MD5 hash to
* reflect the addition of 16 longwords of new data.  MD5Update blocks
* the data and converts bytes into longwords for this routine.
*/
static void MD5Transform(uint32 buf[4], const uint32 in[16]){
	register uint32 a, b, c, d;

	a = buf[0];
	b = buf[1];
	c = buf[2];
	d = buf[3];

	MD5STEP(F1, a, b, c, d, in[ 0]+0xd76aa478,  7);
	MD5STEP(F1, d, a, b, c, in[ 1]+0xe8c7b756, 12);
	MD5STEP(F1, c, d, a, b, in[ 2]+0x242070db, 17);
	MD5STEP(F1, b, c, d, a, in[ 3]+0xc1bdceee, 22);
	MD5STEP(F1, a, b, c, d, in[ 4]+0xf57c0faf,  7);
	MD5STEP(F1, d, a, b, c, in[ 5]+0x4787c62a, 12);
	MD5STEP(F1, c, d, a, b, in[ 6]+0xa8304613, 17);
	MD5STEP(F1, b, c, d, a, in[ 7]+0xfd469501, 22);
	MD5STEP(F1, a, b, c, d, in[ 8]+0x698098d8,  7);
	MD5STEP(F1, d, a, b, c, in[ 9]+0x8b44f7af, 12);
	MD5STEP(F1, c, d, a, b, in[10]+0xffff5bb1, 17);
	MD5STEP(F1, b, c, d, a, in[11]+0x895cd7be, 22);
	MD5STEP(F1, a, b, c, d, in[12]+0x6b901122,  7);
	MD5STEP(F1, d, a, b, c, in[13]+0xfd987193, 12);
	MD5STEP(F1, c, d, a, b, in[14]+0xa679438e, 17);
	MD5STEP(F1, b, c, d, a, in[15]+0x49b40821, 22);

	MD5STEP(F2, a, b, c, d, in[ 1]+0xf61e2562,  5);
	MD5STEP(F2, d, a, b, c, in[ 6]+0xc040b340,  9);
	MD5STEP(F2, c, d, a, b, in[11]+0x265e5a51, 14);
	MD5STEP(F2, b, c, d, a, in[ 0]+0xe9b6c7aa, 20);
	MD5STEP(F2, a, b, c, d, in[ 5]+0xd62f105d,  5);
	MD5STEP(F2, d, a, b, c, in[10]+0x02441453,  9);
	MD5STEP(F2, c, d, a, b, in[15]+0xd8a1e681, 14);
	MD5STEP(F2, b, c, d, a, in[ 4]+0xe7d3fbc8, 20);
	MD5STEP(F2, a, b, c, d, in[ 9]+0x21e1cde6,  5);
	MD5STEP(F2, d, a, b, c, in[14]+0xc33707d6,  9);
	MD5STEP(F2, c, d, a, b, in[ 3]+0xf4d50d87, 14);
	MD5STEP(F2, b, c, d, a, in[ 8]+0x455a14ed, 20);
	MD5STEP(F2, a, b, c, d, in[13]+0xa9e3e905,  5);
	MD5STEP(F2, d, a, b, c, in[ 2]+0xfcefa3f8,  9);
	MD5STEP(F2, c, d, a, b, in[ 7]+0x676f02d9, 14);
	MD5STEP(F2, b, c, d, a, in[12]+0x8d2a4c8a, 20);

	MD5STEP(F3, a, b, c, d, in[ 5]+0xfffa3942,  4);
	MD5STEP(F3, d, a, b, c, in[ 8]+0x8771f681, 11);
	MD5STEP(F3, c, d, a, b, in[11]+0x6d9d6122, 16);
	MD5STEP(F3, b, c, d, a, in[14]+0xfde5380c, 23);
	MD5STEP(F3, a, b, c, d, in[ 1]+0xa4beea44,  4);
	MD5STEP(F3, d, a, b, c, in[ 4]+0x4bdecfa9, 11);
	MD5STEP(F3, c, d, a, b, in[ 7]+0xf6bb4b60, 16);
	MD5STEP(F3, b, c, d, a, in[10]+0xbebfbc70, 23);
	MD5STEP(F3, a, b, c, d, in[13]+0x289b7ec6,  4);
	MD5STEP(F3, d, a, b, c, in[ 0]+0xeaa127fa, 11);
	MD5STEP(F3, c, d, a, b, in[ 3]+0xd4ef3085, 16);
	MD5STEP(F3, b, c, d, a, in[ 6]+0x04881d05, 23);
	MD5STEP(F3, a, b, c, d, in[ 9]+0xd9d4d039,  4);
	MD5STEP(F3, d, a, b, c, in[12]+0xe6db99e5, 11);
	MD5STEP(F3, c, d, a, b, in[15]+0x1fa27cf8, 16);
	MD5STEP(F3, b, c, d, a, in[ 2]+0xc4ac5665, 23);

	MD5STEP(F4, a, b, c, d, in[ 0]+0xf4292244,  6);
	MD5STEP(F4, d, a, b, c, in[ 7]+0x432aff97, 10);
	MD5STEP(F4, c, d, a, b, in[14]+0xab9423a7, 15);
	MD5STEP(F4, b, c, d, a, in[ 5]+0xfc93a039, 21);
	MD5STEP(F4, a, b, c, d, in[12]+0x655b59c3,  6);
	MD5STEP(F4, d, a, b, c, in[ 3]+0x8f0ccc92, 10);
	MD5STEP(F4, c, d, a, b, in[10]+0xffeff47d, 15);
	MD5STEP(F4, b, c, d, a, in[ 1]+0x85845dd1, 21);
	MD5STEP(F4, a, b, c, d, in[ 8]+0x6fa87e4f,  6);
	MD5STEP(F4, d, a, b, c, in[15]+0xfe2ce6e0, 10);
	MD5STEP(F4, c, d, a, b, in[ 6]+0xa3014314, 15);
	MD5STEP(F4, b, c, d, a, in[13]+0x4e0811a1, 21);
	MD5STEP(F4, a, b, c, d, in[ 4]+0xf7537e82,  6);
	MD5STEP(F4, d, a, b, c, in[11]+0xbd3af235, 10);
	MD5STEP(F4, c, d, a, b, in[ 2]+0x2ad7d2bb, 15);
	MD5STEP(F4, b, c, d, a, in[ 9]+0xeb86d391, 21);

	buf[0] += a;
	buf[1] += b;
	buf[2] += c;
	buf[3] += d;
}

/*
* Start MD5 accumulation.  Set bit count to 0 and buffer to mysterious
* initialization constants.
*/
static void MD5Init(MD5Context *ctx){
	ctx->isInit = 1;
	ctx->buf[0] = 0x67452301;
	ctx->buf[1] = 0xefcdab89;
	ctx->buf[2] = 0x98badcfe;
	ctx->buf[3] = 0x10325476;
	ctx->bits[0] = 0;
	ctx->bits[1] = 0;
}

/*
* Update context to reflect the concatenation of another buffer full
* of bytes.
*/
static 
	void MD5Update(MD5Context *ctx, const unsigned char *buf, unsigned int len){
		uint32 t;

		/* Update bitcount */

		t = ctx->bits[0];
		if ((ctx->bits[0] = t + ((uint32)len << 3)) < t)
			ctx->bits[1]++; /* Carry from low to high */
		ctx->bits[1] += len >> 29;

		t = (t >> 3) & 0x3f;    /* Bytes already in shsInfo->data */

		/* Handle any leading odd-sized chunks */

		if ( t ) {
			unsigned char *p = (unsigned char *)ctx->in + t;

			t = 64-t;
			if (len < t) {
				memcpy(p, buf, len);
				return;
			}
			memcpy(p, buf, t);
			byteReverse(ctx->in, 16);
			MD5Transform(ctx->buf, (uint32 *)ctx->in);
			buf += t;
			len -= t;
		}

		/* Process data in 64-byte chunks */

		while (len >= 64) {
			memcpy(ctx->in, buf, 64);
			byteReverse(ctx->in, 16);
			MD5Transform(ctx->buf, (uint32 *)ctx->in);
			buf += 64;
			len -= 64;
		}

		/* Handle any remaining bytes of data. */

		memcpy(ctx->in, buf, len);
}

/*
* Final wrapup - pad to 64-byte boundary with the bit pattern 
* 1 0* (64-bit count of bits processed, MSB-first)
*/
static void MD5Final(unsigned char digest[16], MD5Context *ctx){
	unsigned count;
	unsigned char *p;

	/* Compute number of bytes mod 64 */
	count = (ctx->bits[0] >> 3) & 0x3F;

	/* Set the first char of padding to 0x80.  This is safe since there is
	always at least one byte free */
	p = ctx->in + count;
	*p++ = 0x80;

	/* Bytes of padding needed to make 64 bytes */
	count = 64 - 1 - count;

	/* Pad out to 56 mod 64 */
	if (count < 8) {
		/* Two lots of padding:  Pad the first block to 64 bytes */
		memset(p, 0, count);
		byteReverse(ctx->in, 16);
		MD5Transform(ctx->buf, (uint32 *)ctx->in);

		/* Now fill the next block with 56 bytes */
		memset(ctx->in, 0, 56);
	} else {
		/* Pad block to 56 bytes */
		memset(p, 0, count-8);
	}
	byteReverse(ctx->in, 14);

	/* Append length in bits and transform */
	((uint32 *)ctx->in)[ 14 ] = ctx->bits[0];
	((uint32 *)ctx->in)[ 15 ] = ctx->bits[1];

	MD5Transform(ctx->buf, (uint32 *)ctx->in);
	byteReverse((unsigned char *)ctx->buf, 4);
	memcpy(digest, ctx->buf, 16);
	memset(ctx, 0, sizeof(ctx));    /* In case it is sensitive */
}

/*
** Convert a 128-bit MD5 digest into a 32-digit base-16 number.
*/
static void MD5DigestToBase16(unsigned char *digest, char *zBuf){
	static char const zEncode[] = "0123456789abcdef";
	int i, j;

	for(j=i=0; i<16; i++){
		int a = digest[i];
		zBuf[j++] = zEncode[(a>>4)&0xf];
		zBuf[j++] = zEncode[a & 0xf];
	}
	zBuf[j] = 0;
}


/*
** Convert a 128-bit MD5 digest into sequency of eight 5-digit integers
** each representing 16 bits of the digest and separated from each
** other by a "-" character.
*/
static void MD5DigestToBase10x8(unsigned char digest[16], char zDigest[50]){
	int i, j;
	unsigned int x;
	for(i=j=0; i<16; i+=2){
		x = digest[i]*256 + digest[i+1];
		if( i>0 ) zDigest[j++] = '-';
		sprintf(&zDigest[j], "%05u", x);
		j += 5;
	}
	zDigest[j] = 0;
}

/*
** A TCL command for md5.  The argument is the text to be hashed.  The
** Result is the hash in base64.  
*/
static int md5_cmd(void*cd, Tcl_Interp *interp, int argc, const char **argv){
	MD5Context ctx;
	unsigned char digest[16];
	char zBuf[50];
	void (*converter)(unsigned char*, char*);

	if( argc!=2 ){
		Tcl_AppendResult(interp,"wrong # args: should be \"", argv[0], 
			" TEXT\"", 0);
		return TCL_ERROR;
	}
	MD5Init(&ctx);
	MD5Update(&ctx, (unsigned char*)argv[1], (unsigned)strlen(argv[1]));
	MD5Final(digest, &ctx);
	converter = (void(*)(unsigned char*,char*))cd;
	converter(digest, zBuf);
	Tcl_AppendResult(interp, zBuf, (char*)0);
	return TCL_OK;
}

/*
** A TCL command to take the md5 hash of a file.  The argument is the
** name of the file.
*/
static int md5file_cmd(void*cd, Tcl_Interp*interp, int argc, const char **argv){
	FILE *in;
	MD5Context ctx;
	void (*converter)(unsigned char*, char*);
	unsigned char digest[16];
	char zBuf[10240];

	if( argc!=2 ){
		Tcl_AppendResult(interp,"wrong # args: should be \"", argv[0], 
			" FILENAME\"", 0);
		return TCL_ERROR;
	}
	in = fopen(argv[1],"rb");
	if( in==0 ){
		Tcl_AppendResult(interp,"unable to open file \"", argv[1], 
			"\" for reading", 0);
		return TCL_ERROR;
	}
	MD5Init(&ctx);
	for(;;){
		int n;
		n = (int)fread(zBuf, 1, sizeof(zBuf), in);
		if( n<=0 ) break;
		MD5Update(&ctx, (unsigned char*)zBuf, (unsigned)n);
	}
	fclose(in);
	MD5Final(digest, &ctx);
	converter = (void(*)(unsigned char*,char*))cd;
	converter(digest, zBuf);
	Tcl_AppendResult(interp, zBuf, (char*)0);
	return TCL_OK;
}

/*
** Register the four new TCL commands for generating MD5 checksums
** with the TCL interpreter.
*/
int Md5_Init(Tcl_Interp *interp){
	Tcl_CreateCommand(interp, "md5", (Tcl_CmdProc*)md5_cmd,
		MD5DigestToBase16, 0);
	Tcl_CreateCommand(interp, "md5-10x8", (Tcl_CmdProc*)md5_cmd,
		MD5DigestToBase10x8, 0);
	Tcl_CreateCommand(interp, "md5file", (Tcl_CmdProc*)md5file_cmd,
		MD5DigestToBase16, 0);
	Tcl_CreateCommand(interp, "md5file-10x8", (Tcl_CmdProc*)md5file_cmd,
		MD5DigestToBase10x8, 0);
	return TCL_OK;
}
#endif /* defined(SQLITE_TEST) || defined(SQLITE_TCLMD5) */

#if defined(SQLITE_TEST)
/*
** During testing, the special md5sum() aggregate function is available.
** inside SQLite.  The following routines implement that function.
*/
static void md5step(sqlite3_context *context, int argc, sqlite3_value **argv){
	MD5Context *p;
	int i;
	if( argc<1 ) return;
	p = sqlite3_aggregate_context(context, sizeof(*p));
	if( p==0 ) return;
	if( !p->isInit ){
		MD5Init(p);
	}
	for(i=0; i<argc; i++){
		const char *zData = (char*)sqlite3_value_text(argv[i]);
		if( zData ){
			MD5Update(p, (unsigned char*)zData, (int)strlen(zData));
		}
	}
}
static void md5finalize(sqlite3_context *context){
	MD5Context *p;
	unsigned char digest[16];
	char zBuf[33];
	p = sqlite3_aggregate_context(context, sizeof(*p));
	MD5Final(digest,p);
	MD5DigestToBase16(digest, zBuf);
	sqlite3_result_text(context, zBuf, -1, SQLITE_TRANSIENT);
}
int Md5_Register(sqlite3 *db){
	int rc = sqlite3_create_function(db, "md5sum", -1, SQLITE_UTF8, 0, 0, 
		md5step, md5finalize);
	sqlite3_overload_function(db, "md5sum", -1);  /* To exercise this API */
	return rc;
}
#endif /* defined(SQLITE_TEST) */


/*
** If the macro TCLSH is one, then put in code this for the
** "main" routine that will initialize Tcl and take input from
** standard input, or if a file is named on the command line
** the TCL interpreter reads and evaluates that file.
*/
#if TCLSH==1
static const char *tclsh_main_loop(void){
	static const char zMainloop[] =
		"set line {}\n"
		"while {![eof stdin]} {\n"
		"if {$line!=\"\"} {\n"
		"puts -nonewline \"> \"\n"
		"} else {\n"
		"puts -nonewline \"% \"\n"
		"}\n"
		"flush stdout\n"
		"append line [gets stdin]\n"
		"if {[info complete $line]} {\n"
		"if {[catch {uplevel #0 $line} result]} {\n"
		"puts stderr \"Error: $result\"\n"
		"} elseif {$result!=\"\"} {\n"
		"puts $result\n"
		"}\n"
		"set line {}\n"
		"} else {\n"
		"append line \\n\n"
		"}\n"
		"}\n"
		;
	return zMainloop;
}
#endif
#if TCLSH==2
static const char *tclsh_main_loop(void);
#endif

#ifdef SQLITE_TEST
static void init_all(Tcl_Interp *);
static int init_all_cmd(
	ClientData cd,
	Tcl_Interp *interp,
	int objc,
	Tcl_Obj *CONST objv[]
){

	Tcl_Interp *slave;
	if( objc!=2 ){
		Tcl_WrongNumArgs(interp, 1, objv, "SLAVE");
		return TCL_ERROR;
	}

	slave = Tcl_GetSlave(interp, Tcl_GetString(objv[1]));
	if( !slave ){
		return TCL_ERROR;
	}

	init_all(slave);
	return TCL_OK;
}

/*
** Tclcmd: db_use_legacy_prepare DB BOOLEAN
**
**   The first argument to this command must be a database command created by
**   [sqlite3]. If the second argument is true, then the handle is configured
**   to use the sqlite3_prepare_v2() function to prepare statements. If it
**   is false, sqlite3_prepare().
*/
static int db_use_legacy_prepare_cmd(
	ClientData cd,
	Tcl_Interp *interp,
	int objc,
	Tcl_Obj *CONST objv[]
){
	Tcl_CmdInfo cmdInfo;
	SqliteDb *pDb;
	int bPrepare;

	if( objc!=3 ){
		Tcl_WrongNumArgs(interp, 1, objv, "DB BOOLEAN");
		return TCL_ERROR;
	}

	if( !Tcl_GetCommandInfo(interp, Tcl_GetString(objv[1]), &cmdInfo) ){
		Tcl_AppendResult(interp, "no such db: ", Tcl_GetString(objv[1]), (char*)0);
		return TCL_ERROR;
	}
	pDb = (SqliteDb*)cmdInfo.objClientData;
	if( Tcl_GetBooleanFromObj(interp, objv[2], &bPrepare) ){
		return TCL_ERROR;
	}

	pDb->bLegacyPrepare = bPrepare;

	Tcl_ResetResult(interp);
	return TCL_OK;
}
#endif

/*
** Configure the interpreter passed as the first argument to have access
** to the commands and linked variables that make up:
**
**   * the [sqlite3] extension itself, 
**
**   * If SQLITE_TCLMD5 or SQLITE_TEST is defined, the Md5 commands, and
**
**   * If SQLITE_TEST is set, the various test interfaces used by the Tcl
**     test suite.
*/
static void init_all(Tcl_Interp *interp){
	Sqlite3_Init(interp);

#if defined(SQLITE_TEST) || defined(SQLITE_TCLMD5)
	Md5_Init(interp);
#endif

	/* Install the [register_dbstat_vtab] command to access the implementation
	** of virtual table dbstat (source file test_stat.c). This command is
	** required for testfixture and sqlite3_analyzer, but not by the production
	** Tcl extension.  */
#if defined(SQLITE_TEST) || TCLSH==2
	{
		extern int SqlitetestStat_Init(Tcl_Interp*);
		SqlitetestStat_Init(interp);
	}
#endif

#ifdef SQLITE_TEST
	{
		extern int Sqliteconfig_Init(Tcl_Interp*);
		extern int Sqlitetest1_Init(Tcl_Interp*);
		extern int Sqlitetest2_Init(Tcl_Interp*);
		extern int Sqlitetest3_Init(Tcl_Interp*);
		extern int Sqlitetest4_Init(Tcl_Interp*);
		extern int Sqlitetest5_Init(Tcl_Interp*);
		extern int Sqlitetest6_Init(Tcl_Interp*);
		extern int Sqlitetest7_Init(Tcl_Interp*);
		extern int Sqlitetest8_Init(Tcl_Interp*);
		extern int Sqlitetest9_Init(Tcl_Interp*);
		extern int Sqlitetestasync_Init(Tcl_Interp*);
		extern int Sqlitetest_autoext_Init(Tcl_Interp*);
		extern int Sqlitetest_demovfs_Init(Tcl_Interp *);
		extern int Sqlitetest_func_Init(Tcl_Interp*);
		extern int Sqlitetest_hexio_Init(Tcl_Interp*);
		extern int Sqlitetest_init_Init(Tcl_Interp*);
		extern int Sqlitetest_malloc_Init(Tcl_Interp*);
		extern int Sqlitetest_mutex_Init(Tcl_Interp*);
		extern int Sqlitetestschema_Init(Tcl_Interp*);
		extern int Sqlitetestsse_Init(Tcl_Interp*);
		extern int Sqlitetesttclvar_Init(Tcl_Interp*);
		extern int Sqlitetestfs_Init(Tcl_Interp*);
		extern int SqlitetestThread_Init(Tcl_Interp*);
		extern int SqlitetestOnefile_Init();
		extern int SqlitetestOsinst_Init(Tcl_Interp*);
		extern int Sqlitetestbackup_Init(Tcl_Interp*);
		extern int Sqlitetestintarray_Init(Tcl_Interp*);
		extern int Sqlitetestvfs_Init(Tcl_Interp *);
		extern int Sqlitetestrtree_Init(Tcl_Interp*);
		extern int Sqlitequota_Init(Tcl_Interp*);
		extern int Sqlitemultiplex_Init(Tcl_Interp*);
		extern int SqliteSuperlock_Init(Tcl_Interp*);
		extern int SqlitetestSyscall_Init(Tcl_Interp*);
		extern int Sqlitetestfuzzer_Init(Tcl_Interp*);
		extern int Sqlitetestwholenumber_Init(Tcl_Interp*);
		extern int Sqlitetestregexp_Init(Tcl_Interp*);

#if defined(SQLITE_ENABLE_FTS3) || defined(SQLITE_ENABLE_FTS4)
		extern int Sqlitetestfts3_Init(Tcl_Interp *interp);
#endif

#ifdef SQLITE_ENABLE_ZIPVFS
		extern int Zipvfs_Init(Tcl_Interp*);
		Zipvfs_Init(interp);
#endif

		Sqliteconfig_Init(interp);
		Sqlitetest1_Init(interp);
		Sqlitetest2_Init(interp);
		Sqlitetest3_Init(interp);
		Sqlitetest4_Init(interp);
		Sqlitetest5_Init(interp);
		Sqlitetest6_Init(interp);
		Sqlitetest7_Init(interp);
		Sqlitetest8_Init(interp);
		Sqlitetest9_Init(interp);
		Sqlitetestasync_Init(interp);
		Sqlitetest_autoext_Init(interp);
		Sqlitetest_demovfs_Init(interp);
		Sqlitetest_func_Init(interp);
		Sqlitetest_hexio_Init(interp);
		Sqlitetest_init_Init(interp);
		Sqlitetest_malloc_Init(interp);
		Sqlitetest_mutex_Init(interp);
		Sqlitetestschema_Init(interp);
		Sqlitetesttclvar_Init(interp);
		Sqlitetestfs_Init(interp);
		SqlitetestThread_Init(interp);
		SqlitetestOnefile_Init(interp);
		SqlitetestOsinst_Init(interp);
		Sqlitetestbackup_Init(interp);
		Sqlitetestintarray_Init(interp);
		Sqlitetestvfs_Init(interp);
		Sqlitetestrtree_Init(interp);
		Sqlitequota_Init(interp);
		Sqlitemultiplex_Init(interp);
		SqliteSuperlock_Init(interp);
		SqlitetestSyscall_Init(interp);
		Sqlitetestfuzzer_Init(interp);
		Sqlitetestwholenumber_Init(interp);
		Sqlitetestregexp_Init(interp);

#if defined(SQLITE_ENABLE_FTS3) || defined(SQLITE_ENABLE_FTS4)
		Sqlitetestfts3_Init(interp);
#endif

		Tcl_CreateObjCommand(
			interp, "load_testfixture_extensions", init_all_cmd, 0, 0
			);
		Tcl_CreateObjCommand(
			interp, "db_use_legacy_prepare", db_use_legacy_prepare_cmd, 0, 0
			);

#ifdef SQLITE_SSE
		Sqlitetestsse_Init(interp);
#endif
	}
#endif
}

#define TCLSH_MAIN main   /* Needed to fake out mktclapp */
int TCLSH_MAIN(int argc, char **argv){
	Tcl_Interp *interp;

	/* Call sqlite3_shutdown() once before doing anything else. This is to
	** test that sqlite3_shutdown() can be safely called by a process before
	** sqlite3_initialize() is. */
	sqlite3_shutdown();

	Tcl_FindExecutable(argv[0]);
	interp = Tcl_CreateInterp();

#if TCLSH==2
	sqlite3_config(SQLITE_CONFIG_SINGLETHREAD);
#endif

	init_all(interp);
	if( argc>=2 ){
		int i;
		char zArgc[32];
		sqlite3_snprintf(sizeof(zArgc), zArgc, "%d", argc-(3-TCLSH));
		Tcl_SetVar(interp,"argc", zArgc, TCL_GLOBAL_ONLY);
		Tcl_SetVar(interp,"argv0",argv[1],TCL_GLOBAL_ONLY);
		Tcl_SetVar(interp,"argv", "", TCL_GLOBAL_ONLY);
		for(i=3-TCLSH; i<argc; i++){
			Tcl_SetVar(interp, "argv", argv[i],
				TCL_GLOBAL_ONLY | TCL_LIST_ELEMENT | TCL_APPEND_VALUE);
		}
		if( TCLSH==1 && Tcl_EvalFile(interp, argv[1])!=TCL_OK ){
			const char *zInfo = Tcl_GetVar(interp, "errorInfo", TCL_GLOBAL_ONLY);
			if( zInfo==0 ) zInfo = Tcl_GetStringResult(interp);
			fprintf(stderr,"%s: %s\n", *argv, zInfo);
			return 1;
		}
	}
	if( TCLSH==2 || argc<=1 ){
		Tcl_GlobalEval(interp, tclsh_main_loop());
	}
	return 0;
}
#endif /* TCLSH */
