//#define VISUAL
#include <RuntimeHost.h>
#include <FallocHost.h>

void __testSystemDataBtree(cudaDeviceHeap &r);
void __testSystemDataPager(cudaDeviceHeap &r);
void __testSystemDataVdbe(cudaDeviceHeap &r);

#if __HIPCC__
void GMain(cudaDeviceHeap &r) {
#else
void main(int argc, char **argv) { cudaDeviceHeap r; memset(&r, 0, sizeof(r));
#endif
	__testSystemDataVdbe(r);
}

#if __HIPCC__
void __main(cudaDeviceHeap &r)
{	
	cudaErrorCheck(cudaDeviceHeapSelect(r));
	GMain(r); cudaDeviceHeapSynchronize(r);
}

int main(int argc, char **argv)
{
	cudaErrorCheck(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
	int deviceId = gpuGetMaxGflopsDeviceId();
	cudaErrorCheck(hipSetDevice(deviceId));
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024*8));

	cudaDeviceHeap deviceHeap = cudaDeviceHeapCreate(); //256, 4096);
	//cudaDeviceFalloc fallocHost = cudaDeviceFallocCreate(100, 1024);

	// First initialize OpenGL context, so we can properly set the GL for CUDA. This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	//IVisualRender *render = new RuntimeVisualRender(deviceHeap);
	//IVisualRender *render = new FallocVisualRender(fallocHost);
	//if (!Visual::InitGL(render, &argc, argv)) return 0;
	//cudaErrorCheck(cudaGLSetGLDevice(deviceId));

	// run
	__main(deviceHeap);
	//Visual::Main();
	//Visual::Dispose();
	
	cudaDeviceHeapDestroy(deviceHeap);
	//cudaDeviceFallocDestroy(fallocHost);

	hipDeviceReset();
	printf("\nEnd."); char c; scanf("%c", &c);
	return 0;
}
#endif