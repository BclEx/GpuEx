//#define VISUAL
#include <RuntimeHost.h>
#include <FallocHost.h>

void __testSystemDataBtree(cudaDeviceHeap &r);
void __testSystemDataPager(cudaDeviceHeap &r);
void __testSystemDataVdbe(cudaDeviceHeap &r);

#if __HIPCC__
void GMain(cudaDeviceHeap &r) {
#else
void main(int argc, char **argv) { cudaDeviceHeap r; memset(&r, 0, sizeof(r));
#endif
	__testSystemDataVdbe(r);
}

#if __HIPCC__
void __main(cudaDeviceHeap &r)
{	
	cudaCheckErrors(cudaDeviceHeapSelect(r), );
	GMain(r); cudaDeviceHeapSynchronize(r);
}

int main(int argc, char **argv)
{
	cudaCheckErrors(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax), return -1);
	int deviceId = gpuGetMaxGflopsDeviceId();
	cudaCheckErrors(hipSetDevice(deviceId), return -2);
	cudaCheckErrors(hipDeviceSetLimit(hipLimitStackSize, 1024*10), return -2);

	cudaDeviceHeap deviceHeap = cudaDeviceHeapCreate(256, 4096);
	//cudaDeviceFalloc fallocHost = cudaDeviceFallocCreate(100, 1024);

	// First initialize OpenGL context, so we can properly set the GL for CUDA. This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	//IVisualRender *render = new RuntimeVisualRender(deviceHeap);
	//IVisualRender *render = new FallocVisualRender(fallocHost);
	//if (!Visual::InitGL(render, &argc, argv)) return 0;
	//cudaCheckErrors(cudaGLSetGLDevice(deviceId), return -3);

	// run
	__main(deviceHeap);
	//Visual::Main();
	//Visual::Dispose();
	
	cudaDeviceHeapDestroy(deviceHeap);
	//cudaDeviceFallocDestroy(fallocHost);

	hipDeviceReset();
	printf("\nEnd."); char c; scanf("%c", &c);
	return 0;
}
#endif