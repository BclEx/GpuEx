// This file contains code that modified the OS layer in order to simulate different device types (by overriding the return values of the  xDeviceCharacteristics() and xSectorSize() methods).
#include <Core+Vdbe\Core+Vdbe.cu.h>
#include <new.h>

// Maximum pathname length supported by the devsym backend.
#define DEVSYM_MAX_PATHNAME 512
// Name used to identify this VFS.
#define DEVSYM_VFS_NAME "devsym"

struct DevsymGlobal
{
	VSystem *Vfs;
	VFile::IOCAP DeviceChar;
	int SectorSize;
};
__device__ struct DevsymGlobal _g = { nullptr, (VFile::IOCAP)0, 512 };

class DevSymVFile : public VFile
{
public:
	VFile *Real; // The "real" underlying file descriptor
public:
	__device__ virtual RC Close_() { return Real->Close(); }
	__device__ virtual RC Read(void *buffer, int amount, int64 offset) { return Real->Read(buffer, amount, offset); }
	__device__ virtual RC Write(const void *buffer, int amount, int64 offset) { return Real->Write(buffer, amount, offset); }
	__device__ virtual RC Truncate(int64 size) { return Real->Truncate(size); }
	__device__ virtual RC Sync(SYNC flags) { return Real->Sync(flags); }
	__device__ virtual RC get_FileSize(int64 &size) { return Real->get_FileSize(size); }

	__device__ virtual RC Lock(LOCK lock) { return Real->Lock(lock); }
	__device__ virtual RC Unlock(LOCK lock) { return Real->Unlock(lock); }
	__device__ virtual RC CheckReservedLock(int &lock) { return Real->CheckReservedLock(lock); }
	__device__ virtual RC FileControl(FCNTL op, void *arg) { return Real->FileControl(op, arg); }

	__device__ virtual uint get_SectorSize() { return _g.SectorSize; }
	__device__ virtual IOCAP get_DeviceCharacteristics() { return _g.DeviceChar; }

	__device__ virtual RC ShmLock(int offset, int n, SHM flags) { return Real->ShmLock(offset, n, flags); }
	__device__ virtual void ShmBarrier() { Real->ShmBarrier(); }
	__device__ virtual RC ShmUnmap(bool deleteFlag) { return Real->ShmUnmap(deleteFlag); }
	__device__ virtual RC ShmMap(int region, int sizeRegion, bool isWrite, void volatile **pp) { return Real->ShmMap(region, sizeRegion, isWrite, pp); }
};

class DevSymVSystem : public VSystem
{
public:
	VFile *Real; // The "real" underlying file descriptor
public:
	__device__ virtual VFile *_AttachFile(void *buffer) { return new (buffer) DevSymVFile(); }
	__device__ virtual RC Open(const char *path, VFile *file, OPEN flags, OPEN *outFlags)
	{
		DevSymVFile *p = (DevSymVFile *)file;
		p->Real = (VFile *)&p[1];
		RC rc = _g.Vfs->Open(path, p->Real, flags, outFlags);
		p->Opened = p->Real->Opened;
		return rc;
	}
	__device__ virtual RC Delete(const char *path, bool syncDirectory) { return _g.Vfs->Delete(path, syncDirectory); }
	__device__ virtual RC Access(const char *path, ACCESS flags, int *outRC) { return _g.Vfs->Access(path, flags, outRC); }
	__device__ virtual RC FullPathname(const char *path, int pathOutLength, char *pathOut) { return _g.Vfs->FullPathname(path, pathOutLength, pathOut); }

#ifndef OMIT_LOAD_EXTENSION
	__device__ virtual void *DlOpen(const char *filename) { return _g.Vfs->DlOpen(filename); }
	__device__ virtual void DlError(int bufLength, char *buf) { return _g.Vfs->DlError(bufLength, buf); }
	__device__ virtual void (*DlSym(void *handle, const char *symbol))() { return _g.Vfs->DlSym(handle, symbol); }
	__device__ virtual void DlClose(void *handle) { return _g.Vfs->DlClose(handle); }
#endif
	__device__ virtual int Randomness(int bufLength, char *buf) { return _g.Vfs->Randomness(bufLength, buf); }
	__device__ virtual int Sleep(int microseconds) { return _g.Vfs->Sleep(microseconds); }
	//__device__ virtual RC CurrentTimeInt64(int64 *now);
	__device__ virtual RC CurrentTime(double *now) { return _g.Vfs->CurrentTime(now); }
	//__device__ virtual RC GetLastError(int bufLength, char *buf);
};

// This procedure registers the devsym vfs with SQLite. If the argument is true, the devsym vfs becomes the new default vfs. It is the only publicly
// available function in this file.
__device__ static unsigned char _devsymVfsBuf[sizeof(DevSymVSystem)];
__device__ static DevSymVSystem *_devsymVfs;
__device__ void devsym_register(VFile::IOCAP deviceChar, int sectorSize)
{
	if (!_g.Vfs)
	{
		_g.Vfs = VSystem::FindVfs(nullptr);
		_devsymVfs = new (_devsymVfsBuf) DevSymVSystem();
		_devsymVfs->SizeOsFile = sizeof(DevSymVFile) + _g.Vfs->SizeOsFile;
		_devsymVfs->MaxPathname = DEVSYM_MAX_PATHNAME;
		_devsymVfs->Name = DEVSYM_VFS_NAME;
		VSystem::RegisterVfs(_devsymVfs, false);
	}
	_g.DeviceChar = ((int)deviceChar >= 0 ? deviceChar : (VFile::IOCAP)0);
	_g.SectorSize = (sectorSize >= 0 ? sectorSize : 512);
}
