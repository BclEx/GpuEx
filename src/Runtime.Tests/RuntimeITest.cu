#include "hip/hip_runtime.h"
#include <Runtime.cu.h>

#define _TEST(id) \
	__global__ void runtimeTest##id(void *r); \
	void runtimeTest##id##_host(cudaDeviceHeap &r) { cudaDeviceHeapSelect(r); runtimeTest##id<<<1, 1>>>(r.heap); cudaDeviceHeapSynchronize(r); } \
	__global__ void runtimeTest##id(void *r) \
{ \
	_runtimeSetHeap(r);

//////////////////////////////////////////////////

// printf outputs
_TEST(0) {
	_printf("test");
}}
