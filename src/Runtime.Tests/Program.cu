#include <string.h>
#include <RuntimeHost.h>
//http://thelegendofrandom.com/blog/archives/2231

void __testRuntime(cudaDeviceHeap &r);
void __testRuntimeEx(cudaDeviceHeap &r);
//void __testRegex1(cudaDeviceHeap &r);

#if __HIPCC__
void GMain(cudaDeviceHeap &r) {
#else
void main(int argc, char **argv) { cudaDeviceHeap r; memset(&r, 0, sizeof(r));
#endif
__testRuntime(r);
__testRuntimeEx(r);
//__testRegex1(r);
}

#if __HIPCC__
void __main(cudaDeviceHeap &r)
{	
	cudaDeviceHeapSelect(r);
	GMain(r); cudaDeviceHeapSynchronize(r);
}

cudaDeviceHeap _deviceHeap;
int main(int argc, char **argv)
{
	cudaErrorCheck(hipSetDeviceFlags(hipDeviceMapHost));
	int deviceId = gpuGetMaxGflopsDeviceId();
	cudaErrorCheck(hipSetDevice(deviceId));
	//cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024*4));

	_deviceHeap = cudaDeviceHeapCreate(256, 4096);

	// First initialize OpenGL context, so we can properly set the GL for CUDA. This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	//IVisualRender *render = new RuntimeVisualRender(deviceHeap);
	//if (!Visual::InitGL(render, &argc, argv)) return 0;
	//cudaErrorCheck(cudaGLSetGLDevice(deviceId));

	// run
	__main(_deviceHeap);
	//Visual::Main();
	//Visual::Dispose();

	cudaDeviceHeapDestroy(_deviceHeap);

	hipDeviceReset();
	printf("\nEnd"); char c; scanf("%c", &c);
	return 0;
}
#endif